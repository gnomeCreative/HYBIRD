#include "hip/hip_runtime.h"
#include "LB2.h"

#include <cstdlib>

#include "cuda_helper.h"

#include "DEM.h"

/**
 * Storage for static members must be defined
 */
std::unique_ptr<CubTempMem> CubTempMem::_singletonT;
std::unique_ptr<CubTempMem> CubTempMem::_singletonB;

/**
 * (Temporary) DEM data synchronisation
 * Reformat DEM data to structure of arrays (for CPU), and copy it to device (for CUDA)
 */
template<>
bool LB2::syncElements<CPU>(const elmtList &elements) {
    bool componentsHasGrown = false;
    if (h_elements.count < elements.size()) {
        // Grow host buffers
         if (h_elements.x1) {
             free(h_elements.x1);
             free(h_elements.wGlobal);
             free(h_elements.FHydro);
             free(h_elements.MHydro);
             free(h_elements.fluidVolume);
         }
         h_elements.x1 = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.wGlobal = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.FHydro = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.MHydro = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.fluidVolume = (double*)malloc(elements.size() * sizeof(double));
    }
    // Update size
    h_elements.count = static_cast<unsigned int>(elements.size());
    // Repackage host particle data from array of structures, to structure of arrays
     for (unsigned int i = 0; i < h_elements.count; ++i) {
         h_elements.x1[i] = elements[i].x1;
         h_elements.wGlobal[i] = elements[i].wGlobal;
         h_elements.FHydro[i] = elements[i].FHydro;
         // h_elements.MHydro[i] = elements[i].MHydro; // This is zero'd before use in latticeBoltzmannStep()
         // h_elements.fluidVolume[i] = elements[i].fluidVolume; // This is zero'd before use in latticeBoltzmannStep()
     }
    // Construct the components storage
    {
        // Allocate memory for componentsData
        unsigned int totalComponents = 0;
        for (const auto& e : elements)
            totalComponents += static_cast<unsigned int>(e.components.size());
        if (!h_elements.componentsIndex || totalComponents >= h_elements.componentsIndex[elements.size()]) {
            if (h_elements.componentsData)
                free(h_elements.componentsData);
            h_elements.componentsData = (unsigned int*)malloc(totalComponents * sizeof(unsigned int));
            componentsHasGrown = true;
        }
        // Allocate componentsIndex if first pass
        if (!h_elements.componentsIndex)
            h_elements.componentsIndex = (unsigned int*)malloc((elements.size() + 1) * sizeof(unsigned int));
        // Fill componentsIndex and componentsData
        totalComponents = 0;
        for (int i = 0; i < elements.size(); ++i) {
            h_elements.componentsIndex[i] = totalComponents;
            if (!elements[i].components.empty()) {
                memcpy(h_elements.componentsData + totalComponents, elements[i].components.data(), elements[i].components.size() * sizeof(unsigned int));
                totalComponents += static_cast<unsigned int>(elements[i].components.size());
            }
        }
        h_elements.componentsIndex[elements.size()] = totalComponents;
    }
    return componentsHasGrown;
}
template<>
void LB2::syncParticles<CPU>(const particleList &particles) {
    if (h_particles.count < particles.size()) {
        // Grow host buffers
        if (h_particles.clusterIndex) {
            free(h_particles.clusterIndex);
            free(h_particles.r);
            free(h_particles.x0);
            free(h_particles.radiusVec);
        }
        h_particles.clusterIndex = (unsigned int*)malloc(particles.size() * sizeof(unsigned int));
        h_particles.r = (double*)malloc(particles.size() * sizeof(double));
        h_particles.x0 = (tVect*)malloc(particles.size() * sizeof(tVect));
        h_particles.radiusVec = (tVect*)malloc(particles.size() * sizeof(tVect));
    }
    // Update size
    h_particles.count = static_cast<unsigned int>(particles.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (int i = 0; i < h_particles.count; ++i) {
        h_particles.clusterIndex[i] = particles[i].clusterIndex;
        h_particles.r[i] = particles[i].r;
        h_particles.x0[i] = particles[i].x0;
        h_particles.radiusVec[i] = particles[i].radiusVec;
    }
}
template<>
void LB2::syncCylinders<CPU>(const cylinderList &cylinders) {
    if (h_cylinders.count < cylinders.size()) {
        // Grow host buffers
        if (h_cylinders.p1) {
            free(h_cylinders.p1);
            free(h_cylinders.p2);
            free(h_cylinders.R);
            free(h_cylinders.naxes);
            free(h_cylinders.omega);
            free(h_cylinders.moving);
        }
        h_cylinders.p1 = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.p2 = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.R = (double*)malloc(cylinders.size() * sizeof(double));
        h_cylinders.naxes = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.omega = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.moving = (bool*)malloc(cylinders.size() * sizeof(bool));
    }
    // Update size
    h_cylinders.count = static_cast<unsigned int>(cylinders.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_cylinders.count; ++i) {
        h_cylinders.p1[i] = cylinders[i].p1;
        h_cylinders.p2[i] = cylinders[i].p2;
        h_cylinders.R[i] = cylinders[i].R;
        h_cylinders.naxes[i] = cylinders[i].naxes;
        h_cylinders.omega[i] = cylinders[i].omega;
        h_cylinders.moving[i] = cylinders[i].moving;
    }
}
template<>
void LB2::syncWalls<CPU>(const wallList &walls) {
    if (h_walls.count < walls.size()) {
        // Grow host buffers
        if (h_walls.n) {
            free(h_walls.n);
            free(h_walls.p);
            free(h_walls.rotCenter);
            free(h_walls.omega);
            free(h_walls.vel);
            free(h_walls.FHydro);
        }
        h_walls.n = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.p = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.rotCenter = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.omega = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.vel = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.FHydro = (tVect*)malloc(walls.size() * sizeof(tVect));
    }
    // Update size
    h_walls.count = static_cast<unsigned int>(walls.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_walls.count; ++i) {
        h_walls.n[i] = walls[i].n;
        h_walls.p[i] = walls[i].p;
        h_walls.rotCenter[i] = walls[i].rotCenter;
        h_walls.omega[i] = walls[i].omega;
        h_walls.vel[i] = walls[i].vel;
        // h_walls.FHydro[i] = walls[i].FHydro; // Zero'd before use in streaming()
    }
}
template<>
void LB2::syncObjects<CPU>(const objectList &objects) {
    if (h_objects.count < objects.size()) {
        // Grow host buffers
        if (h_objects.r) {
            free(h_objects.r);
            free(h_objects.x0);
            free(h_objects.x1);
            free(h_objects.FHydro);
        }
        h_objects.r = (double*)malloc(objects.size() * sizeof(double));
        h_objects.x0 = (tVect*)malloc(objects.size() * sizeof(tVect));
        h_objects.x1 = (tVect*)malloc(objects.size() * sizeof(tVect));
        h_objects.FHydro = (tVect*)malloc(objects.size() * sizeof(tVect));
    }
    // Update size
    h_objects.count = static_cast<unsigned int>(objects.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_objects.count; ++i) {
        h_objects.r[i] = objects[i].r;
        h_objects.x0[i] = objects[i].x0;
        h_objects.x1[i] = objects[i].x1;
        // h_objects.FHydro[i] = objects[i].FHydro; // Zero'd before use in streaming()
    }
}
#ifdef USE_CUDA
template<>
bool LB2::syncElements<CUDA>(const elmtList &elements) {
    if (!d_elements) {
        CUDA_CALL(hipMalloc(&d_elements, sizeof(Element2)));
    }
    // @todo copy hd_elements to d_elements
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    bool componentsHasGrown = this->syncElements<CPU>(elements);
    bool updateDeviceStruct = false;
    if (hd_elements.count < elements.size()) {
        if (hd_elements.x1) {
            CUDA_CALL(hipFree(hd_elements.x1));
            CUDA_CALL(hipFree(hd_elements.wGlobal));
            CUDA_CALL(hipFree(hd_elements.FHydro));
            CUDA_CALL(hipFree(hd_elements.MHydro));
            CUDA_CALL(hipFree(hd_elements.fluidVolume));
        }
        // Initially allocate device buffers except components
        CUDA_CALL(hipMalloc(&hd_elements.x1, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.wGlobal, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.FHydro, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.MHydro, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.fluidVolume, hd_elements.count * sizeof(double)));
        updateDeviceStruct = true;
    }
    if (componentsHasGrown || !hd_elements.componentsIndex) {
        // Allocate components
        if (hd_elements.componentsIndex)
            CUDA_CALL(hipFree(hd_elements.componentsIndex));
        if (hd_elements.componentsData)
            CUDA_CALL(hipFree(hd_elements.componentsData));
        // Allocate components
        CUDA_CALL(hipMalloc(&hd_elements.componentsIndex, (h_elements.count + 1) * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&hd_elements.componentsData, h_elements.componentsIndex[h_elements.count] * sizeof(unsigned int)));
        updateDeviceStruct = true;
    }
    // Update size
    hd_elements.count = elements.size();
    if (updateDeviceStruct) {
        // Copy updated device pointers to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(d_elements, &hd_elements, sizeof(Element2), hipMemcpyHostToDevice));
    } else {
        // Copy updated device pointers to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(&d_elements->count, &hd_elements.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_elements.x1, &h_elements.x1, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.wGlobal, &h_elements.wGlobal, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.FHydro, &h_elements.FHydro, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_elements.MHydro, &h_elements.MHydro, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice)); // This is zero'd before use in latticeBoltzmannStep()
    // CUDA_CALL(hipMemcpy(hd_elements.fluidVolume, &h_elements.fluidVolume, h_elements.count * sizeof(double), hipMemcpyHostToDevice)); // This is zero'd before use in latticeBoltzmannStep()
    CUDA_CALL(hipMemcpy(hd_elements.componentsIndex, h_elements.componentsIndex, (h_elements.count + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.componentsData, h_elements.componentsData, h_elements.componentsIndex[h_elements.count] * sizeof(unsigned int), hipMemcpyHostToDevice));
    return componentsHasGrown;
}
template<>
void LB2::syncParticles<CUDA>(const particleList &particles) {
    if (!d_particles) {
        CUDA_CALL(hipMalloc(&d_particles, sizeof(Particle2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncParticles<CPU>(particles);
    if (hd_particles.count < particles.size()) {
        // Grow device buffers
        if (hd_particles.clusterIndex) {
            CUDA_CALL(hipFree(hd_particles.clusterIndex));
            CUDA_CALL(hipFree(hd_particles.r));
            CUDA_CALL(hipFree(hd_particles.x0));
            CUDA_CALL(hipFree(hd_particles.radiusVec));
        }
        CUDA_CALL(hipMalloc(&hd_particles.clusterIndex, h_particles.count * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&hd_particles.r, h_particles.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_particles.x0, h_particles.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_particles.radiusVec, h_particles.count * sizeof(tVect)));
        hd_particles.count = h_particles.count;
        // Copy updated device pointers to device (@todo When/where is d_particles allocated??)
        CUDA_CALL(hipMemcpy(d_particles, &h_particles, sizeof(Particle2), hipMemcpyHostToDevice));
    } else if(hd_particles.count != particles.size()) {
        // Buffer has shrunk, so just update size
        hd_particles.count = static_cast<unsigned int>(particles.size());
        // Copy updated particle count to device
        CUDA_CALL(hipMemcpy(&d_particles->count, &h_particles.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_particles.clusterIndex, h_particles.clusterIndex, h_particles.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.r, h_particles.r, h_particles.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.x0, h_particles.x0, h_particles.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.radiusVec, h_particles.radiusVec, h_particles.count * sizeof(tVect), hipMemcpyHostToDevice));
}
template<>
void LB2::syncCylinders<CUDA>(const cylinderList &cylinders) {
    if (!d_cylinders) {
        CUDA_CALL(hipMalloc(&d_cylinders, sizeof(Cylinder2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncCylinders<CPU>(cylinders);
    if (hd_cylinders.count < cylinders.size()) {
        // Grow device buffers
        if (hd_cylinders.p1) {
            CUDA_CALL(hipFree(hd_cylinders.p1));
            CUDA_CALL(hipFree(hd_cylinders.p2));
            CUDA_CALL(hipFree(hd_cylinders.R));
            CUDA_CALL(hipFree(hd_cylinders.naxes));
            CUDA_CALL(hipFree(hd_cylinders.omega));
            CUDA_CALL(hipFree(hd_cylinders.moving));
        }
        CUDA_CALL(hipMalloc(&hd_cylinders.p1, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.p2, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.R, h_cylinders.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_cylinders.naxes, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.omega, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.moving, h_cylinders.count * sizeof(bool)));
        hd_cylinders.count = h_cylinders.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_cylinders, &hd_cylinders, sizeof(Cylinder2), hipMemcpyHostToDevice));
    } else if(hd_cylinders.count != cylinders.size()) {
        // Buffer has shrunk, so just update size
        hd_cylinders.count = static_cast<unsigned int>(cylinders.size());
        // Copy updated particle count to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(&d_cylinders->count, &h_walls.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_cylinders.p1, h_cylinders.p1, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.p2, h_cylinders.p2, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.R, h_cylinders.R, h_cylinders.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.naxes, h_cylinders.naxes, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.omega, h_cylinders.omega, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.moving, h_cylinders.moving, h_cylinders.count * sizeof(bool), hipMemcpyHostToDevice));
}
template<>
void LB2::syncWalls<CUDA>(const wallList &walls) {
    if (!d_walls) {
        CUDA_CALL(hipMalloc(&d_walls, sizeof(Wall2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncWalls<CPU>(walls);
    if (hd_walls.count < walls.size()) {
        // Grow device buffers
        if (hd_walls.n) {
            CUDA_CALL(hipFree(hd_walls.n));
            CUDA_CALL(hipFree(hd_walls.p));
            CUDA_CALL(hipFree(hd_walls.rotCenter));
            CUDA_CALL(hipFree(hd_walls.omega));
            CUDA_CALL(hipFree(hd_walls.vel));
            CUDA_CALL(hipFree(hd_walls.FHydro));
        }
        CUDA_CALL(hipMalloc(&hd_walls.n, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.p, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.rotCenter, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.omega, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.vel, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.FHydro, h_walls.count * sizeof(tVect)));
        hd_walls.count = h_walls.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_walls, &hd_walls, sizeof(Wall2), hipMemcpyHostToDevice));
    } else if(hd_walls.count != walls.size()) {
        // Buffer has shrunk, so just update size
        hd_walls.count = static_cast<unsigned int>(walls.size());
        // Copy updated particle count to device (@todo When/where is d_walls allocated??)
        CUDA_CALL(hipMemcpy(&d_walls->count, &h_walls.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_walls.n, h_walls.n, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.p, h_walls.p, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.rotCenter, h_walls.rotCenter, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.omega, h_walls.omega, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.vel, h_walls.vel, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_walls.FHydro, h_walls.FHydro, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice)); // Zero'd before use in streaming()
}
template<>
void LB2::syncObjects<CUDA>(const objectList &walls) {
    if (!d_objects) {
        CUDA_CALL(hipMalloc(&d_objects, sizeof(Object2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncObjects<CPU>(walls);
    if (hd_objects.count < walls.size()) {
        // Grow device buffers
        if (hd_objects.r) {
            CUDA_CALL(hipFree(hd_objects.r));
            CUDA_CALL(hipFree(hd_objects.x0));
            CUDA_CALL(hipFree(hd_objects.x1));
            CUDA_CALL(hipFree(hd_objects.FHydro));
        }
        CUDA_CALL(hipMalloc(&hd_objects.r, h_objects.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_objects.x0, h_objects.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_objects.x1, h_objects.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_objects.FHydro, h_objects.count * sizeof(tVect)));
        hd_objects.count = h_objects.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_objects, &hd_objects, sizeof(Object2), hipMemcpyHostToDevice));
    } else if(hd_objects.count != walls.size()) {
        // Buffer has shrunk, so just update size
        hd_objects.count = static_cast<unsigned int>(walls.size());
        // Copy updated particle count to device (@todo When/where is d_objects allocated??)
        CUDA_CALL(hipMemcpy(&d_objects->count, &h_objects.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_objects.r, h_objects.r, h_objects.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_objects.x0, h_objects.x0, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_objects.x1, h_objects.x1, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_objects.FHydro, h_objects.FHydro, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice)); // Zero'd before use in streaming()
}
#endif


/**
 * initializeParticleBoundaries()
 */
__host__ __device__ __forceinline__ double common_initializeParticleBoundaries(const unsigned int i, Node2* nodes, Particle2* particles) {
    // Fetch the index of the (active) node being processed
    const unsigned int an_i = nodes->activeI[i];
    const tVect node_position = nodes->getPosition(an_i);
    for (unsigned int p_i = 0; p_i < particles->count; ++p_i) {
        const tVect convertedPosition = particles->x0[p_i] / PARAMS.unit.Length;
        // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
        const double convertedRadius = particles->r[p_i] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
        if (node_position.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
            nodes->setInsideParticle(an_i, true);
            nodes->solidIndex[an_i] = p_i;
            return nodes->mass[an_i];  // @todo in original code it doesn't break after setting
        }
    }
    return 0.0;
}
template<>
double LB2::initializeParticleBoundaries<CPU>() {
    // Reset all nodes to outside
    memset(hd_nodes.p, 0, h_nodes.count * sizeof(bool));

    // @todo can we parallelise at a higher level?
    double totalParticleMass = 0;
#pragma omp parallel for reduction(+:totalParticleMass) 
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        totalParticleMass += common_initializeParticleBoundaries(i, d_nodes, d_particles);
    }
    return totalParticleMass;
}
#ifdef USE_CUDA
__global__ void d_initializeParticleBoundaries(Node2* d_nodes, Particle2* d_particles, double *node_in_particle_mass) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    const double t = common_initializeParticleBoundaries(i, d_nodes, d_particles);

    if (t != 0.0) {
        atomicAdd(node_in_particle_mass, t);
    }
}
template<>
double LB2::initializeParticleBoundaries<CUDA>() {
    // Reset all nodes to outside
    CUDA_CALL(hipMemset(hd_nodes.p, 0, h_nodes.count * sizeof(bool)));
    // Initialise reduction variable
    auto &t = CubTempMem::GetTempSingleton();
    t.resize(sizeof(double));
    double *d_return = static_cast<double*>(t.getPtr());
    double h_return = 0;
    CUDA_CALL(hipMemcpy(d_return, &h_return, sizeof(double), hipMemcpyHostToDevice));

    // Launch cuda kernel to update
    // @todo Try unrolling this, so 1 thread per node+particle combination (2D launch?)
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_initializeParticleBoundaries << <gridSize, blockSize >> > (d_nodes, d_particles, d_return);
    CUDA_CHECK();

    // Copy back return value
    CUDA_CALL(hipMemcpy(&h_return, d_return, sizeof(double), hipMemcpyDeviceToHost));
    return h_return;
}
#endif

/**
 * findNewActive()
 */
__host__ __device__ __forceinline__ void common_findNewActive(const unsigned int i, Node2* nodes, Particle2* particles, Element2* elements) {
    // Fetch the index of the (active) node being processed
    const unsigned int an_i = nodes->activeI[i];
    if (nodes->p[an_i]) {
        const tVect nodePosition = nodes->getPosition(an_i);
        // solid index to identify cluster
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // in this case check if it has been uncovered (must be out of all particles of the cluster) - we start with a true hypothesis
        // cycling through component particles
        const unsigned int first_component = elements->componentsIndex[clusterIndex];
        const unsigned int last_component = elements->componentsIndex[clusterIndex + 1];
        for (unsigned int j = first_component; j < last_component; ++j) {
            // getting indexes from particle composing the cluster
            const unsigned int componentIndex = elements->componentsData[j];
            // checking if it has been uncovered in component j of the cluster
            // radius need to be increased by half a lattice unit
            // this is because solid boundaries are located halfway between solid and fluid nodes
            const tVect convertedPosition = particles->x0[componentIndex] / PARAMS.unit.Length;
            // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
            const double convertedRadius = particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
            if (nodePosition.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
                // if the node is still inside the element, the hypothesis of new active is not true anymore
                // and we can get out of the cycle
                return;
            }
        }
        // turning up the cell as we didn't exit early
        nodes->setInsideParticle(an_i, false);
    }
}
template<>
void LB2::findNewActive<CPU>() {
    // @todo can we parallelise at a higher level?
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        common_findNewActive(i, d_nodes, d_particles, d_elements);
    }
}
#ifdef USE_CUDA
__global__ void d_findNewActive(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    common_findNewActive(i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::findNewActive<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_findNewActive << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * findNewSolid()
 */
__host__ __device__ __forceinline__ void common_findNewSolid(const unsigned int i, Node2* nodes, Particle2* particles, Element2* elements) {
    const unsigned int an_i = nodes->activeI[i];
    if (nodes->isInsideParticle(an_i)) {  // If node is inside particle
        // solid index to identify cluster
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // cycle through first neighbors
        const unsigned int nodeCount = nodes->count;
        for (int k = 1; k < lbmMainDirec; ++k) {
            const unsigned int l_i = nodes->d[nodeCount * k + an_i];
            if (l_i != std::numeric_limits<unsigned int>::max()) {
                // checking if solid particle is close to an active one -> we have an active node to check
                if (!nodes->isInsideParticle(l_i) && nodes->isActive(l_i)) {
                    const tVect linkPosition = nodes->getPosition(l_i);
                    // check if neighbors has been covered (by any of the particles of the cluster) - we start with a false hypothesis
                    // cycling through all components of the cluster
                    const unsigned int first_component = elements->componentsIndex[clusterIndex];
                    const unsigned int last_component = elements->componentsIndex[clusterIndex + 1];
                    for (unsigned int j = first_component; j < last_component; ++j) {
                        // getting component particle index
                        const unsigned int componentIndex = elements->componentsData[j];
                        // check if it getting inside
                        // radius need to be increased by half a lattice unit
                        // this is because solid boundaries are located halfway between solid and fluid nodes
                        // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
                        if (linkPosition.insideSphere(particles->x0[componentIndex] / PARAMS.unit.Length, particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length)) { //-0.5?
                            // if so, then the false hypothesis does not hold true anymore
                            nodes->solidIndex[l_i] = componentIndex;
                            // By setting particle to inside, it won't be checked again, newSolidNodes hence becomes redundant
                            nodes->setInsideParticle(l_i, true);  // @todo Is this a race condition? Multiple nodes may share a link node?
                            // and we exit the cycle
                            break;
                        }
                    }
                }
            }
        }
    }
}
template<>
void LB2::findNewSolid<CPU>() {
    // @todo can we parallelise at a higher level?
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        common_findNewSolid(i, d_nodes, d_particles, d_elements);
    }
}
#ifdef USE_CUDA
__global__ void d_findNewSolid(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    common_findNewSolid(i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::findNewSolid<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_findNewSolid, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_findNewSolid << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * checkNewInterfaceParticles()
 */
__host__ __device__ __forceinline__ void common_checkNewInterfaceParticles(const unsigned int e_i, Node2* nodes, Particle2* particles, Element2* elements) {
    // INITIAL PARTICLE POSITION ////////////////////////
    if (elements->FHydro[e_i].norm2() == 0.0) {
        const unsigned int first_component = elements->componentsIndex[e_i];
        const unsigned int last_component = elements->componentsIndex[e_i + 1];
        for (unsigned int n = first_component; n < last_component; ++n) {
            const unsigned int componentIndex = elements->componentsData[n];
            const tVect convertedPosition = particles->x0[componentIndex] / PARAMS.unit.Length;
            // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
            const double convertedRadius = particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
            for (unsigned int i_i = 0; i_i < nodes->interfaceCount; ++i_i) {
                const unsigned int nodeHere = nodes->interfaceI[i_i];
                if (!nodes->isInsideParticle(nodeHere)) {
                    // checking if node is inside a particle
                    const tVect nodePosition = nodes->getPosition(nodeHere);
                    if (nodePosition.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
                        nodes->setInsideParticle(nodeHere, true);
                        nodes->solidIndex[nodeHere] = componentIndex;
                    }
                }
            }
        }
    }
}
template<>
void LB2::checkNewInterfaceParticles<CPU>() {
#pragma omp parallel for
    for (unsigned int e_i = 0; e_i < d_elements->count; ++e_i) {
        common_checkNewInterfaceParticles(e_i, d_nodes, d_particles, d_elements);
    }
}
#ifdef USE_CUDA
__global__ void d_checkNewInterfaceParticles(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to element 
    const unsigned int e_i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (e_i >= d_elements->count) return;
    // Pass the active node index to the common implementation
    common_checkNewInterfaceParticles(e_i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::checkNewInterfaceParticles<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_checkNewInterfaceParticles, 0, h_elements.count);
    // Round up to accommodate required threads
    gridSize = (h_elements.count + blockSize - 1) / blockSize;
    // @todo Are there more elements or particles? This may want to be inverted, and we can go straight to particles rather than components?
    d_checkNewInterfaceParticles << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * reconstruct()
 * computeHydroForces()
 * collision()
 */
__host__ __device__ __forceinline__ void common_computeHydroForces(const unsigned int an_i, Node2* nodes, Particle2* particles, Element2* elements) {
    // resetting hydrodynamic forces on nodes
    nodes->hydroForce[an_i].reset();
    if (nodes->isInsideParticle(an_i)) {
        // getting the index of the particle to compute force in the right object
        const unsigned int index = an_i;
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // calculating velocity of the solid boundary at the node (due to rotation of particles)
        // vectorized radius (real units)
        const tVect radius = nodes->getPosition(index) - particles->x0[particleIndex] / PARAMS.unit.Length + particles->radiusVec[particleIndex] / PARAMS.unit.Length;
        // update velocity of the particle node (u=v_center+omega x radius) (real units)
        const tVect localVel = elements->x1[clusterIndex] / PARAMS.unit.Speed + (elements->wGlobal[clusterIndex].cross(radius)) / PARAMS.unit.AngVel;

        // calculate differential velocity
        const tVect diffVel = nodes->age[an_i] * nodes->age[an_i] * nodes->liquidFraction(an_i) * (nodes->u[an_i] - localVel);

        // force on fluid
        nodes->hydroForce[an_i] += -1.0 * diffVel;

        // force on particle
#ifdef __CUDA_ARCH__
        // CUDA atomics
        atomicAdd(&elements->fluidVolume[clusterIndex], nodes->mass[an_i]);
        atomicAdd(&elements->FHydro[clusterIndex].x, 1.0 * diffVel.x);
        atomicAdd(&elements->FHydro[clusterIndex].y, 1.0 * diffVel.y);
        atomicAdd(&elements->FHydro[clusterIndex].z, 1.0 * diffVel.z);
        const tVect t = 1.0 * radius.cross(diffVel);
        atomicAdd(&elements->MHydro[clusterIndex].x, t.x);
        atomicAdd(&elements->MHydro[clusterIndex].y, t.y);
        atomicAdd(&elements->MHydro[clusterIndex].z, t.z);
#else
        // CPU atomics
#pragma omp atomic update
        elements->fluidVolume[clusterIndex] += nodes->mass[an_i];
#pragma omp atomic update
        elements->FHydro[clusterIndex] += 1.0 * diffVel;
#pragma omp atomic update
        elements->MHydro[clusterIndex] += 1.0 * radius.cross(diffVel);
#endif
    }
}
template<>
void LB2::reconstructHydroCollide<CPU>() {
    // @todo the inside of this loop could be merged with d_reconstructHydroCollide()
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Convert index to active node index
        const unsigned int an_i = d_nodes->activeI[i];

        // reconstruction of macroscopic variables from microscopic distribution
        // this step is necessary to proceed to the collision step
        d_nodes->reconstruct(an_i);

        // compute interaction forces
        if (d_elements->count) {
            common_computeHydroForces(an_i, d_nodes, d_particles, d_elements);
        }

        //collision operator
        d_nodes->collision(an_i);
    }
}
#ifdef USE_CUDA
__global__ void d_reconstructHydroCollide(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;

    // Convert index to active node index
    const unsigned int an_i = d_nodes->activeI[i];

    // reconstruction of macroscopic variables from microscopic distribution
    // this step is necessary to proceed to the collision step
    d_nodes->reconstruct(an_i);

    // compute interaction forces
    if (d_elements->count) {
        common_computeHydroForces(an_i, d_nodes, d_particles, d_elements);
    }

    //collision operator
    d_nodes->collision(an_i);
}
template<>
void LB2::reconstructHydroCollide<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_reconstructHydroCollide, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_reconstructHydroCollide << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * streaming()
 */
__host__ __device__ __forceinline__ void common_streaming(const unsigned int i, Node2* nodes, Wall2* walls) {
    // Convert index to active node index
    const unsigned int an_i = nodes->activeI[i];

    // coefficient for free-surface
    constexpr double C2x2 = 9.0;
    constexpr double C3x2 = 3.0;
    // coefficient for slip conditions
    const double S1 = PARAMS.slipCoefficient;
    const double S2 = (1.0 - PARAMS.slipCoefficient);
    // creating list for collision function @todo can this be precomputed, rather than once per node?
    std::array<double, lbmDirec> staticPres;
    for (int j = 0; j < lbmDirec; j++) {
        staticPres[j] = PARAMS.fluidMaterial.initDensity * coeff[j];
    }

    // coefficient for bounce-back
    constexpr double BBCoeff = 2.0 * 3.0;

    const unsigned int A_OFFSET = an_i * lbmDirec;
    // cycling through neighbours
    for (unsigned int j = 1; j < lbmDirec; ++j) {
        // getting neighbour index
        const unsigned int ln_i = nodes->d[nodes->count * j + an_i];
        // if neighbour is normal fluid cell what follows is true

        if (ln_i == std::numeric_limits<unsigned int>::max()) { // is gas
            // additional variables for equilibrium f computation
            const double usq = nodes->u[an_i].norm2();
            const double vuj = nodes->u[an_i].dot(v[j]);
            // streaming with constant pressure interface
            nodes->f[A_OFFSET + opp[j]] = -nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq);
        } else {
            const unsigned int L_OFFSET = ln_i * lbmDirec;
            // @todo this could be improved by stacking matching cases to reduce divergence
            switch (nodes->type[ln_i]) {
            case LIQUID:
            {
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[L_OFFSET + opp[j]];
                break;
            }
            case INTERFACE:
            {
#ifdef DEBUG
                // TEST USING AGE //////////////////////////////////////
                const double usq = nodes->u[an_i].norm2();
                const double vuj = nodes->u[an_i].dot(v[j]);
                nodes->f[A_OFFSET + opp[j]] = nodes->age[ln_i] * nodes->fs[L_OFFSET + opp[j]] +
                    (1.0 - nodes->age[ln_i]) * (-nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq));
#else

                nodes->f[A_OFFSET + opp[j]] = nodes->fs[L_OFFSET + opp[j]];
#endif
                break;

            }
            // for walls there is simple bounce-back
            case STAT_WALL:
            {
#ifndef DEBUG 
                if (nodes->type[an_i] == INTERFACE) {
                    // additional variables for equilibrium f computation
                    const double usq = nodes->u[an_i].norm2();
                    const double vuj = nodes->u[an_i].dot(v[j]);
                    //streaming with constant pressure interface
                    nodes->f[A_OFFSET + opp[j]] = -nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq);
                    break;
                }
#endif      
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];

                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, 0.0);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                break;
            }
            // for curved walls there is the rule of Mei-Luo-Shyy
            case TOPO:
            {
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                break;
            }
            case OUTLET:
            {
                nodes->f[A_OFFSET + opp[j]] = std::min(nodes->fs[A_OFFSET + opp[j]], nodes->fs[A_OFFSET + j]);
                break;
            }
            // for moving walls there is simple bounce-back with velocity correction
            case DYN_WALL:
            {
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];
                // velocity of the wall
                const tVect vel = nodes->u[ln_i];
                // variation in Bounce-Back due to moving object
                const double BBi = BBCoeff * nodes->n[an_i] * coeff[j] * vel.dot(v[j]); // mass!!!!!

                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, BBi);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                // adding the extra mass to the surplus //@todo extraMass required for parity
                // extraMass = BBi * nodes->mass[an_i];  // redistributeMass() currently not used, so this isn't implemented properly
                break;
            }// for walls there is simple bounce-back
            case OBJ:
            {
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];
                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, 0.0);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                break;
            }
            case SLIP_STAT_WALL:
            {
                if (j > 6) {
                    const unsigned int nodeCheck1 = nodes->d[slip1Check[j] * nodes->count + an_i];
                    const unsigned int nodeCheck2 = nodes->d[slip2Check[j] * nodes->count + an_i];
                    // check for the environment
                    const bool active1 = nodeCheck1 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck1);
                    const bool active2 = nodeCheck2 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck2);
                    // given the environment, perform the right operation
                    if (active1 && !active2) {
                        // first
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck1 * lbmDirec + slip1[j]] + S2 * nodes->fs[A_OFFSET + j];
                    }
                    else if (!active1 && active2) {
                        // second
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck2 * lbmDirec + slip2[j]] + S2 * nodes->fs[A_OFFSET + j];
                    }
                    else {
                        // standard BB
                        nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                    }
                }
                else {
                    // standard BB
                    nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                }
                break;
            }
            case SLIP_DYN_WALL:
            {
                // velocity of the wall
                const tVect vel = nodes->u[ln_i];
                // variation in Bounce-Back due to moving object
                const double BBi = BBCoeff * nodes->n[an_i] * coeff[j] * vel.dot(v[j]);
                if (j > 6) {
                    const unsigned int nodeCheck1 = nodes->d[slip1Check[j] * nodes->count + an_i];
                    const unsigned int nodeCheck2 = nodes->d[slip2Check[j] * nodes->count + an_i];
                    // check for the environment
                    const bool active1 = nodeCheck1 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck1);
                    const bool active2 = nodeCheck2 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck2);
                    // given the environment, perform the right operation
                    if (active1 && !active2) {
                        // first
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck1 * lbmDirec + slip1[j]] + S2 * (nodes->fs[A_OFFSET + j] - BBi);
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += S2 * nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                    else if (!active1 && active2) {
                        // second
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck2 * lbmDirec + slip2[j]] + S2 * (nodes->fs[A_OFFSET + j] - BBi);
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += S2 * nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                    else {
                        // standard BB
                        nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                }
                else {
                    // standard BB
                    nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                    // adding the extra mass to the surplus //@todo extraMass required for parity
                    // extraMass += nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                }
                break;
            }
            case UNUSED:
            case GAS:
            case PERIODIC:
            case CYL:
            default:
            {
                {
                    // @todo This may print out of order if multiple threads break in parallel
                    tVect pos = nodes->getPosition(an_i);
                    printf("%u(%f, %f, %f) %s TYPE ERROR:\n", an_i, pos.x, pos.y, pos.z, typeString(nodes->type[an_i]));
                    for (unsigned int k = 1; k < lbmDirec; ++k) {
                        printf("before error: j=%u link=%u\n", k, nodes->d[k * nodes->count + an_i]);
                    }
                    pos = nodes->getPosition(ln_i);
                    printf("(%f, %f, %f) %s TYPE ERROR\n", pos.x, pos.y, pos.z, typeString(nodes->type[ln_i]));
                    // @todo aborting from CUDA is harder, especially if the printf() is to be saved
#ifndef __CUDA_ARCH__
                    std::abort();
#endif
                    return;
                }
                break;

            }
            }
        }
    }
}
template<>
void LB2::streaming<CPU>() {
    // STREAMING STEP
    // Init forces to zero
    hd_walls.initForces<CPU>();
    hd_objects.initForces<CPU>();
    // Init streaming support vector
    hd_nodes.store<CPU>();

#pragma omp parallel for // @note extraMass reduction is not currently implemented
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        common_streaming(i, d_nodes, d_walls);
    }

    // redistributing extra mass due to bounce back to interface cells
    // redistributeMass(extraMass);  // extraMass hasn't been implemented properly
}
#ifdef USE_CUDA
__global__ void d_streaming(Node2* d_nodes, Wall2* d_walls) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;

    common_streaming(i, d_nodes, d_walls);
}
template<>
void LB2::streaming<CUDA>() {
    // STREAMING STEP
    // Init forces to zero
    hd_walls.initForces<CUDA>();
    hd_objects.initForces<CUDA>();
    // Init streaming support vector
    hd_nodes.store<CUDA>();

    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_streaming << <gridSize, blockSize >> > (d_nodes, d_walls);
    CUDA_CHECK();

#ifdef _DEBUG
    CUDA_CALL(hipMemcpy(h_nodes.f, hd_nodes.f, sizeof(double) * h_nodes.count * lbmDirec, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.activeI, hd_nodes.activeI, sizeof(unsigned int) * h_nodes.activeCount, hipMemcpyDeviceToHost));
    for (unsigned int in = 0; in < h_nodes.activeCount; ++in) {
        const unsigned int a_i = h_nodes.activeI[in];
        for (unsigned int j = 1; j < lbmDirec; ++j) {
            if (h_nodes.f[a_i * lbmDirec + j] == 0) {
                cout << "Error!" << endl;
            }
        }
    }
#endif

    // redistributing extra mass due to bounce back to interface cells
    // redistributeMass(extraMass);  // extraMass hasn't been implemented properly
}
#endif

/**
 * shiftToPhysical(), originally part of latticeBoltzmannStep()
 */
template<>
void LB2::shiftToPhysical<CPU>() {
    for (unsigned int i = 0; i < d_elements->count; ++i) {
        d_elements->FHydro[i] *= PARAMS.unit.Force;
        d_elements->MHydro[i] *= PARAMS.unit.Torque;
        d_elements->fluidVolume[i] *= PARAMS.unit.Volume;
    }
    for (unsigned int i = 0; i < d_walls->count; ++i) {
        d_walls->FHydro[i] *= PARAMS.unit.Force;
    }
    for (unsigned int i = 0; i < d_objects->count; ++i) {
        d_objects->FHydro[i] *= PARAMS.unit.Force;
    }
}
#ifdef USE_CUDA
__global__ void d_shiftToPhysical(Element2* d_elements, Wall2* d_walls, Object2* d_objects) {
    // Get unique CUDA thread index
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_elements->count) {
        d_elements->FHydro[i] *= PARAMS.unit.Force;
        d_elements->MHydro[i] *= PARAMS.unit.Torque;
        d_elements->fluidVolume[i] *= PARAMS.unit.Volume;
    }
    if (i < d_walls->count) {
        d_walls->FHydro[i] *= PARAMS.unit.Force;
    }
    if (i < d_objects->count) {
        d_objects->FHydro[i] *= PARAMS.unit.Force;
    }
}
template<>
void LB2::shiftToPhysical<CUDA>() {
    // Launch enough threads to accomodate everything
    const unsigned int maxCount = std::max(std::max(h_elements.count, h_walls.count), h_objects.count);
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, maxCount);
    // Round up to accommodate required threads
    gridSize = (maxCount + blockSize - 1) / blockSize;
    d_shiftToPhysical << <gridSize, blockSize >> > (d_elements, d_walls, d_objects);
    CUDA_CHECK();
}
#endif


///
/// latticeBoltzmannFreeSurfaceStep() subroutines
///

/**
 * redistributeMass()
 */
template<>
void LB2::redistributeMass<CPU>(const double& massSurplus) {
    const double addMass = massSurplus / d_nodes->interfaceCount;

#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        const unsigned int in_i = d_nodes->activeI[i];
        d_nodes->mass[in_i] += addMass;
    }
}
#ifdef USE_CUDA
__global__ void d_redistributeMass(Node2 *d_nodes, const double addMass) {
    // Get unique CUDA thread index
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads
    if (i >= d_nodes->interfaceCount)
        return;
    // Increase mass
    const unsigned int in_i = d_nodes->interfaceI[i];
    d_nodes->mass[in_i] += addMass;    
}
template<>
void LB2::redistributeMass<CUDA>(const double& massSurplus) {
    const double addMass = massSurplus / d_nodes->interfaceCount;
    // Launch enough threads to accommodate all interface nodes
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.interfaceCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_redistributeMass<<<gridSize, blockSize>>>(d_nodes, addMass);
    CUDA_CHECK();
}
#endif

/**
 * enforceMassConservation()
 */
template<>
void LB2::enforceMassConservation<CPU>() {
    // calculate total mass of active nodes
    double thisMass = 0.0;
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        const unsigned int an_i = d_nodes->activeI[i];
        if (!d_nodes->isInsideParticle(an_i)) {
            thisMass += d_nodes->mass[an_i];
        }
    }

    // mass deficit
    const double massDeficit = (thisMass - PARAMS.totalMass);

    // fix it
    redistributeMass<CPU>(-0.01 * massDeficit);
}
#ifdef USE_CUDA
/**
 * This unary operator allows us to reduce across a mapped array
 */
template<typename T>
struct unmapper : thrust::unary_function<unsigned int, T> {
    T* d_map;
    unmapper(T* d_map_init)
        : d_map(d_map_init) { }
    __host__ __device__ T operator()(const unsigned int& x) const {
        return d_map[x];
    }
};
template<>
void LB2::enforceMassConservation<CUDA>() {
    // calculate total mass of active nodes
    // This could be switched to use cub in a future cuda version
    const double thisMass = thrust::transform_reduce(hd_nodes.activeI, hd_nodes.activeI + hd_nodes.activeCount,
        unmapper(hd_nodes.mass),
        0.0,
        thrust::plus<double>());

    // mass deficit
    const double massDeficit = (thisMass - PARAMS.totalMass);

    // fix it
    redistributeMass<CUDA>(-0.01 * massDeficit);

}
#endif

/**
 * updateMass()
 */
__host__ __device__ __forceinline__ void common_updateMassInterface(const unsigned int in_i, Node2 *nodes) {
    // mass for interface nodes is regulated by the evolution equation
    nodes->newMass[in_i] = nodes->mass[in_i];
    // additional mass streaming to/from interface
    double deltaMass = 0.0;
    const unsigned int nodeCount = nodes->count;
    // cycling through neighbors
    for (unsigned int j = 1; j < lbmDirec; ++j) {
        // getting neighbor index
        const unsigned int ln_i = nodes->d[nodeCount * j + in_i];
        // average liquid fraction
        if (ln_i == std::numeric_limits<unsigned int>::max()) {
            // do nothing
        } else if (nodes->type[ln_i] == INTERFACE) {
            // average liquid fraction
            const double averageMass = 0.5 * (nodes->mass[ln_i] / nodes->n[ln_i] + nodes->mass[in_i] / nodes->n[in_i]);
            deltaMass += averageMass * nodes->massStream(in_i, j);
        } else if (nodes->type[ln_i] == LIQUID) {
            const double averageMass = 1.0;
            deltaMass += averageMass * nodes->massStream(in_i, j);
        } else if (nodes->type[ln_i] == DYN_WALL) {
            const double averageMass = 1.0 * nodes->mass[in_i];
            deltaMass += averageMass * nodes->massStream(in_i, j);
        } else if (nodes->type[ln_i] == CYL) {
            const double averageMass = 1.0 * nodes->mass[in_i];
            deltaMass += averageMass * nodes->massStream(in_i, j);
        } else if (nodes->type[ln_i] == SLIP_DYN_WALL) {
            if (j > 6) {
                bool active1 = false;
                bool active2 = false;
                const unsigned int c1_i = nodes->d[nodeCount * slip1Check[j] + in_i];
                const unsigned int c2_i = nodes->d[nodeCount * slip2Check[j] + in_i];
                // check for the environment
                if (c1_i != std::numeric_limits<unsigned int>::max()) {
                    if (nodes->isActive(c1_i)) {
                        active1 = true;
                    }
                }
                if (c2_i != std::numeric_limits<unsigned int>::max()) {
                    if (nodes->isActive(c2_i)) {
                        active2 = true;
                    }
                }
                // given the environment, perform the right operation
                double averageMass = 0.0;
                if (active1 && !active2) {
                    // adding the extra mass to the surplus
                    averageMass += 1.0 * (1.0 - PARAMS.slipCoefficient) * nodes->mass[in_i];
                } else if (!active1 && active2) {
                    // adding the extra mass to the surplus
                    averageMass += 1.0 * (1.0 - PARAMS.slipCoefficient) * nodes->mass[in_i];
                } else {
                    // adding the extra mass to the surplus
                    averageMass += 1.0 * nodes->mass[in_i];
                }
                deltaMass += averageMass * nodes->massStream(in_i, j);
            } else {
                // adding the extra mass to the surplus
                const double averageMass = 1.0 * nodes->mass[in_i];
                deltaMass += averageMass * nodes->massStream(in_i, j);
            }
        }
    }
    nodes->newMass[in_i] += deltaMass;

    nodes->mass[in_i] = nodes->newMass[in_i];
    nodes->age[in_i] = min(nodes->age[in_i] + PARAMS.ageRatio, 1.0f);
}
__host__ __device__ __forceinline__ void common_updateMassFluid(const unsigned int fn_i, Node2 *nodes) {
    nodes->mass[fn_i] = nodes->n[fn_i];
    nodes->age[fn_i] = min(nodes->age[fn_i] + PARAMS.ageRatio, 1.0f);
}
template<>
void LB2::updateMass<CPU>() {
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to active node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        common_updateMassInterface(in_i, d_nodes);
    }
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->fluidCount; ++i) {
        // Convert index to active node index
        const unsigned int fn_i = d_nodes->fluidI[i];
        common_updateMassFluid(fn_i, d_nodes);
    }
}
#ifdef USE_CUDA
__global__ void d_updateMass(Node2* d_nodes) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < d_nodes->interfaceCount) {
        // Convert index to active node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        common_updateMassInterface(in_i, d_nodes);
    }
    if (i < d_nodes->fluidCount) {
        // Convert index to active node index
        const unsigned int fn_i = d_nodes->fluidI[i];
        common_updateMassFluid(fn_i, d_nodes);
    }
}
template<>
void LB2::updateMass<CUDA>() {
    // Enough threads for interface or fluid
    const unsigned int maxThreads = std::max(h_nodes.interfaceCount, h_nodes.fluidCount);
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_updateMass, 0, maxThreads);
    // Round up to accommodate required threads
    gridSize = (maxThreads + blockSize - 1) / blockSize;
    d_updateMass<<<gridSize, blockSize>>>(d_nodes);
    CUDA_CHECK();
}
#endif

__host__ __device__ __forceinline__ void common_findInterfaceMutants(const unsigned int in_i, Node2* nodes) {
    // CHECKING FOR NEW FLUID NODES from filling
    if (nodes->mass[in_i] > nodes->n[in_i]) {
        // updating type
        nodes->type[in_i] = INTERFACE_FILLED;
    }// CHECKING FOR NEW GAS NODES from emptying
    else if (nodes->mass[in_i] < 0.0) {
        // updating type
        nodes->type[in_i] = INTERFACE_EMPTY;
    }
}
__host__ __device__ __forceinline__ void common_smoothenInterface_find(const unsigned int in_i, Node2* nodes) {
    constexpr double marginalMass = 1.0e-2;
    // CHECKING FOR NEW INTERFACE NODES from neighboring a new fluid node
    if (nodes->type[in_i] == INTERFACE_FILLED) {
        // neighor indices
        const std::array<unsigned int, lbmDirec> neighborCoord = nodes->findNeighbors(in_i);
        // cycling through neighbors
        for (int j = 1; j < lbmDirec; ++j) {
            // neighbor index
            const unsigned int ln_i = neighborCoord[j];
            // checking if node is gas (so to be transformed into interface)
            if (ln_i < nodes->count && nodes->type[ln_i] == GAS) { // @todo this should probably include INTERFACE_EMPTY (see issue #5)
                nodes->type[ln_i] = GAS_TO_INTERFACE;
            }
        }
    }

    // CHECKING FOR NEW INTERFACE NODES from neighboring a new gas node
    // tested unordered_set, was slower
    if (nodes->type[in_i] == INTERFACE_EMPTY) {
        // neighor indices
        const std::array<unsigned int, lbmDirec> neighborCoord = nodes->findNeighbors(in_i);
        // cycling through neighbors
        for (int j = 1; j < lbmDirec; ++j) {
            // neighbor node
            const unsigned int ln_i = neighborCoord[j];
            if (ln_i < nodes->count && (nodes->type[ln_i] == LIQUID || nodes->type[ln_i] == INTERFACE_FILLED)) {
                nodes->type[ln_i] = FLUID_TO_INTERFACE;
            }
        }
    }
}
__host__ __device__ __forceinline__ void common_smoothenInterface_update(const unsigned int in_i, Node2* nodes) {
    constexpr double marginalMass = 1.0e-2;
    // CHECKING FOR NEW INTERFACE NODES from neighboring a new fluid node
    if (nodes->type[in_i] == GAS_TO_INTERFACE) {
        // create new interface node
        nodes->generateNode(in_i, INTERFACE);
        // add it to interface node list
        // node is becoming active and needs to be initialized
        double massSurplusHere = -marginalMass * PARAMS.fluidMaterial.initDensity;
        // neighor indices
        const std::array<unsigned int, lbmDirec> neighborCoord = nodes->findNeighbors(in_i);
        unsigned int src_i = std::numeric_limits<unsigned int>::max();
        // cycling through neighbors
        for (int j = 1; j < lbmDirec; ++j) {
            // neighbor index
            const unsigned int ln_i = neighborCoord[j];
            // checking if node is gas (so to be transformed into interface)
            if (ln_i < nodes->count && nodes->type[ln_i] == INTERFACE_FILLED) { // @todo this should probably include INTERFACE_EMPTY (see issue #5)
                src_i = ln_i;
            }
        }
        if (src_i == std::numeric_limits<unsigned int>::max()) {
            printf("Error\n");
        }
        // same density and velocity; 1% of the mass
        nodes->copy(in_i, src_i);
        nodes->mass[in_i] = -massSurplusHere;
        // the 1% of the mass is taken form the surplus
        nodes->scatterMass(in_i, massSurplusHere);  // @TODO race condition on extraMass (not currently enabled as redundant)?
        // massSurplus += massSurplusHere;
    }


    // CHECKING FOR NEW INTERFACE NODES from neighboring a new gas node
    // tested unordered_set, was slower
    else if (nodes->type[in_i] == FLUID_TO_INTERFACE) {
        // ln_i should equal nodes->d[in_i * nodes.count + j];
        nodes->type[in_i] = INTERFACE;
        double massSurplusHere = marginalMass * nodes->n[in_i];
        // characteristics are inherited by previous fluid cell. Only mass must be updated to 99% of initial mass
        nodes->mass[in_i] = nodes->n[in_i] - massSurplusHere;
        // the remaining 1% of the mass is added to the surplus
        nodes->scatterMass(in_i, massSurplusHere);
        //massSurplus += massSurplusHere;
    }
}
__host__ __device__ __forceinline__ void common_updateMutants(const unsigned int in_i, Node2* nodes, double *massSurplus) {
    // resetting new gas macroscopic quantities
    if (nodes->type[in_i] == INTERFACE_EMPTY) {
        // updating mass surplus
#ifdef __CUDA_ARCH__
        // CUDA atomics
        atomicAdd(massSurplus, nodes->mass[in_i]);
#else
        // CPU atomics
        #pragma omp atomic update
        *massSurplus += nodes->mass[in_i];
#endif
        // deleting node
        nodes->eraseNode(in_i);
    }

    // resetting new fluid macroscopic quantities
    if (nodes->type[in_i] == INTERFACE_FILLED) {
        // updating mass surplus
#ifdef __CUDA_ARCH__
        // CUDA atomics
        atomicAdd(massSurplus, nodes->mass[in_i] - nodes->n[in_i]);
#else
        // CPU atomics
        #pragma omp atomic update
        *massSurplus += (nodes->mass[in_i] - nodes->n[in_i]);
#endif
        // setting liquid fraction for new fluid cell (other macroscopic characteristics stay the same)
        nodes->mass[in_i] = nodes->n[in_i];
        // Complete it's conversion to type LIQUID
        nodes->type[in_i] = LIQUID;
    }
}
__host__ __device__ __forceinline__ void common_removeIsolated(const unsigned int in_i, Node2* nodes, double *massSurplus) {
    // remove isolated interface cells (surrounded either by only fluid or only solid cells)

    // checking if it is surrounded by fluid (in that case is converted to fluid). Solid is an exception
    // reverse cycle is needed because of deletion function
    {
        bool surroundedFluid = true;
        for (int j = 1; j < lbmDirec; ++j) {
            const unsigned int ln_i = nodes->d[nodes->count * j + in_i];
            if (ln_i == std::numeric_limits<unsigned int>::max() || nodes->type[ln_i] == GAS) {
                surroundedFluid = false;
                break;
            }
        }
        if (surroundedFluid) {
            // update mass storage for balance
#ifdef __CUDA_ARCH__
            // CUDA atomics
            atomicAdd(massSurplus, nodes->mass[in_i] - nodes->n[in_i]);
#else
            // CPU atomics
            #pragma omp atomic update
            *massSurplus += (nodes->mass[in_i] - nodes->n[in_i]);
#endif
            // update characteristics (inherited from the gas node)
            nodes->mass[in_i] = nodes->n[in_i];
            nodes->type[in_i] = LIQUID;
        }
    }

    // checking if it is surrounded by gas (in that case is converted to gas)
    // or, better, if it is not connected to fluid (could be connected to walls or particles)
    {
        bool surroundedGas = true;
        for (int j = 1; j < lbmDirec; ++j) {
            const unsigned int ln_i = nodes->d[nodes->count * j + in_i];
            if (ln_i != std::numeric_limits<unsigned int>::max()) {
                if (nodes->type[ln_i] == LIQUID) {
                    surroundedGas = false;
                    break;
                }
            }
        }
        // updating mass surplus
        if (surroundedGas) {
            // update mass
#ifdef __CUDA_ARCH__
            // CUDA atomics
            atomicAdd(massSurplus, nodes->mass[in_i]);
#else
            // CPU atomics
            #pragma omp atomic update
            *massSurplus += nodes->mass[in_i];
#endif
            nodes->eraseNode(in_i);
        }
    }
}

template<>
void LB2::updateInterface<CPU>() {
    // Initialise reduction variable
    double h_massSurplus = 0.0;
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to interface node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        // filling lists of mutant nodes and changing their type
        common_findInterfaceMutants(in_i, d_nodes);
    }
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to interface node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        // fixing the interface (always one interface between fluid and gas)
        common_smoothenInterface_find(in_i, d_nodes);
    }
    // @todo build temporary list of new/interface/new_gas
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to interface node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        // fixing the interface (always one interface between fluid and gas)
        common_smoothenInterface_update(in_i, d_nodes);
    }
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to interface node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        // updating characteristics of mutant nodes
        common_updateMutants(in_i, d_nodes, &h_massSurplus);
    }
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        // Convert index to interface node index
        const unsigned int in_i = d_nodes->interfaceI[i];
        // remove isolated interface cells (both surrounded by gas and by fluid)
        common_removeIsolated(in_i, d_nodes, &h_massSurplus);
    }
    // @todo Rebuild interface list
    const double addMass = h_massSurplus / d_nodes->interfaceCount;
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->interfaceCount; ++i) {
        const unsigned int in_i = d_nodes->interfaceI[i];
        // distributing surplus to interface cells
        d_nodes->mass[in_i] += addMass;
    }
    // @todo Rebuild all lists
}
#ifdef USE_CUDA
__global__ void d_findInterfaceMutants(Node2* d_nodes) {
    // Get unique CUDA thread index, which corresponds to interface node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->interfaceCount)
        return;
    // Convert index to interface node index
    const unsigned int in_i = d_nodes->interfaceI[i];
    common_findInterfaceMutants(in_i, d_nodes);
}
__global__ void d_smoothenInterface_find(Node2* d_nodes) {
    // Get unique CUDA thread index, which corresponds to interface node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->interfaceCount)
        return;
    // Convert index to interface node index
    const unsigned int in_i = d_nodes->interfaceI[i];
    common_smoothenInterface_find(in_i, d_nodes);
}
__global__ void d_smoothenInterface_update(Node2* d_nodes, unsigned int *count, unsigned int *list) {
    // Get unique CUDA thread index, which corresponds to interface node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= *count)
        return;
    // Convert index to interface node index
    const unsigned int in_i = list[i];
    common_smoothenInterface_update(in_i, d_nodes);
}
__global__ void d_updateMutants(Node2* d_nodes, double* d_massSurplus) {
    // Get unique CUDA thread index, which corresponds to interface node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->interfaceCount)
        return;
    // Convert index to interface node index
    const unsigned int in_i = d_nodes->interfaceI[i];
    common_updateMutants(in_i, d_nodes, d_massSurplus);
}
__global__ void d_removeIsolated(Node2* d_nodes, double* d_massSurplus) {
    // Get unique CUDA thread index, which corresponds to interface node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->interfaceCount)
        return;
    // Convert index to interface node index
    const unsigned int in_i = d_nodes->interfaceI[i];
    common_removeIsolated(in_i, d_nodes, d_massSurplus);
}

__global__ void d_buildList(unsigned int *counter, unsigned int *buffer, const types type_check, const types *types_buffer, const unsigned int threadCount) {
    // Grid stride loop
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < threadCount;
        i += blockDim.x * gridDim.x)
    {
        if (types_buffer[i] == type_check) {
            const unsigned int offset = atomicInc(counter, UINT_MAX);
            buffer[offset] = i;
        }
    }
}
__global__ void d_buildDualList(unsigned int* counter, unsigned int* buffer, const types type_check1, const types type_check2, const types* types_buffer, const unsigned int threadCount) {
    // Grid stride loop
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < threadCount;
        i += blockDim.x * gridDim.x)
    {
        if (types_buffer[i] == type_check1 || types_buffer[i] == type_check2) {
            const unsigned int offset = atomicInc(counter, UINT_MAX);
            buffer[offset] = i;
        }
    }
}

template<>
void LB2::buildInterfaceList<CUDA>(unsigned int max_len, bool update_device_struct) {
    // This is a simple implementation, there may be faster approaches
    // Alternate approach, stable pair-sort indices by type, then scan to identify boundaries

    // Ensure builder list is atleast min(19*h_nodes.activeCount, count)
    auto& ctb = CubTempMem::GetBufferSingleton();
    const unsigned int max_interface = min(max_len, hd_nodes.count) + 1;
    ctb.resize(max_interface * sizeof(unsigned int));
    unsigned int* builderI = reinterpret_cast<unsigned int*>(ctb.getPtr());
    // Init index 0 to 0, this will be used as an atomic counter
    CUDA_CALL(hipMemset(builderI, 0, sizeof(unsigned int)));
    // Launch kernel as grid stride loop
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); // TODO Assumes device 0 in multi device system
    d_buildList << <32 * numSMs, 256 >> > (builderI, &builderI[1], INTERFACE, hd_nodes.type, hd_nodes.count);
    CUDA_CHECK();
    // Copy back result to main list
    unsigned int new_interface_count = 0;
    CUDA_CALL(hipMemcpy(&new_interface_count, builderI, sizeof(unsigned int), hipMemcpyDeviceToHost));
    if (new_interface_count > hd_nodes.interfaceAlloc) {
        // Resize interface buffer (it doesn't currently ever scale back down)
        if (hd_nodes.interfaceI) {
            CUDA_CALL(hipFree(hd_nodes.interfaceI));
        }
        CUDA_CALL(hipMalloc(&hd_nodes.interfaceI, new_interface_count * sizeof(unsigned int)));
        hd_nodes.interfaceAlloc = new_interface_count;
    }
    hd_nodes.interfaceCount = new_interface_count;
    // Sort list into it's new storage
    size_t temp_storage_bytes = 0;
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes, &builderI[1], hd_nodes.interfaceI, new_interface_count));
    auto& ctm = CubTempMem::GetTempSingleton();
    ctm.resize(temp_storage_bytes);
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(ctm.getPtr(), temp_storage_bytes, &builderI[1], hd_nodes.interfaceI, new_interface_count));
    // Update device struct (new size and ptr, but whole struct because eh)
    if (update_device_struct) {
        CUDA_CALL(hipMemcpy(d_nodes, &hd_nodes, sizeof(Node2), hipMemcpyHostToDevice));
    }
}
template<>
void LB2::buildFluidList<CUDA>(unsigned int max_len, bool update_device_struct) {
    // This is a simple implementation, there may be faster approaches
    // Alternate approach, stable pair-sort indices by type, then scan to identify boundaries

    // Ensure builder list is atleast min(19*h_nodes.activeCount, count)
    auto& ctb = CubTempMem::GetBufferSingleton();
    const unsigned int max_interface = min(max_len, hd_nodes.count) + 1;
    ctb.resize(max_interface * sizeof(unsigned int));
    unsigned int* builderI = reinterpret_cast<unsigned int*>(ctb.getPtr());
    // Init index 0 to 0, this will be used as an atomic counter
    CUDA_CALL(hipMemset(builderI, 0, sizeof(unsigned int)));
    // Launch kernel as grid stride loop
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); // TODO Assumes device 0 in multi device system
    d_buildList << <32 * numSMs, 256 >> > (builderI, &builderI[1], LIQUID, hd_nodes.type, hd_nodes.count);
    CUDA_CHECK();
    // Copy back result to main list
    unsigned int new_fluid_count = 0;
    CUDA_CALL(hipMemcpy(&new_fluid_count, builderI, sizeof(unsigned int), hipMemcpyDeviceToHost));
    if (new_fluid_count > hd_nodes.fluidAlloc) {
        // Resize buffer (it doesn't currently ever scale back down)
        if (hd_nodes.fluidI) {
            CUDA_CALL(hipFree(hd_nodes.fluidI));
        }
        CUDA_CALL(hipMalloc(&hd_nodes.fluidI, new_fluid_count * sizeof(unsigned int)));
        hd_nodes.fluidAlloc = new_fluid_count;
    }
    hd_nodes.fluidCount = new_fluid_count;
    // Sort list into it's new storage
    size_t temp_storage_bytes = 0;
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes, &builderI[1], hd_nodes.fluidI, new_fluid_count));
    auto& ctm = CubTempMem::GetTempSingleton();
    ctm.resize(temp_storage_bytes);
    CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(ctm.getPtr(), temp_storage_bytes, &builderI[1], hd_nodes.fluidI, new_fluid_count));
    // Update device struct (new size and ptr, but whole struct because eh)
    if (update_device_struct) {
        CUDA_CALL(hipMemcpy(d_nodes, &hd_nodes, sizeof(Node2), hipMemcpyHostToDevice));
    }
}
template<>
void LB2::buildActiveList<CUDA>() {
    // Merge fluid and interface lists into active list
    // This could be done with cub with CCCL 2.7.0, but that's 3 weeks old so not currently provided by CUDA, whilst retaining sortedness
    // Instead we'll just pack the two buffers one after the other, probably good enough.

    // Resize active list
    const unsigned int new_active_count = hd_nodes.interfaceCount + hd_nodes.fluidCount;
    if (new_active_count > hd_nodes.activeAlloc) {
        // Resize buffer (it doesn't currently ever scale back down)
        if (hd_nodes.activeI) {
            CUDA_CALL(hipFree(hd_nodes.activeI));
        }
        CUDA_CALL(hipMalloc(&hd_nodes.activeI, new_active_count * sizeof(unsigned int)));
        hd_nodes.activeAlloc = new_active_count;
    }
    hd_nodes.activeCount = new_active_count;
    // Copy data to buffer
    CUDA_CALL(hipMemcpy(hd_nodes.activeI, hd_nodes.interfaceI, hd_nodes.interfaceCount * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(&hd_nodes.activeI[hd_nodes.interfaceCount], hd_nodes.fluidI, hd_nodes.fluidCount * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    // Update device struct (new size and ptr, but whole struct because eh)
    CUDA_CALL(hipMemcpy(d_nodes, &hd_nodes, sizeof(Node2), hipMemcpyHostToDevice));
}
template<>
unsigned int *LB2::buildTempNewList<CUDA>(unsigned int max_len, bool update_device_struct) {
    // This is a simple implementation, there may be faster approaches
    // Alternate approach, stable pair-sort indices by type, then scan to identify boundaries

    // Ensure builder list is atleast min(19*h_nodes.activeCount, count)
    auto& ctb = CubTempMem::GetBufferSingleton();
    const unsigned int max_interface = min(max_len, hd_nodes.count) + 1;
    ctb.resize(max_interface * sizeof(unsigned int));
    unsigned int* builderI = reinterpret_cast<unsigned int*>(ctb.getPtr());
    // Init index 0 to 0, this will be used as an atomic counter
    CUDA_CALL(hipMemset(builderI, 0, sizeof(unsigned int)));
    // Launch kernel as grid stride loop
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); // TODO Assumes device 0 in multi device system
    d_buildDualList<<<32 * numSMs, 256>>>(builderI, &builderI[1], GAS_TO_INTERFACE, FLUID_TO_INTERFACE, hd_nodes.type, hd_nodes.count);
    CUDA_CHECK();
    // This is a temporary list, so just return the device pointer
    return builderI;
}

template<>
void LB2::updateInterface<CUDA>() {
    // Initialise reduction variable
    auto& t = CubTempMem::GetTempSingleton();
    t.resize(sizeof(double));
    double *d_massSurplus = static_cast<double*>(t.getPtr());
    double h_massSurplus = 0;
    CUDA_CALL(hipMemcpy(d_massSurplus, &h_massSurplus, sizeof(double), hipMemcpyHostToDevice));

    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    // Separate kernels, in the same (default) stream, synchronisation is required between each kernel launch
    // filling lists of mutant nodes and changing their type
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_findInterfaceMutants, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_findInterfaceMutants<<<gridSize, blockSize>>>(d_nodes);
    // fixing the interface (always one interface between fluid and gas)
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_smoothenInterface_find, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_smoothenInterface_find<<<gridSize, blockSize>>>(d_nodes);
    unsigned int *d_templist = buildTempNewList<CUDA>(lbmDirec * hd_nodes.interfaceCount);
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_smoothenInterface_update, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_smoothenInterface_update<<<gridSize, blockSize>>>(d_nodes, d_templist, d_templist+1);
    // updating characteristics of mutant nodes
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_updateMutants, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_updateMutants<<<gridSize, blockSize>>>(d_nodes, d_massSurplus);
    // Rebuild interface list
    buildInterfaceList<CUDA>(lbmDirec * hd_nodes.activeCount);
    // remove isolated interface cells (both surrounded by gas and by fluid)
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_removeIsolated, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_removeIsolated<<<gridSize, blockSize>>>(d_nodes, d_massSurplus);
    // Rebuild all lists
    buildInterfaceList<CUDA>(hd_nodes.interfaceCount, false);
    buildFluidList<CUDA>(hd_nodes.fluidCount + lbmDirec * hd_nodes.interfaceCount, false);
    buildActiveList<CUDA>();
    // distributing surplus to interface cells
    CUDA_CALL(hipMemcpy(&h_massSurplus, d_massSurplus, sizeof(double), hipMemcpyDeviceToHost));
    const double addMass = h_massSurplus / hd_nodes.interfaceCount;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_redistributeMass, 0, hd_nodes.interfaceCount);
    gridSize = (hd_nodes.interfaceCount + blockSize - 1) / blockSize;
    d_redistributeMass<<<gridSize, blockSize>>>(d_nodes, addMass);
#ifdef DEBUG
    // computeSurfaceNormal()
#endif
    CUDA_CHECK();
}
#endif


void LB2::latticeBoltzmannCouplingStep(bool& newNeighbourList) {
    // identifies which nodes need to have an update due to particle movement
    // the complexity arises from trying to keep the scaling as close to linear as possible
    // maybe the best idea is to do this in two step:
    // 1) the first is to check for new active nodes and initialise them
    // 2) the second is checking for new solid nodes.
    // this automatically check also for the hideous case of particle to particle double transition

    /**
     * @todo The parallelisation of each of these methods should be reviewed
     *       Most are 2D loops, the range of each being unclear
     *       Likewise, can OpenMP parallel block be moved outside of each member?
     */

     // first we check if a new neighbour table has been defined. In that case, the indexing needs to be reinitialised
    if (newNeighbourList) {
        cout << endl << "New neighbour list" << endl;
        this->initializeParticleBoundaries<IMPL>();
        newNeighbourList = false;
    }
    else {
        // SOLID TO ACTIVE CHECK
        // @note Calling this directly after initializeParticleBoundaries() is redundant, hence else
        this->findNewActive<IMPL>();
    }

    // ACTIVE TO SOLID CHECK
    this->findNewSolid<IMPL>();

    if (PARAMS.freeSurface) {
        this->checkNewInterfaceParticles<IMPL>();
    }
}
void LB2::latticeBoltzmannStep() {
    // Reconstruct active list
    hd_nodes.cleanLists<IMPL>();

    // Initializing the elements forces (lattice units)
    hd_elements.initElements<IMPL>();

    // Initialise lattice boltzmann force vector
    if (!h_PARAMS.forceField) {
        h_PARAMS.lbF.reset();
        syncParams();
    }

    // reconstruct(), computeHydroForces(), collision()
    // Reconstruct macroscopic variables from microscopic distribution
    // Compute interaction forces with DEM elmts
    // Collision step
    this->reconstructHydroCollide<IMPL>();

    // Streaming operator
    this->streaming<IMPL>();

    // Shift element/wall/object forces and torques to physical units
    this->shiftToPhysical<IMPL>();
}
extern ProblemName problemName;
void LB2::latticeBoltzmannFreeSurfaceStep() {
    // in case mass needs to be kept constant, call enforcing function here
    if (PARAMS.imposeFluidVolume) {
        this->enforceMassConservation<IMPL>();
    } else if (PARAMS.increaseVolume) {
        if (PARAMS.time < PARAMS.deltaTime) {
            this->redistributeMass<IMPL>(PARAMS.deltaVolume / PARAMS.deltaTime);
        }
    } else {
        switch (problemName) {
        case DRUM:
        case STAVA:
        {
            this->enforceMassConservation<IMPL>();
            break;
        }
        }
    }

    // mass and free surface update
    this->updateMass<IMPL>();
    this->updateInterface<IMPL>();
    hd_nodes.cleanLists<IMPL>();
}

Node2& LB2::getNodes() {
#ifdef USE_CUDA
    // If using CUDA, data is on device by default, so sync back.
    if (hd_nodes.count > h_nodes.count) {
        // Resize main buffers
        if (h_nodes.f) free(h_nodes.f);
        h_nodes.f = static_cast<double*>(malloc(hd_nodes.count * lbmDirec * sizeof(double)));
        if (h_nodes.fs) free(h_nodes.fs);
        h_nodes.fs = static_cast<double*>(malloc(hd_nodes.count * lbmDirec * sizeof(double)));
        if (h_nodes.n) free(h_nodes.n);
        h_nodes.n = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.u) free(h_nodes.u);
        h_nodes.u = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.hydroForce) free(h_nodes.hydroForce);
        h_nodes.hydroForce = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.centrifugalForce) free(h_nodes.centrifugalForce);
        h_nodes.centrifugalForce = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.mass) free(h_nodes.mass);
        h_nodes.mass = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.newMass) free(h_nodes.newMass);
        h_nodes.newMass = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.visc) free(h_nodes.visc);
        h_nodes.visc = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.basal) free(h_nodes.basal);
        h_nodes.basal = static_cast<bool*>(malloc(hd_nodes.count * sizeof(bool)));
        if (h_nodes.friction) free(h_nodes.friction);
        h_nodes.friction = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.age) free(h_nodes.age);
        h_nodes.age = static_cast<float*>(malloc(hd_nodes.count * sizeof(float)));
        if (h_nodes.solidIndex) free(h_nodes.solidIndex);
        h_nodes.solidIndex = static_cast<unsigned int*>(malloc(hd_nodes.count * sizeof(unsigned int)));
        if (h_nodes.d) free(h_nodes.d);
        h_nodes.d = static_cast<unsigned int*>(malloc(hd_nodes.count * lbmDirec * sizeof(unsigned int)));
        if (h_nodes.type) free(h_nodes.type);
        h_nodes.type = static_cast<types*>(malloc(hd_nodes.count * sizeof(types)));
        if (h_nodes.p) free(h_nodes.p);
        h_nodes.p = static_cast<bool*>(malloc(hd_nodes.count * sizeof(bool)));
    }
    h_nodes.count = hd_nodes.count;
    // Resize misc buffers
    if (hd_nodes.activeCount > h_nodes.activeAlloc) {
        if (h_nodes.activeI) free(h_nodes.activeI);
        h_nodes.activeI = static_cast<unsigned int*>(malloc(hd_nodes.activeCount * sizeof(unsigned int)));
    }
    h_nodes.activeCount = hd_nodes.activeCount;
    if (hd_nodes.interfaceCount > h_nodes.interfaceAlloc) {
        if (h_nodes.interfaceI) free(h_nodes.interfaceI);
        h_nodes.interfaceI = static_cast<unsigned int*>(malloc(hd_nodes.interfaceCount * sizeof(unsigned int)));
        h_nodes.interfaceAlloc = hd_nodes.interfaceCount;
    }
    h_nodes.interfaceCount = hd_nodes.interfaceCount;
    if (hd_nodes.fluidCount > h_nodes.fluidAlloc) {
        if (h_nodes.fluidI) free(h_nodes.fluidI);
        h_nodes.fluidI = static_cast<unsigned int*>(malloc(hd_nodes.fluidCount * sizeof(unsigned int)));
        h_nodes.fluidAlloc = hd_nodes.fluidCount;
    }
    h_nodes.fluidCount = hd_nodes.fluidCount;
    if (hd_nodes.wallCount > h_nodes.wallCount) {
        if (h_nodes.wallI) free(h_nodes.wallI);
        h_nodes.wallI = static_cast<unsigned int*>(malloc(hd_nodes.wallCount * sizeof(unsigned int)));
    }
    h_nodes.wallCount = hd_nodes.wallCount;
    // Copy main buffers back to host
    // CUDA_CALL(hipMemcpy(h_nodes.coord, hd_nodes.coord, h_nodes.count * sizeof(unsigned int), hipMemcpyDeviceToHost)); // redundant?
    CUDA_CALL(hipMemcpy(h_nodes.f, hd_nodes.f, h_nodes.count * lbmDirec * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.fs, hd_nodes.fs, h_nodes.count * lbmDirec * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.n, hd_nodes.n, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.u, hd_nodes.u, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.hydroForce, hd_nodes.hydroForce, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.centrifugalForce, hd_nodes.centrifugalForce, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.mass, hd_nodes.mass, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.newMass, hd_nodes.newMass, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.visc, hd_nodes.visc, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.basal, hd_nodes.basal, h_nodes.count * sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.friction, hd_nodes.friction, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.age, hd_nodes.age, h_nodes.count * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.solidIndex, hd_nodes.solidIndex, h_nodes.count * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.d, hd_nodes.d, h_nodes.count * lbmDirec * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.type, hd_nodes.type, h_nodes.count * sizeof(types), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.p, hd_nodes.p, h_nodes.count * sizeof(bool), hipMemcpyDeviceToHost));
    // Copy misc buffers back to host
    CUDA_CALL(hipMemcpy(h_nodes.activeI, hd_nodes.activeI, h_nodes.activeCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.interfaceI, hd_nodes.interfaceI, h_nodes.interfaceCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.fluidI, hd_nodes.fluidI, h_nodes.fluidCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.wallI, hd_nodes.wallI, h_nodes.wallCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
#endif
    return h_nodes;
}
void LB2::initDeviceNodes() {
#ifdef USE_CUDA
    // Allocate the main storage
    if (d_nodes) {
        fprintf(stderr, "LB2::initDeviceNodes() should only be called once.");
        throw std::exception();
    }
    cout << "Initialising device nodes..";
    CUDA_CALL(hipMalloc(&d_nodes, sizeof(Node2)));
    // Build HD struct
    hd_nodes.activeCount = h_nodes.activeCount;
    hd_nodes.activeAlloc = h_nodes.activeCount;
    CUDA_CALL(hipMalloc(&hd_nodes.activeI, hd_nodes.activeCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.activeI, h_nodes.activeI, hd_nodes.activeCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.interfaceCount = h_nodes.interfaceCount;
    hd_nodes.interfaceAlloc = h_nodes.interfaceCount;
    CUDA_CALL(hipMalloc(&hd_nodes.interfaceI, hd_nodes.interfaceCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.interfaceI, h_nodes.interfaceI, hd_nodes.interfaceCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.fluidCount = h_nodes.fluidCount;
    hd_nodes.fluidAlloc = h_nodes.fluidCount;
    CUDA_CALL(hipMalloc(&hd_nodes.fluidI, hd_nodes.fluidCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.fluidI, h_nodes.fluidI, hd_nodes.fluidCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.wallCount = h_nodes.wallCount;
    CUDA_CALL(hipMalloc(&hd_nodes.wallI, hd_nodes.wallCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.wallI, h_nodes.wallI, hd_nodes.wallCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.count = h_nodes.count;
    CUDA_CALL(hipMalloc(&hd_nodes.f, hd_nodes.count * lbmDirec * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.f, h_nodes.f, hd_nodes.count * lbmDirec * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.fs, hd_nodes.count * lbmDirec * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.fs, h_nodes.fs, hd_nodes.count * lbmDirec * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.n, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.n, h_nodes.n, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.u, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.u, h_nodes.u, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.hydroForce, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.hydroForce, h_nodes.hydroForce, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.centrifugalForce, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.centrifugalForce, h_nodes.centrifugalForce, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.mass, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.mass, h_nodes.mass, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.newMass, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.newMass, h_nodes.newMass, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.visc, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.visc, h_nodes.visc, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.basal, hd_nodes.count * sizeof(bool)));
    CUDA_CALL(hipMemcpy(hd_nodes.basal, h_nodes.basal, hd_nodes.count * sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.friction, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.friction, h_nodes.friction, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.age, hd_nodes.count * sizeof(float)));
    CUDA_CALL(hipMemcpy(hd_nodes.age, h_nodes.age, hd_nodes.count * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.solidIndex, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.solidIndex, h_nodes.solidIndex, hd_nodes.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.d, hd_nodes.count * lbmDirec * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.d, h_nodes.d, hd_nodes.count * lbmDirec * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.type, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.type, h_nodes.type, hd_nodes.count * sizeof(types), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.p, hd_nodes.count * sizeof(bool)));
    CUDA_CALL(hipMemcpy(hd_nodes.p, h_nodes.p, hd_nodes.count * sizeof(bool), hipMemcpyHostToDevice));
    // Copy struct containing device pointers and counts
    CUDA_CALL(hipMemcpy(d_nodes, &hd_nodes, sizeof(Node2), hipMemcpyHostToDevice));
    cout << "..complete" << std::endl;
#endif    
}

void LB2::init(cylinderList& cylinders, wallList& walls, particleList& particles, objectList& objects, bool externalSolveCoriolis, bool externalSolveCentrifugal) {
    // Convert from AoS format to SoA and copy to device
    syncCylinders<IMPL>(cylinders);
    syncWalls<IMPL>(walls);
    syncParticles<IMPL>(particles);
    syncObjects<IMPL>(objects);

    //  Lattice Boltzmann initialization steps

    // switchers for apparent accelerations
    h_PARAMS.solveCoriolis = externalSolveCoriolis;
    h_PARAMS.solveCentrifugal = externalSolveCentrifugal;

    // first comes the initialization of the data structures
    cout << "Initializing nodes containers and types" << endl;
    // total number of nodes
    h_PARAMS.totPossibleNodes = h_PARAMS.lbSize[0] * h_PARAMS.lbSize[1] * h_PARAMS.lbSize[2];

    // Allocate nodes (dense matrix, even gas nodes are represented)
    // This initialises them all as GAS
    allocateHostNodes(h_PARAMS.totPossibleNodes);

    // application of lattice boundaries
    initializeLatticeBoundaries();
    // then the initial node type must be identified for every node (if not specified, it is already Fluid)
    initializeTypes(walls, cylinders, objects);

    ifstream fluidFileID;
    if (h_PARAMS.lbRestart) {
        // open fluid restart file
        fluidFileID.open(init_params.lbRestartFile.c_str(), ios::in);
        ASSERT(fluidFileID.is_open());
        // check if the restart file size is ok
        unsigned int restartX, restartY, restartZ, restartNodes;
        fluidFileID >> restartX;
        fluidFileID >> restartY;
        fluidFileID >> restartZ;
        fluidFileID >> restartNodes;
        ASSERT(restartX == h_PARAMS.lbSize[0]);
        ASSERT(restartY == h_PARAMS.lbSize[1]);
        ASSERT(restartZ == h_PARAMS.lbSize[2]);
        // read active nodes from file and generate
        // @todo
        fprintf(stderr, "lbRestart is not yet supported.\n");
        throw std::exception();
        // restartInterface(fluidFileID, restartNodes);
    } else {
        // initialize interface
        initializeInterface();
        // initialize variables for active nodes
        initializeVariables();
    }

    // initialize variables for wall nodes
    initializeWalls();

    // initialize h_nodes's fluid, interface and active lists
    initializeLists();

    // Setup hd_nodes, copy it to d_nodes
    initDeviceNodes();

    // application of particle initial position
    const double inside_mass = initializeParticleBoundaries<IMPL>();

    // in case mass needs to be kept constant, compute it here
    h_PARAMS.totalMass = 0.0;
    if (h_PARAMS.imposeFluidVolume) {
        // volume and mass is the same in lattice units
        h_PARAMS.totalMass = h_PARAMS.imposedFluidVolume / h_PARAMS.unit.Volume;
    } else {
        switch (problemName) {
        case DRUM:
        {
            h_PARAMS.totalMass = h_PARAMS.fluidMass / h_PARAMS.unit.Mass;
            break;
        }
        case STAVA:
        {
            h_PARAMS.totalMass = 200000.0 / h_PARAMS.unit.Volume;
            break;
        }
        default:
        {
            // @todo This needs to be calculated on device if using CUDA, h_nodes is out of date follow initializeParticleBoundaries
            for (int i = 0; i < h_nodes.activeCount; ++i) {
                const unsigned int a_i = h_nodes.activeI[i];
                if (!h_nodes.isInsideParticle(a_i)) {
                    h_PARAMS.totalMass += h_nodes.mass[a_i];
                }
            }
            break;
        }
        }
    }
    if (h_PARAMS.increaseVolume) {
        h_PARAMS.deltaVolume /= h_PARAMS.unit.Volume;
        h_PARAMS.deltaTime /= h_PARAMS.unit.Time;
    }

    syncParams();

    cout << "Done with initialization" << endl;
}

void LB2::allocateHostNodes(const unsigned int count) {
    // Allocate enough memory for these nodes
    assert(h_nodes.count == 0);  // No nodes should exist at the time this is called
    h_nodes.count = count;
    // Allocate host buffers
    //h_nodes.coord = static_cast<unsigned int*>(malloc(h_nodes.count * sizeof(unsigned int))); // TODO nolonger required
    h_nodes.f = static_cast<double*>(malloc(h_nodes.count * lbmDirec * sizeof(double)));
    h_nodes.fs = static_cast<double*>(malloc(h_nodes.count * lbmDirec * sizeof(double)));
    h_nodes.n = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.u = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.hydroForce = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.centrifugalForce = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.mass = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.newMass = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.visc = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.basal = static_cast<bool*>(malloc(h_nodes.count * sizeof(bool)));
    h_nodes.friction = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.age = static_cast<float*>(malloc(h_nodes.count * sizeof(float)));
    h_nodes.solidIndex = static_cast<unsigned int*>(malloc(h_nodes.count * sizeof(unsigned int)));
    h_nodes.d = static_cast<unsigned int*>(malloc(h_nodes.count * lbmDirec * sizeof(unsigned int)));
    h_nodes.type = static_cast<types*>(malloc(h_nodes.count * sizeof(types)));
    h_nodes.p = static_cast<bool*>(malloc(h_nodes.count * sizeof(bool)));
    // Zero initialisation
    memset(h_nodes.f, 0, h_nodes.count * lbmDirec * sizeof(double));
    memset(h_nodes.fs, 0, h_nodes.count * lbmDirec * sizeof(double));
    memset(h_nodes.n, 0, h_nodes.count * sizeof(double));
    memset(h_nodes.u, 0, h_nodes.count * sizeof(tVect));
    memset(h_nodes.hydroForce, 0, h_nodes.count * sizeof(tVect));
    memset(h_nodes.centrifugalForce, 0, h_nodes.count * sizeof(tVect));
    memset(h_nodes.mass, 0, h_nodes.count * sizeof(double));
    memset(h_nodes.newMass, 0, h_nodes.count * sizeof(double));
    std::fill(h_nodes.visc, h_nodes.visc + h_nodes.count, 1.0);
    memset(h_nodes.basal, 0, h_nodes.count * sizeof(bool));
    memset(h_nodes.friction, 0, h_nodes.count * sizeof(double));
    memset(h_nodes.age, 0, h_nodes.count * sizeof(float));
    memset(h_nodes.solidIndex, std::numeric_limits<unsigned int>::max(), h_nodes.count * sizeof(unsigned int));
    memset(h_nodes.d, std::numeric_limits<unsigned int>::max(), h_nodes.count * lbmDirec * sizeof(unsigned int));
    std::fill(h_nodes.type, h_nodes.type + h_nodes.count, GAS);
    memset(h_nodes.p, 0, h_nodes.count * sizeof(bool));
}

void LB2::initializeLatticeBoundaries() {
    // assign boundary characteristic to nodes (see class)
    // if not differently defined, type is 0 (fluid)

    // BOUNDARY CONDITIONS ///////////////////////////
    // solid boundary wins over all in corners, where more than 1 bc is defined
    cout << "Initializing boundaries" << endl;

    unsigned int indexHere = 0;
    // XY
    for (unsigned int x = 0; x < h_PARAMS.lbSize[0]; ++x) {
        for (unsigned int y = 0; y < h_PARAMS.lbSize[1]; ++y) {
            // bottom
            indexHere = h_PARAMS.getIndex(x, y, 0);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[4]);
            }
            // top
            indexHere = h_PARAMS.getIndex(x, y, h_PARAMS.lbSize[2] - 1);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[5]);
            }
        }
    }

    // YZ
    for (unsigned int y = 0; y < h_PARAMS.lbSize[1]; ++y) {
        for (unsigned int z = 0; z < h_PARAMS.lbSize[2]; ++z) {
            // bottom
            indexHere = h_PARAMS.getIndex(0, y, z);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[0]);
            }
            // top
            indexHere = h_PARAMS.getIndex(h_PARAMS.lbSize[0] - 1, y, z);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[1]);
            }
        }
    }

    // ZX
    for (unsigned int z = 0; z < h_PARAMS.lbSize[2]; ++z) {
        for (unsigned int x = 0; x < h_PARAMS.lbSize[0]; ++x) {
            // bottom
            indexHere = h_PARAMS.getIndex(x, 0, z);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[2]);
            }
            // top
            indexHere = h_PARAMS.getIndex(x, h_PARAMS.lbSize[1] - 1, z);
            if (h_nodes.type[indexHere] == GAS) {
                generateNode(indexHere, h_PARAMS.boundary[3]);
            }
        }
    }
}
void LB2::initializeTypes(const wallList& walls, const cylinderList& cylinders, const objectList& objects) {
    initializeWallBoundaries(walls);
    // application of solid cylinders
    initializeCylinderBoundaries(cylinders);
    // application of objects
    initializeObjectBoundaries(objects);
    // initializing topography if one is present
    initializeTopography();
}
void LB2::initializeWallBoundaries(const wallList& walls) {
    // const double wallThickness = 2.0 * h_PARAMS.unit.Length;
    // SOLID WALLS ////////////////////////
    for (unsigned int iw = 0; iw < walls.size(); ++iw) {
        const tVect convertedWallp = walls[iw].p / h_PARAMS.unit.Length;
        const tVect normHere = walls[iw].n;
        const unsigned int indexHere = walls[iw].index;
        const bool slipHere = walls[iw].slip;
        const bool movingHere = walls[iw].moving;
        // @todo This was previously OpenMP parallel, but could be race condition in generateNode?
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            // check if the node is solid
            // all walls have max thickness 2 nodes
            const tVect pos = h_PARAMS.getPosition(it);
            const double wallDistance = pos.distance2Plane(convertedWallp, normHere);
            if (wallDistance > -2.0 && wallDistance < 0.0) {
                //check for borders in limted walls
                if (walls[iw].limited) {
                    const double xHere = pos.x * h_PARAMS.unit.Length;
                    const double yHere = pos.y * h_PARAMS.unit.Length;
                    const double zHere = pos.z * h_PARAMS.unit.Length;
                    // check if beyond limits
                    if (xHere < walls[iw].xMin || xHere > walls[iw].xMax ||
                        yHere < walls[iw].yMin || yHere > walls[iw].yMax ||
                        zHere < walls[iw].zMin || zHere > walls[iw].zMax) {
                        continue;
                    }
                }
                // Node is inside a wall
                // generate node (tentatively as static wall)
                generateNode(it, STAT_WALL);
                // setting solidIndex
                h_nodes.solidIndex[it] = indexHere; // TODO indexHere is redundant, use iw?
                // setting type: 5-6=slip, 7-8=no-slip
                if (slipHere) {
                    // setting type for slip: 5=static, 6=moving
                    if (movingHere) {
                        h_nodes.type[it] = SLIP_DYN_WALL;
                    } else {
                        h_nodes.type[it] = SLIP_STAT_WALL;
                    }
                } else {
                    // setting type for no-slip: 7=static, 8=moving
                    if (movingHere) {
                        h_nodes.type[it] = DYN_WALL;
                    } else {
                        h_nodes.type[it] = STAT_WALL;
                    }
                }
            }
        }
    }

}
void LB2::initializeObjectBoundaries(const objectList& objects) {
    // SOLID WALLS ////////////////////////
    for (int io = 0; io < objects.size(); ++io) {
        const tVect convertedPosition = objects[io].x0 / h_PARAMS.unit.Length;
        const double convertedRadius = objects[io].r / h_PARAMS.unit.Length;
        const unsigned int indexHere = objects[io].index;
        // @todo This was previously OpenMP parallel, but could be race condition in generateNode?
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            const tVect nodePosition = h_PARAMS.getPosition(it);
            if (nodePosition.insideSphere(convertedPosition, convertedRadius)) {
                generateNode(it, OBJ);
                h_nodes.solidIndex[it] = indexHere; // TODO indexHere is redundant, use io?
            }
        }
    }
}
void LB2::initializeCylinderBoundaries(const cylinderList& cylinders) {
    // SOLID CYLINDERS ////////////////////////
    for (int ic = 0; ic < cylinders.size(); ++ic) {
        const tVect convertedCylinderp1 = cylinders[ic].p1 / h_PARAMS.unit.Length;
        const tVect naxesHere = cylinders[ic].naxes;
        const double convertedRadius = cylinders[ic].R / h_PARAMS.unit.Length;
        const unsigned int indexHere = cylinders[ic].index;
        const bool slipHere = cylinders[ic].slip;
        const bool movingHere = cylinders[ic].moving;
        // @todo This was previously OpenMP parallel, but could be race condition in generateNode?
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            // creating solid cells
            const bool isOutside = h_PARAMS.getPosition(it).insideCylinder(convertedCylinderp1, naxesHere, convertedRadius, convertedRadius + 3.0);
            const bool isInside = h_PARAMS.getPosition(it).insideCylinder(convertedCylinderp1, naxesHere, max(convertedRadius - 3.0, 0.0), convertedRadius);
            if ((cylinders[ic].type == FULL && isInside) ||
                (cylinders[ic].type == EMPTY && isOutside)) {
                //check for borders in limted walls
                if (cylinders[ic].limited) {
                    const tVect here = h_PARAMS.getPosition(it) * h_PARAMS.unit.Length;
                    // check if beyond limits
                    if (here.x < cylinders[ic].xMin || here.x > cylinders[ic].xMax ||
                        here.y < cylinders[ic].yMin || here.y > cylinders[ic].yMax ||
                        here.z < cylinders[ic].zMin || here.z > cylinders[ic].zMax) {
                        continue;
                    }
                }
                // Node is inside a cylinder
                // tentatively static
                generateNode(it, STAT_WALL);
                // setting solidIndex
                h_nodes.solidIndex[it] = indexHere;  // TODO indexHere is redundant, use ic?
                // setting type: 5-6=slip, 7-8=no-slip
                if (slipHere) {
                    // setting type for slip: 5=static, 6=moving
                    if (movingHere) {
                        h_nodes.type[it] = SLIP_DYN_WALL;
                    } else {
                        h_nodes.type[it] = SLIP_STAT_WALL;
                    }
                } else {
                    // setting type for no-slip: 7=static, 8=moving
                    if (movingHere) {
                        h_nodes.type[it] = DYN_WALL;
                    } else {
                        h_nodes.type[it] = STAT_WALL;
                    }
                }
            }
        }
    }
}
void LB2::initializeTopography() {
    // Based on initializeTopography()
    
    const double surfaceThickness = 1.75 * h_PARAMS.unit.Length;

    // TOPOGRAPHY ////////////////////////
    if (h_PARAMS.lbTopography) {
        lbTop.readFromFile(init_params.lbTopographyFile, h_PARAMS.translateTopographyX, h_PARAMS.translateTopographyY, h_PARAMS.translateTopographyZ);
        lbTop.show();
        // check if topography grid contains the fluid domain
        ASSERT(lbTop.coordX[0] < h_PARAMS.unit.Length);
        ASSERT(lbTop.coordY[0] < h_PARAMS.unit.Length);

        cout << "lbTop.coordX[lbTop.sizeX - 1]=" << lbTop.coordX[lbTop.sizeX - 1] << endl;
        cout << "lbSize[0]) * unit.Length=" << h_PARAMS.lbSize[0] * h_PARAMS.unit.Length << endl;
        ASSERT(lbTop.coordX[lbTop.sizeX - 1] > h_PARAMS.lbSize[0] * h_PARAMS.unit.Length);
        cout << "lbTop.coordY[lbTop.sizeY - 1]=" << lbTop.coordY[lbTop.sizeY - 1] << endl;
        cout << "lbSize[1]) * unit.Length=" << h_PARAMS.lbSize[1] * h_PARAMS.unit.Length << endl;
        ASSERT(lbTop.coordY[lbTop.sizeY - 1] > h_PARAMS.lbSize[1] * h_PARAMS.unit.Length);
        
        // @todo This was previously OpenMP parallel, critical section around generateNode()
        for (unsigned int ix = 1; ix < h_PARAMS.lbSize[0] - 1; ++ix) {
            for (unsigned int iy = 1; iy < h_PARAMS.lbSize[1] - 1; ++iy) {
                for (unsigned int iz = 1; iz < h_PARAMS.lbSize[2] - 1; ++iz) {
                    const tVect nodePosition = tVect(ix, iy, iz) * h_PARAMS.unit.Length;
                    const double distanceFromTopography = lbTop.distance(nodePosition);
                    
                    if (distanceFromTopography < 0.0 && distanceFromTopography>-1.0 * surfaceThickness) {
                        const unsigned int it = ix + iy * h_PARAMS.lbSize[0] + iz * h_PARAMS.lbSize[0] * h_PARAMS.lbSize[1];
                        generateNode(it, STAT_WALL);
                        h_nodes.type[it] = TOPO;
                    }
                }
            }
        }
    }
}
void LB2::initializeInterface() {
    // TODO Currently only default case is supported
    // creates an interface electing interface cells from active cells
    if (h_PARAMS.lbTopographySurface) {
        // Formerly setTopographySurface()
        // @todo This was previously OpenMP parallel, critical section around generateNode()
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            if (h_nodes.type[it] == GAS) {
                // control is done in real coordinates
                const tVect nodePosition = h_PARAMS.getPosition(it) * h_PARAMS.unit.Length;
                const double surfaceIsoparameterHere = lbTop.surfaceIsoparameter(nodePosition);
                if (surfaceIsoparameterHere > 0.0 && surfaceIsoparameterHere <= 1.0) {// setting solidIndex
                    generateNode(it, LIQUID);
                }
            }
        }
    } else {
        switch (problemName) {
        case NONE:
        case SHEARCELL:
        case AVALANCHE:
        case DRUM:
        case NET:
        case BARRIER:
        case ZHOU:
        case OPENBARRIER:
        case HONGKONG:
        case STVINCENT:
        case STAVA:
        case NIGRO:
        case CAROLINE:
        case DAMBREAK:
        case GRAY_DAMBREAK:
        case GRAY_DAMBREAK_2D:
        case INCLINEFLOW:
        case HOURGLASS:
        case IERVOLINO:
        case IERVOLINO_2D:
        case IERVOLINO_CYLINDERTEST:
        case HEAP:
        case TRIAXIAL:
        case JOP:
        case WILL:
        case WILL_SETTLING:
        case MANGENEY:
        case GRAY:
        case ESERCITAZIONE:
        case FILIPPO_SILOS:
        case HK_SMALL:
        case HK_LARGE:
        case KELVIN:
        case SHEARCELL2023:
        case INTRUDER:
        case OBJMOVING:
        default:
            {
                cout << "Initializing interface using box defined in config file:" << endl;
                cout << "X=(" << double(h_PARAMS.freeSurfaceBorders[0]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[1]) * h_PARAMS.unit.Length << ")" << endl;
                cout << "Y=(" << double(h_PARAMS.freeSurfaceBorders[2]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[3]) * h_PARAMS.unit.Length << ")" << endl;
                cout << "Z=(" << double(h_PARAMS.freeSurfaceBorders[4]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[5]) * h_PARAMS.unit.Length << ")" << endl;
                for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
                    if (h_nodes.type[it] == GAS) {
                        // creating fluid cells
                        const tVect pos = h_PARAMS.getPosition(it);
                        if ((pos.x > h_PARAMS.freeSurfaceBorders[0]) &&
                            (pos.x < h_PARAMS.freeSurfaceBorders[1]) &&
                            (pos.y > h_PARAMS.freeSurfaceBorders[2]) &&
                            (pos.y < h_PARAMS.freeSurfaceBorders[3]) &&
                            (pos.z > h_PARAMS.freeSurfaceBorders[4]) &&
                            (pos.z < h_PARAMS.freeSurfaceBorders[5])) {
                            generateNode(it, LIQUID);
                        }
                    }
                }
                break;
            }
        }
    }
}
void LB2::generateNode(unsigned int coord, types typeHere) {

    // set type
    h_nodes.type[coord] = typeHere;
    h_nodes.p[coord] = false;  // setOutsideParticle()
    h_nodes.age[coord] = 0.0;

    // TODO Add it to list of known non-gas nodes?

    // find neighbor indices
    const std::array<unsigned int, lbmDirec> neighborCoord = h_nodes.findNeighbors(coord);

    h_nodes.basal[coord] = false;

    // set centrifugal acceleration
    h_nodes.centrifugalForce[coord] = computeCentrifugal(h_nodes.getPosition(coord), PARAMS.rotationCenter, PARAMS.rotationSpeed);

    // assign neighbor nodes
    for (unsigned int j = 1; j < lbmDirec; ++j) {
        // linearized coordinate of neighbor nodes
        const unsigned int link = neighborCoord[j];
        // check if node at that location exists
        if (link < h_nodes.count && h_nodes.type[link] != GAS) {
            // assign neighbor for local node
            h_nodes.d[j * h_nodes.count + coord] = link;
            // if neighbor node is also active, link it to local node
            if (h_nodes.isActive(coord)) {
                h_nodes.d[opp[j] * h_nodes.count + link] = coord;
                if (h_nodes.isWall(link)) {
                    h_nodes.basal[coord] = true;
                }
            }
        } else {
            h_nodes.d[j * h_nodes.count + coord] = std::numeric_limits<unsigned int>::max();
        }
    }
}
void LB2::initializeVariables() { 
    cout << "Initializing variables" << endl;
    // note that interface is not defined here. All fluid, interface and gas cells are uninitialized at the moment
    // calculate maximum height of the fluid

    // find "taller" and "deepest" points
    double minProjection = std::numeric_limits<double>::max();
    double maxProjection = -std::numeric_limits<double>::max();
        
    if (!PARAMS.solveCentrifugal) {
        // TODO openmp reduction?
        for (unsigned int i = 0; i < h_nodes.count; ++i) {
            if (h_nodes.isActive(i)) {
                const tVect position = h_nodes.getPosition(i);
                const double projection = position.dot(PARAMS.lbF);
                minProjection = std::min(minProjection, projection);
                maxProjection = std::max(maxProjection, projection);
            }
        }
        cout << "minProjection = " << minProjection << endl;
    } else {
        // TODO openmp reduction?
        for (unsigned int i = 0; i < h_nodes.count; ++i) {
            if (h_nodes.isActive(i)) {
                const tVect position = h_nodes.getPosition(i);
                const double projection = position.dot(h_nodes.centrifugalForce[i]);
                minProjection = std::min(minProjection, projection);
                maxProjection = std::max(maxProjection, projection);
            }
        }
        cout << "minProjection = " << minProjection << endl;
    }

    // checking for boundary between gas and fluid and assigning interface properties
    // at this point fluid cells contain actual fluid cells and potential interface cells, so we create the node anyway
    double massFluid = 0.0;
    double massInterface = 0.0;
    // TODO openmp?
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.type[i] == LIQUID) {
            // check if it is interface
            for (int j = 1; j < lbmDirec; ++j) {
                unsigned int linkNode = h_nodes.d[j * h_nodes.count + i];
                if (linkNode == std::numeric_limits<unsigned int>::max()) {
                    h_nodes.type[i] = INTERFACE;
                    break;
                }
            }
        }
        // now assign macroscopic quantities accordingly
        // FLUID NODES ////
        if (h_nodes.type[i] == LIQUID) {
            massFluid += 1.0;
            // setting macroscopic variables
            // density is calculated using hydrostatic profile
            const tVect position = h_nodes.getPosition(i);
            if (!PARAMS.solveCentrifugal) {
                const double projection = position.dot(PARAMS.lbF);
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity + 3.0 * PARAMS.fluidMaterial.initDensity * (projection-minProjection), PARAMS.initVelocity, PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, Zero);
            } else {
                const double projection = position.dot(h_nodes.centrifugalForce[i]);
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity + 3.0 * PARAMS.fluidMaterial.initDensity * (projection-minProjection), PARAMS.initVelocity, PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, PARAMS.rotationSpeed);
            }
        }// INTERFACE NODES ////
        else if (h_nodes.type[i] == INTERFACE) {
            massInterface += 0.5;
            // setting macroscopic variables
            h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, PARAMS.initVelocity, 0.5 * PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, PARAMS.rotationSpeed);
        }

    }
    cout << "Approximate volume = " << massFluid * PARAMS.unit.Volume << " (fluid body), " << massInterface * PARAMS.unit.Volume << " (interface), " << (massFluid + massInterface) * PARAMS.unit.Volume << " (tot), " << endl;
}
void LB2::initializeWalls() {
    cout << "Initializing wall nodes" << endl;
    const double zero = 0.0;

    std::vector<unsigned int> wallNodes;

    // initializing wall nodes
    // note that, in the hypothesis that these walls are not evolving, only nodes at the interface need creation
    // TODO openmp?
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.isWall(i)) {
            // initialize node
            // STATIC WALL NODES ////
            if (h_nodes.type[i] == STAT_WALL ||
                h_nodes.type[i] == SLIP_STAT_WALL ||
                h_nodes.type[i] == OBJ ||
                h_nodes.type[i] == TOPO) {
                // reset velocity and mass (useful for plotting)
                // density=0.0; velocity=(0.0,0.0,0.0), mass=0.0; viscosity=0.0; force=(0.0,0.0,0.0)
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, Zero, zero, zero, Zero, 1.0, Zero);
            }// DYNAMIC WALL NODES ////
            else if (h_nodes.type[i] == DYN_WALL || 
                     h_nodes.type[i] == SLIP_DYN_WALL || 
                     h_nodes.type[i] == CYL) {
                // need to define velocity. It could be part of a cylinder or wall, we check both
                tVect solidVelocity;
                const tVect nodePosition = h_nodes.getPosition(i);
                unsigned int solidIndex = h_nodes.solidIndex[i];
                // wall
                if (solidIndex < h_walls.count && nodePosition.insidePlane(h_walls.p[solidIndex] / PARAMS.unit.Length, h_walls.n[solidIndex])) {
                    solidVelocity = h_walls.getSpeed(solidIndex, nodePosition * PARAMS.unit.Length) / PARAMS.unit.Speed;
                }// cylinder
                else if (solidIndex < h_cylinders.count && !nodePosition.insideCylinder(h_cylinders.p1[solidIndex] / PARAMS.unit.Length, h_cylinders.naxes[solidIndex], 0.0, h_cylinders.R[solidIndex] / PARAMS.unit.Length)) {
                    solidVelocity = h_cylinders.getSpeed(solidIndex, nodePosition * PARAMS.unit.Length) / PARAMS.unit.Speed;
                }// objects
                else if (solidIndex < h_objects.count && nodePosition.insideSphere(h_objects.x0[solidIndex] / PARAMS.unit.Length, h_objects.r[solidIndex] / PARAMS.unit.Length)) {
                    solidVelocity = h_objects.x1[solidIndex] / PARAMS.unit.Speed;
                }
                // reset velocity and mass (useful for plotting)
                // density=0.0; velocity=solidVelocity, mass=0.0; viscosity=0.0; force=(0.0,0.0,0.0)
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, solidVelocity, zero, zero, Zero, 1.0, PARAMS.rotationSpeed);
            }
            // add node to list
            wallNodes.push_back(i);
        }
    }
    // Allocate the wall nodes storage
    h_nodes.wallCount = static_cast<unsigned int>(wallNodes.size());
    h_nodes.wallI = static_cast<unsigned int*>(malloc(h_nodes.wallCount * sizeof(unsigned int)));
    memcpy(h_nodes.wallI, wallNodes.data(), h_nodes.wallCount * sizeof(unsigned int));    
}
void LB2::initializeLists() {
    cout << "Resetting lists ...";

    // note that interface is not defined here. All fluid, interface and gas cells are 0 at the moment
    std::vector<unsigned int> fluidNodes;
    std::vector<unsigned int> interfaceNodes;

    // creating list and initialize macroscopic variables for all nodes except walls
    // TODO OpenMP?
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.type[i] == LIQUID) {
            fluidNodes.push_back(i);
        } else if (h_nodes.type[i] == INTERFACE) {
            interfaceNodes.push_back(i);
        }
    }

    // Array to Buffer
    assert(!h_nodes.fluidI);
    h_nodes.fluidCount = static_cast<unsigned int>(fluidNodes.size());
    h_nodes.fluidI = static_cast<unsigned int*>(malloc(h_nodes.fluidCount * sizeof(unsigned int)));
    h_nodes.fluidAlloc = h_nodes.fluidCount;
    memcpy(h_nodes.fluidI, fluidNodes.data(), h_nodes.fluidCount * sizeof(unsigned int));

    assert(!h_nodes.interfaceI);
    h_nodes.interfaceCount = static_cast<unsigned int>(interfaceNodes.size());
    h_nodes.interfaceI = static_cast<unsigned int*>(malloc(h_nodes.interfaceCount * sizeof(unsigned int)));
    h_nodes.interfaceAlloc = h_nodes.interfaceCount;
    memcpy(h_nodes.interfaceI, interfaceNodes.data(), h_nodes.interfaceCount * sizeof(unsigned int));

    // Build a sorted active nodes list
    fluidNodes.insert(fluidNodes.end(), interfaceNodes.begin(), interfaceNodes.end());
    std::sort(fluidNodes.begin(), fluidNodes.end());

    // Array to buffer
    assert(!h_nodes.activeI);
    h_nodes.activeCount = static_cast<unsigned int>(fluidNodes.size());
    h_nodes.activeI = static_cast<unsigned int*>(malloc(h_nodes.activeCount * sizeof(unsigned int)));
    memcpy(h_nodes.activeI, fluidNodes.data(), h_nodes.activeCount * sizeof(unsigned int));
    
    cout << " done" << endl;
}
void LB2::step(const DEM &dem, bool io_demSolver) {
    this->syncDEM(dem.elmts, dem.particles, dem.walls, dem.objects);

    if (io_demSolver) {
        this->latticeBoltzmannCouplingStep(dem.newNeighborList);
    }

    if (dem.demTime >= dem.demInitialRepeat) {
        this->latticeBoltzmannStep();

        // Lattice Boltzmann core steps
        if (PARAMS.freeSurface) {
            this->latticeBoltzmannFreeSurfaceStep();
        }
    }
}

void LB2::syncDEM(const elmtList &elmts, const particleList &particles, const wallList &walls, const objectList &objects) {
    // Sync DEM data to structure of arrays format (and device memory)
    syncElements<IMPL>(elmts);
    syncParticles<IMPL>(particles);
    syncWalls<IMPL>(walls);
    syncObjects<IMPL>(objects);
}
void LB2::setParams(const LBParams& params, const LBInitParams& initParams, bool skip_sync) {
    // CPU
    h_PARAMS = params;
    init_params = initParams;
    // CUDA
    if (!skip_sync)
        syncParams();
}
void LB2::syncParams() {
#ifdef USE_CUDA
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PARAMS), &h_PARAMS, sizeof(LBParams)));
#endif
}
