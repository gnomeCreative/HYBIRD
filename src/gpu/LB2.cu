#include "hip/hip_runtime.h"
#include "LB2.h"

#include <cstdlib>

#include "cuda_helper.h"

#include "DEM.h"

/**
 * Storage for static members must be defined
 */
std::unique_ptr<CubTempMem> CubTempMem::_singletonT;
std::unique_ptr<CubTempMem> CubTempMem::_singletonB;

/**
 * (Temporary) DEM data synchronisation
 * Reformat DEM data to structure of arrays (for CPU), and copy it to device (for CUDA)
 */
template<>
bool LB2::syncElements<CPU>(const elmtList &elements) {
    bool componentsHasGrown = false;
    if (h_elements.count < elements.size()) {
        // Grow host buffers
         if (h_elements.x1) {
             free(h_elements.x1);
             free(h_elements.wGlobal);
             free(h_elements.FHydro);
             free(h_elements.MHydro);
             free(h_elements.fluidVolume);
         }
         h_elements.x1 = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.wGlobal = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.FHydro = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.MHydro = (tVect*)malloc(elements.size() * sizeof(tVect));
         h_elements.fluidVolume = (double*)malloc(elements.size() * sizeof(double));
    }
    // Update size
    h_elements.count = static_cast<unsigned int>(elements.size());
    // Repackage host particle data from array of structures, to structure of arrays
     for (unsigned int i = 0; i < h_elements.count; ++i) {
         h_elements.x1[i] = elements[i].x1;
         h_elements.wGlobal[i] = elements[i].wGlobal;
         h_elements.FHydro[i] = elements[i].FHydro;
         // h_elements.MHydro[i] = elements[i].MHydro; // This is zero'd before use in latticeBoltzmannStep()
         // h_elements.fluidVolume[i] = elements[i].fluidVolume; // This is zero'd before use in latticeBoltzmannStep()
     }
    // Construct the components storage
    {
        // Allocate memory for componentsData
        unsigned int totalComponents = 0;
        for (const auto& e : elements)
            totalComponents += static_cast<unsigned int>(e.components.size());
        if (!h_elements.componentsIndex || totalComponents >= h_elements.componentsIndex[elements.size()]) {
            if (h_elements.componentsData)
                free(h_elements.componentsData);
            h_elements.componentsData = (unsigned int*)malloc(totalComponents * sizeof(unsigned int));
            componentsHasGrown = true;
        }
        // Allocate componentsIndex if first pass
        if (!h_elements.componentsIndex)
            h_elements.componentsIndex = (unsigned int*)malloc((elements.size() + 1) * sizeof(unsigned int));
        // Fill componentsIndex and componentsData
        totalComponents = 0;
        for (int i = 0; i < elements.size(); ++i) {
            h_elements.componentsIndex[i] = totalComponents;
            if (!elements[i].components.empty()) {
                memcpy(h_elements.componentsData + totalComponents, elements[i].components.data(), elements[i].components.size() * sizeof(unsigned int));
                totalComponents += static_cast<unsigned int>(elements[i].components.size());
            }
        }
        h_elements.componentsIndex[elements.size()] = totalComponents;
    }
    return componentsHasGrown;
}
template<>
void LB2::syncParticles<CPU>(const particleList &particles) {
    if (h_particles.count < particles.size()) {
        // Grow host buffers
        if (h_particles.clusterIndex) {
            free(h_particles.clusterIndex);
            free(h_particles.r);
            free(h_particles.x0);
            free(h_particles.radiusVec);
        }
        h_particles.clusterIndex = (unsigned int*)malloc(particles.size() * sizeof(unsigned int));
        h_particles.r = (double*)malloc(particles.size() * sizeof(double));
        h_particles.x0 = (tVect*)malloc(particles.size() * sizeof(tVect));
        h_particles.radiusVec = (tVect*)malloc(particles.size() * sizeof(tVect));
    }
    // Update size
    h_particles.count = static_cast<unsigned int>(particles.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (int i = 0; i < h_particles.count; ++i) {
        h_particles.clusterIndex[i] = particles[i].clusterIndex;
        h_particles.r[i] = particles[i].r;
        h_particles.x0[i] = particles[i].x0;
        h_particles.radiusVec[i] = particles[i].radiusVec;
    }
}
template<>
void LB2::syncCylinders<CPU>(const cylinderList &cylinders) {
    if (h_cylinders.count < cylinders.size()) {
        // Grow host buffers
        if (h_cylinders.p1) {
            free(h_cylinders.p1);
            free(h_cylinders.p2);
            free(h_cylinders.R);
            free(h_cylinders.naxes);
            free(h_cylinders.omega);
            free(h_cylinders.moving);
        }
        h_cylinders.p1 = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.p2 = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.R = (double*)malloc(cylinders.size() * sizeof(double));
        h_cylinders.naxes = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.omega = (tVect*)malloc(cylinders.size() * sizeof(tVect));
        h_cylinders.moving = (bool*)malloc(cylinders.size() * sizeof(bool));
    }
    // Update size
    h_cylinders.count = static_cast<unsigned int>(cylinders.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_cylinders.count; ++i) {
        h_cylinders.p1[i] = cylinders[i].p1;
        h_cylinders.p2[i] = cylinders[i].p2;
        h_cylinders.R[i] = cylinders[i].R;
        h_cylinders.naxes[i] = cylinders[i].naxes;
        h_cylinders.omega[i] = cylinders[i].omega;
        h_cylinders.moving[i] = cylinders[i].moving;
    }
}
template<>
void LB2::syncWalls<CPU>(const wallList &walls) {
    if (h_walls.count < walls.size()) {
        // Grow host buffers
        if (h_walls.n) {
            free(h_walls.n);
            free(h_walls.p);
            free(h_walls.rotCenter);
            free(h_walls.omega);
            free(h_walls.vel);
            free(h_walls.FHydro);
        }
        h_walls.n = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.p = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.rotCenter = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.omega = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.vel = (tVect*)malloc(walls.size() * sizeof(tVect));
        h_walls.FHydro = (tVect*)malloc(walls.size() * sizeof(tVect));
    }
    // Update size
    h_walls.count = static_cast<unsigned int>(walls.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_walls.count; ++i) {
        h_walls.n[i] = walls[i].n;
        h_walls.p[i] = walls[i].p;
        h_walls.rotCenter[i] = walls[i].rotCenter;
        h_walls.omega[i] = walls[i].omega;
        h_walls.vel[i] = walls[i].vel;
        // h_walls.FHydro[i] = walls[i].FHydro; // Zero'd before use in streaming()
    }
}
template<>
void LB2::syncObjects<CPU>(const objectList &objects) {
    if (h_objects.count < objects.size()) {
        // Grow host buffers
        if (h_objects.r) {
            free(h_objects.r);
            free(h_objects.x0);
            free(h_objects.x1);
            free(h_objects.FHydro);
        }
        h_objects.r = (double*)malloc(objects.size() * sizeof(double));
        h_objects.x0 = (tVect*)malloc(objects.size() * sizeof(tVect));
        h_objects.x1 = (tVect*)malloc(objects.size() * sizeof(tVect));
        h_objects.FHydro = (tVect*)malloc(objects.size() * sizeof(tVect));
    }
    // Update size
    h_objects.count = static_cast<unsigned int>(objects.size());
    // Repackage host particle data from array of structures, to structure of arrays
    for (unsigned int i = 0; i < h_objects.count; ++i) {
        h_objects.r[i] = objects[i].r;
        h_objects.x0[i] = objects[i].x0;
        h_objects.x1[i] = objects[i].x1;
        // h_objects.FHydro[i] = objects[i].FHydro; // Zero'd before use in streaming()
    }
}
#ifdef USE_CUDA
template<>
bool LB2::syncElements<CUDA>(const elmtList &elements) {
    if (!d_elements) {
        CUDA_CALL(hipMalloc(&d_elements, sizeof(Element2)));
    }
    // @todo copy hd_elements to d_elements
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    bool componentsHasGrown = this->syncElements<CPU>(elements);
    bool updateDeviceStruct = false;
    if (hd_elements.count < elements.size()) {
        if (hd_elements.x1) {
            CUDA_CALL(hipFree(hd_elements.x1));
            CUDA_CALL(hipFree(hd_elements.wGlobal));
            CUDA_CALL(hipFree(hd_elements.FHydro));
            CUDA_CALL(hipFree(hd_elements.MHydro));
            CUDA_CALL(hipFree(hd_elements.fluidVolume));
        }
        // Initially allocate device buffers except components
        CUDA_CALL(hipMalloc(&hd_elements.x1, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.wGlobal, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.FHydro, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.MHydro, hd_elements.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_elements.fluidVolume, hd_elements.count * sizeof(double)));
        updateDeviceStruct = true;
    }
    if (componentsHasGrown || !hd_elements.componentsIndex) {
        // Allocate components
        if (hd_elements.componentsIndex)
            CUDA_CALL(hipFree(hd_elements.componentsIndex));
        if (hd_elements.componentsData)
            CUDA_CALL(hipFree(hd_elements.componentsData));
        // Allocate components
        CUDA_CALL(hipMalloc(&hd_elements.componentsIndex, (h_elements.count + 1) * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&hd_elements.componentsData, h_elements.componentsIndex[h_elements.count] * sizeof(unsigned int)));
        updateDeviceStruct = true;
    }
    // Update size
    hd_elements.count = elements.size();
    if (updateDeviceStruct) {
        // Copy updated device pointers to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(d_elements, &hd_elements, sizeof(Element2), hipMemcpyHostToDevice));
    } else {
        // Copy updated device pointers to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(&d_elements->count, &hd_elements.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_elements.x1, &h_elements.x1, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.wGlobal, &h_elements.wGlobal, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.FHydro, &h_elements.FHydro, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_elements.MHydro, &h_elements.MHydro, h_elements.count * sizeof(tVect), hipMemcpyHostToDevice)); // This is zero'd before use in latticeBoltzmannStep()
    // CUDA_CALL(hipMemcpy(hd_elements.fluidVolume, &h_elements.fluidVolume, h_elements.count * sizeof(double), hipMemcpyHostToDevice)); // This is zero'd before use in latticeBoltzmannStep()
    CUDA_CALL(hipMemcpy(hd_elements.componentsIndex, h_elements.componentsIndex, (h_elements.count + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_elements.componentsData, h_elements.componentsData, h_elements.componentsIndex[h_elements.count] * sizeof(unsigned int), hipMemcpyHostToDevice));
    return componentsHasGrown;
}
template<>
void LB2::syncParticles<CUDA>(const particleList &particles) {
    if (!d_particles) {
        CUDA_CALL(hipMalloc(&d_particles, sizeof(Particle2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncParticles<CPU>(particles);
    if (hd_particles.count < particles.size()) {
        // Grow device buffers
        if (hd_particles.clusterIndex) {
            CUDA_CALL(hipFree(hd_particles.clusterIndex));
            CUDA_CALL(hipFree(hd_particles.r));
            CUDA_CALL(hipFree(hd_particles.x0));
            CUDA_CALL(hipFree(hd_particles.radiusVec));
        }
        CUDA_CALL(hipMalloc(&hd_particles.clusterIndex, h_particles.count * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&hd_particles.r, h_particles.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_particles.x0, h_particles.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_particles.radiusVec, h_particles.count * sizeof(tVect)));
        hd_particles.count = h_particles.count;
        // Copy updated device pointers to device (@todo When/where is d_particles allocated??)
        CUDA_CALL(hipMemcpy(d_particles, &h_particles, sizeof(Particle2), hipMemcpyHostToDevice));
    } else if(hd_particles.count != particles.size()) {
        // Buffer has shrunk, so just update size
        hd_particles.count = static_cast<unsigned int>(particles.size());
        // Copy updated particle count to device
        CUDA_CALL(hipMemcpy(&d_particles->count, &h_particles.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_particles.clusterIndex, h_particles.clusterIndex, h_particles.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.r, h_particles.r, h_particles.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.x0, h_particles.x0, h_particles.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_particles.radiusVec, h_particles.radiusVec, h_particles.count * sizeof(tVect), hipMemcpyHostToDevice));
}
template<>
void LB2::syncCylinders<CUDA>(const cylinderList &cylinders) {
    if (!d_cylinders) {
        CUDA_CALL(hipMalloc(&d_cylinders, sizeof(Cylinder2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncCylinders<CPU>(cylinders);
    if (hd_cylinders.count < cylinders.size()) {
        // Grow device buffers
        if (hd_cylinders.p1) {
            CUDA_CALL(hipFree(hd_cylinders.p1));
            CUDA_CALL(hipFree(hd_cylinders.p2));
            CUDA_CALL(hipFree(hd_cylinders.R));
            CUDA_CALL(hipFree(hd_cylinders.naxes));
            CUDA_CALL(hipFree(hd_cylinders.omega));
            CUDA_CALL(hipFree(hd_cylinders.moving));
        }
        CUDA_CALL(hipMalloc(&hd_cylinders.p1, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.p2, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.R, h_cylinders.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_cylinders.naxes, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.omega, h_cylinders.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_cylinders.moving, h_cylinders.count * sizeof(bool)));
        hd_cylinders.count = h_cylinders.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_cylinders, &hd_cylinders, sizeof(Cylinder2), hipMemcpyHostToDevice));
    } else if(hd_cylinders.count != cylinders.size()) {
        // Buffer has shrunk, so just update size
        hd_cylinders.count = static_cast<unsigned int>(cylinders.size());
        // Copy updated particle count to device (@todo When/where is d_elements allocated??)
        CUDA_CALL(hipMemcpy(&d_cylinders->count, &h_walls.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_cylinders.p1, h_cylinders.p1, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.p2, h_cylinders.p2, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.R, h_cylinders.R, h_cylinders.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.naxes, h_cylinders.naxes, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.omega, h_cylinders.omega, h_cylinders.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_cylinders.moving, h_cylinders.moving, h_cylinders.count * sizeof(bool), hipMemcpyHostToDevice));
}
template<>
void LB2::syncWalls<CUDA>(const wallList &walls) {
    if (!d_walls) {
        CUDA_CALL(hipMalloc(&d_walls, sizeof(Wall2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncWalls<CPU>(walls);
    if (hd_walls.count < walls.size()) {
        // Grow device buffers
        if (hd_walls.n) {
            CUDA_CALL(hipFree(hd_walls.n));
            CUDA_CALL(hipFree(hd_walls.p));
            CUDA_CALL(hipFree(hd_walls.rotCenter));
            CUDA_CALL(hipFree(hd_walls.omega));
            CUDA_CALL(hipFree(hd_walls.vel));
            CUDA_CALL(hipFree(hd_walls.FHydro));
        }
        CUDA_CALL(hipMalloc(&hd_walls.n, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.p, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.rotCenter, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.omega, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.vel, h_walls.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_walls.FHydro, h_walls.count * sizeof(tVect)));
        hd_walls.count = h_walls.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_walls, &hd_walls, sizeof(Wall2), hipMemcpyHostToDevice));
    } else if(hd_walls.count != walls.size()) {
        // Buffer has shrunk, so just update size
        hd_walls.count = static_cast<unsigned int>(walls.size());
        // Copy updated particle count to device (@todo When/where is d_walls allocated??)
        CUDA_CALL(hipMemcpy(&d_walls->count, &h_walls.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_walls.n, h_walls.n, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.p, h_walls.p, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.rotCenter, h_walls.rotCenter, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.omega, h_walls.omega, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_walls.vel, h_walls.vel, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_walls.FHydro, h_walls.FHydro, h_walls.count * sizeof(tVect), hipMemcpyHostToDevice)); // Zero'd before use in streaming()
}
template<>
void LB2::syncObjects<CUDA>(const objectList &walls) {
    if (!d_objects) {
        CUDA_CALL(hipMalloc(&d_objects, sizeof(Object2)));
    }
    // Copy latest particle data from HOST DEM to the device
    // @todo Can these copies be done ahead of time async?
    // @todo These copies will be redundant when DEM is moved to CUDA
    this->syncObjects<CPU>(walls);
    if (hd_objects.count < walls.size()) {
        // Grow device buffers
        if (hd_objects.r) {
            CUDA_CALL(hipFree(hd_objects.r));
            CUDA_CALL(hipFree(hd_objects.x0));
            CUDA_CALL(hipFree(hd_objects.x1));
            CUDA_CALL(hipFree(hd_objects.FHydro));
        }
        CUDA_CALL(hipMalloc(&hd_objects.r, h_objects.count * sizeof(double)));
        CUDA_CALL(hipMalloc(&hd_objects.x0, h_objects.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_objects.x1, h_objects.count * sizeof(tVect)));
        CUDA_CALL(hipMalloc(&hd_objects.FHydro, h_objects.count * sizeof(tVect)));
        hd_objects.count = h_objects.count;
        // Copy updated device pointers to device
        CUDA_CALL(hipMemcpy(d_objects, &hd_objects, sizeof(Object2), hipMemcpyHostToDevice));
    } else if(hd_objects.count != walls.size()) {
        // Buffer has shrunk, so just update size
        hd_objects.count = static_cast<unsigned int>(walls.size());
        // Copy updated particle count to device (@todo When/where is d_objects allocated??)
        CUDA_CALL(hipMemcpy(&d_objects->count, &h_objects.count, sizeof(unsigned int), hipMemcpyHostToDevice));
    }
    // Copy data to device buffers
    CUDA_CALL(hipMemcpy(hd_objects.r, h_objects.r, h_objects.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_objects.x0, h_objects.x0, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(hd_objects.x1, h_objects.x1, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice));
    // CUDA_CALL(hipMemcpy(hd_objects.FHydro, h_objects.FHydro, h_objects.count * sizeof(tVect), hipMemcpyHostToDevice)); // Zero'd before use in streaming()
}
#endif


/**
 * initializeParticleBoundaries()
 */
__host__ __device__ __forceinline__ inline double common_initializeParticleBoundaries(const unsigned int i, Node2* nodes, Particle2* particles) {
    // Fetch the index of the (active) node being processed
    const unsigned int an_i = nodes->activeI[i];
    const tVect node_position = nodes->getPosition(an_i);
    for (unsigned int p_i = 0; p_i < particles->count; ++p_i) {
        const tVect convertedPosition = particles->x0[p_i] / PARAMS.unit.Length;
        // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
        const double convertedRadius = particles->r[p_i] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
        if (node_position.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
            nodes->setInsideParticle(an_i, true);
            nodes->solidIndex[an_i] = p_i;
            return nodes->mass[an_i];  // @todo in original code it doesn't break after setting
        }
    }
    return 0.0;
}
template<>
double LB2::initializeParticleBoundaries<CPU>() {
    // Reset all nodes to outside
    memset(hd_nodes.p, 0, h_nodes.count * sizeof(bool));

    // @todo can we parallelise at a higher level?
    double totalParticleMass = 0;
#pragma omp parallel for reduction(+:totalParticleMass) 
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        totalParticleMass += common_initializeParticleBoundaries(i, d_nodes, d_particles);
    }
    return totalParticleMass;
}
#ifdef USE_CUDA
__global__ void d_initializeParticleBoundaries(Node2* d_nodes, Particle2* d_particles, double *node_in_particle_mass) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    const double t = common_initializeParticleBoundaries(i, d_nodes, d_particles);

    if (t != 0.0) {
        atomicAdd(node_in_particle_mass, t);
    }
}
template<>
double LB2::initializeParticleBoundaries<CUDA>() {
    // Reset all nodes to outside
    CUDA_CALL(hipMemset(hd_nodes.p, 0, h_nodes.count * sizeof(bool)));
    // Initialise reduction variable
    auto &t = CubTempMem::GetTempSingleton();
    t.resize(sizeof(double));
    double *d_return = static_cast<double*>(t.getPtr());
    double h_return = 0;
    CUDA_CALL(hipMemcpy(d_return, &h_return, sizeof(double), hipMemcpyHostToDevice));

    // Launch cuda kernel to update
    // @todo Try unrolling this, so 1 thread per node+particle combination (2D launch?)
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_initializeParticleBoundaries << <gridSize, blockSize >> > (d_nodes, d_particles, d_return);
    CUDA_CHECK();

    // Copy back return value
    CUDA_CALL(hipMemcpy(&h_return, d_return, sizeof(double), hipMemcpyDeviceToHost));
    return h_return;
}
#endif

/**
 * findNewActive()
 */
__host__ __device__ __forceinline__ inline void common_findNewActive(const unsigned int i, Node2* nodes, Particle2* particles, Element2* elements) {
    // Fetch the index of the (active) node being processed
    const unsigned int an_i = nodes->activeI[i];
    if (nodes->p[an_i]) {
        const tVect nodePosition = nodes->getPosition(an_i);
        // solid index to identify cluster
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // in this case check if it has been uncovered (must be out of all particles of the cluster) - we start with a true hypothesis
        // cycling through component particles
        const unsigned int first_component = elements->componentsIndex[clusterIndex];
        const unsigned int last_component = elements->componentsIndex[clusterIndex + 1];
        for (unsigned int j = first_component; j < last_component; ++j) {
            // getting indexes from particle composing the cluster
            const unsigned int componentIndex = elements->componentsData[j];
            // checking if it has been uncovered in component j of the cluster
            // radius need to be increased by half a lattice unit
            // this is because solid boundaries are located halfway between solid and fluid nodes
            const tVect convertedPosition = particles->x0[componentIndex] / PARAMS.unit.Length;
            // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
            const double convertedRadius = particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
            if (nodePosition.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
                // if the node is still inside the element, the hypothesis of new active is not true anymore
                // and we can get out of the cycle
                return;
            }
        }
        // turning up the cell as we didn't exit early
        nodes->setInsideParticle(an_i, false);
    }
}
template<>
void LB2::findNewActive<CPU>() {
    // @todo can we parallelise at a higher level?
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        common_findNewActive(i, d_nodes, d_particles, d_elements);
    }
}
#ifdef USE_CUDA
__global__ void d_findNewActive(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    common_findNewActive(i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::findNewActive<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_findNewActive << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * findNewSolid()
 */
__host__ __device__ __forceinline__ inline void common_findNewSolid(const unsigned int i, Node2* nodes, Particle2* particles, Element2* elements) {
    const unsigned int an_i = nodes->activeI[i];
    if (nodes->isInsideParticle(an_i)) {  // If node is inside particle
        // solid index to identify cluster
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // cycle through first neighbors
        const unsigned int nodeCount = nodes->count;
        for (int k = 1; k < lbmMainDirec; ++k) {
            const unsigned int l_i = nodes->d[nodeCount * k + an_i];
            if (l_i != std::numeric_limits<unsigned int>::max()) {
                // checking if solid particle is close to an active one -> we have an active node to check
                if (!nodes->isInsideParticle(l_i) && nodes->isActive(l_i)) {
                    const tVect linkPosition = nodes->getPosition(l_i);
                    // check if neighbors has been covered (by any of the particles of the cluster) - we start with a false hypothesis
                    // cycling through all components of the cluster
                    const unsigned int first_component = elements->componentsIndex[clusterIndex];
                    const unsigned int last_component = elements->componentsIndex[clusterIndex + 1];
                    for (unsigned int j = first_component; j < last_component; ++j) {
                        // getting component particle index
                        const unsigned int componentIndex = elements->componentsData[j];
                        // check if it getting inside
                        // radius need to be increased by half a lattice unit
                        // this is because solid boundaries are located halfway between solid and fluid nodes
                        // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
                        if (linkPosition.insideSphere(particles->x0[componentIndex] / PARAMS.unit.Length, particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length)) { //-0.5?
                            // if so, then the false hypothesis does not hold true anymore
                            nodes->solidIndex[l_i] = componentIndex;
                            // By setting particle to inside, it won't be checked again, newSolidNodes hence becomes redundant
                            nodes->setInsideParticle(l_i, true);  // @todo Is this a race condition? Multiple nodes may share a link node?
                            // and we exit the cycle
                            break;
                        }
                    }
                }
            }
        }
    }
}
template<>
void LB2::findNewSolid<CPU>() {
    // @todo can we parallelise at a higher level?
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Pass the active node index to the common implementation
        common_findNewSolid(i, d_nodes, d_particles, d_elements);
    }
}

#ifdef USE_CUDA
__global__ void d_findNewSolid(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;
    // Pass the active node index to the common implementation
    common_findNewSolid(i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::findNewSolid<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_findNewSolid, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_findNewSolid << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * checkNewInterfaceParticles()
 */
__host__ __device__ __forceinline__ inline void common_checkNewInterfaceParticles(const unsigned int e_i, Node2* nodes, Particle2* particles, Element2* elements) {
    // INITIAL PARTICLE POSITION ////////////////////////
    if (elements->FHydro[e_i].norm2() == 0.0) {
        const unsigned int first_component = elements->componentsIndex[e_i];
        const unsigned int last_component = elements->componentsIndex[e_i + 1];
        for (unsigned int n = first_component; n < last_component; ++n) {
            const unsigned int componentIndex = elements->componentsData[n];
            const tVect convertedPosition = particles->x0[componentIndex] / PARAMS.unit.Length;
            // @todo pre-compute PARAMS.hydrodynamicRadius / PARAMS.unit.Length ?
            const double convertedRadius = particles->r[componentIndex] * PARAMS.hydrodynamicRadius / PARAMS.unit.Length;
            for (unsigned int i_i = 0; i_i < nodes->interfaceCount; ++i_i) {
                const unsigned int nodeHere = nodes->interfaceI[i_i];
                if (!nodes->isInsideParticle(nodeHere)) {
                    // checking if node is inside a particle
                    const tVect nodePosition = nodes->getPosition(nodeHere);
                    if (nodePosition.insideSphere(convertedPosition, convertedRadius)) { //-0.5?
                        nodes->setInsideParticle(nodeHere, true);
                        nodes->solidIndex[nodeHere] = componentIndex;
                    }
                }
            }
        }
    }
}
template<>
void LB2::checkNewInterfaceParticles<CPU>() {
#pragma omp parallel for
    for (unsigned int e_i = 0; e_i < d_elements->count; ++e_i) {
        common_checkNewInterfaceParticles(e_i, d_nodes, d_particles, d_elements);
    }
}
#ifdef USE_CUDA
__global__ void d_checkNewInterfaceParticles(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to element 
    const unsigned int e_i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (e_i >= d_elements->count) return;
    // Pass the active node index to the common implementation
    common_checkNewInterfaceParticles(e_i, d_nodes, d_particles, d_elements);
}
template<>
void LB2::checkNewInterfaceParticles<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_checkNewInterfaceParticles, 0, h_elements.count);
    // Round up to accommodate required threads
    gridSize = (h_elements.count + blockSize - 1) / blockSize;
    // @todo Are there more elements or particles? This may want to be inverted, and we can go straight to particles rather than components?
    d_checkNewInterfaceParticles << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}
#endif

/**
 * reconstruct()
 * computeHydroForces()
 * collision()
 */
__host__ __device__ __forceinline__ inline void common_computeHydroForces(const unsigned int an_i, Node2* nodes, Particle2* particles, Element2* elements) {
    // resetting hydrodynamic forces on nodes
    nodes->hydroForce[an_i].reset();
    if (nodes->isInsideParticle(an_i)) {
        // getting the index of the particle to compute force in the right object
        const unsigned int index = nodes->coord[an_i];
        const unsigned int particleIndex = nodes->solidIndex[an_i];
        const unsigned int clusterIndex = particles->clusterIndex[particleIndex];
        // calculating velocity of the solid boundary at the node (due to rotation of particles)
        // vectorized radius (real units)
        const tVect radius = nodes->getPosition(index) - particles->x0[particleIndex] / PARAMS.unit.Length + particles->radiusVec[particleIndex] / PARAMS.unit.Length;
        // update velocity of the particle node (u=v_center+omega x radius) (real units)
        const tVect localVel = elements->x1[clusterIndex] / PARAMS.unit.Speed + (elements->wGlobal[clusterIndex].cross(radius)) / PARAMS.unit.AngVel;

        // calculate differential velocity
        const tVect diffVel = nodes->age[an_i] * nodes->age[an_i] * nodes->liquidFraction(an_i) * (nodes->u[an_i] - localVel);

        // force on fluid
        nodes->hydroForce[an_i] += -1.0 * diffVel;

        // force on particle
#ifdef __CUDA_ARCH__
        // CUDA atomics
        atomicAdd(&elements->fluidVolume[clusterIndex], nodes->mass[an_i]);
        atomicAdd(&elements->FHydro[clusterIndex].x, 1.0 * diffVel.x);
        atomicAdd(&elements->FHydro[clusterIndex].y, 1.0 * diffVel.y);
        atomicAdd(&elements->FHydro[clusterIndex].z, 1.0 * diffVel.z);
        const tVect t = 1.0 * radius.cross(diffVel);
        atomicAdd(&elements->MHydro[clusterIndex].x, t.x);
        atomicAdd(&elements->MHydro[clusterIndex].y, t.y);
        atomicAdd(&elements->MHydro[clusterIndex].z, t.z);
#else
        // CPU atomics
#pragma omp atomic update
        elements->fluidVolume[clusterIndex] += nodes->mass[an_i];
#pragma omp atomic update
        elements->FHydro[clusterIndex] += 1.0 * diffVel;
#pragma omp atomic update
        elements->MHydro[clusterIndex] += 1.0 * radius.cross(diffVel);
#endif
    }
}
template<>
void LB2::reconstructHydroCollide<CPU>() {
    // @todo the inside of this loop could be merged with d_reconstructHydroCollide()
#pragma omp parallel for
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        // Convert index to active node index
        const unsigned int an_i = d_nodes->activeI[i];

        // reconstruction of macroscopic variables from microscopic distribution
        // this step is necessary to proceed to the collision step
        d_nodes->reconstruct(an_i);

        // compute interaction forces
        if (d_elements->count) {
            common_computeHydroForces(an_i, d_nodes, d_particles, d_elements);
        }

        //collision operator
        d_nodes->collision(an_i);
    }
}
#ifdef USE_CUDA
__global__ void d_reconstructHydroCollide(Node2* d_nodes, Particle2* d_particles, Element2* d_elements) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;

    // Convert index to active node index
    const unsigned int an_i = d_nodes->activeI[i];

    // reconstruction of macroscopic variables from microscopic distribution
    // this step is necessary to proceed to the collision step
    d_nodes->reconstruct(an_i);

    // compute interaction forces
    if (d_elements->count) {
        common_computeHydroForces(an_i, d_nodes, d_particles, d_elements);
    }

    //collision operator
    d_nodes->collision(an_i);
}
template<>
void LB2::reconstructHydroCollide<CUDA>() {
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_reconstructHydroCollide, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_reconstructHydroCollide << <gridSize, blockSize >> > (d_nodes, d_particles, d_elements);
    CUDA_CHECK();
}

__host__ __device__ void common_streaming(const unsigned int i, Node2* nodes, Wall2* walls) {
    // Convert index to active node index
    const unsigned int an_i = nodes->activeI[i];

    // coefficient for free-surface
    constexpr double C2x2 = 9.0;
    constexpr double C3x2 = 3.0;
    // coefficient for slip conditions
    const double S1 = PARAMS.slipCoefficient;
    const double S2 = (1.0 - PARAMS.slipCoefficient);
    // creating list for collision function @todo can this be precomputed, rather than once per node?
    std::array<double, lbmDirec> staticPres;
    for (int j = 0; j < lbmDirec; j++) {
        staticPres[j] = PARAMS.fluidMaterial.initDensity * coeff[j];
    }

    // coefficient for bounce-back
    constexpr double BBCoeff = 2.0 * 3.0;

    const unsigned int A_OFFSET = an_i * lbmDirec;
    // cycling through neighbours
    for (unsigned int j = 1; j < lbmDirec; ++j) {
        // getting neighbour index
        const unsigned int ln_i = nodes->d[nodes->count * j + an_i];
        // if neighbour is normal fluid cell what follows is true

        if (ln_i == std::numeric_limits<unsigned int>::max()) { // is gas
            // additional variables for equilibrium f computation
            const double usq = nodes->u[an_i].norm2();
            const double vuj = nodes->u[an_i].dot(v[j]);
            // streaming with constant pressure interface
            nodes->f[A_OFFSET + opp[j]] = -nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq);
        } else {
            const unsigned int L_OFFSET = ln_i * lbmDirec;
            // @todo this could be greatly improved by stacking matching cases to reduce divergence
            switch (nodes->type[ln_i]) {
            case INTERFACE:
#ifdef DEBUG
            {
                // TEST USING AGE //////////////////////////////////////
                const double usq = nodes->u[an_i].norm2();
                const double vuj = nodes->u[an_i].dot(v[j]);
                nodes->f[A_OFFSET + opp[j]] = nodes->age[ln_i] * nodes->fs[L_OFFSET + opp[j]] +
                    (1.0 - nodes->age[ln_i]) * (-nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq));
                break;
            }
#endif // INTERFACE falls through to LIQUID if DEBUG not defined
            case LIQUID:
            {
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[L_OFFSET + opp[j]];
                break;
            }
            // for walls there is simple bounce-back
            case STAT_WALL:
            {
#ifndef DEBUG 
                if (nodes->type[an_i] == INTERFACE) {
                    // additional variables for equilibrium f computation
                    const double usq = nodes->u[an_i].norm2();
                    const double vuj = nodes->u[an_i].dot(v[j]);
                    //streaming with constant pressure interface
                    nodes->f[A_OFFSET + opp[j]] = -nodes->fs[A_OFFSET + j] + coeff[j] * PARAMS.fluidMaterial.initDensity * (2.0 + C2x2 * (vuj * vuj) - C3x2 * usq);
                    break;
                }
#endif      
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];

                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, 0.0);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                // Fall through to TOPO
            }
            // for curved walls there is the rule of Mei-Luo-Shyy
            case TOPO:
            {
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                break;
            }
            case OUTLET:
            {
                nodes->f[A_OFFSET + opp[j]] = std::min(nodes->fs[A_OFFSET + opp[j]], nodes->fs[A_OFFSET + j]);
                break;
            }
            // for moving walls there is simple bounce-back with velocity correction
            case DYN_WALL:
            {
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];
                // velocity of the wall
                const tVect vel = nodes->u[ln_i];
                // variation in Bounce-Back due to moving object
                const double BBi = BBCoeff * nodes->n[an_i] * coeff[j] * vel.dot(v[j]); // mass!!!!!

                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, BBi);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                // adding the extra mass to the surplus //@todo extraMass required for parity
                // extraMass = BBi * nodes->mass[an_i];  // redistributeMass() currently not used, so this isn't implemented properly
                break;
            }// for walls there is simple bounce-back
            case OBJ:
            {
                // getting the index of the wall to compute force in the right object
                const unsigned int solidIndex = nodes->solidIndex[ln_i];
                // static pressure is subtracted in order to correctly compute buoyancy for floating objects
                const tVect BBforce = nodes->bounceBackForce(an_i, j, staticPres, 0.0);
                // updating force and torque on the object (lattice units). This point is critical since many nodes update the force on the same object (lattice units)
#ifdef __CUDA_ARCH__
                    // CUDA atomics
                atomicAdd(&walls->FHydro[solidIndex].x, BBforce.x);
                atomicAdd(&walls->FHydro[solidIndex].y, BBforce.y);
                atomicAdd(&walls->FHydro[solidIndex].z, BBforce.z);
#else
                    // CPU atomics
#pragma omp atomic update
                walls->FHydro[solidIndex] += BBforce;
#endif
                nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                break;
            }
            case SLIP_STAT_WALL:
            {
                if (j > 6) {
                    const unsigned int nodeCheck1 = nodes->d[slip1Check[j] * nodes->count + an_i];
                    const unsigned int nodeCheck2 = nodes->d[slip2Check[j] * nodes->count + an_i];
                    // check for the environment
                    const bool active1 = nodeCheck1 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck1);
                    const bool active2 = nodeCheck2 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck2);
                    // given the environment, perform the right operation
                    if (active1 && !active2) {
                        // first
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck1 * lbmDirec + slip1[j]] + S2 * nodes->fs[A_OFFSET + j];
                    }
                    else if (!active1 && active2) {
                        // second
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck2 * lbmDirec + slip2[j]] + S2 * nodes->fs[A_OFFSET + j];
                    }
                    else {
                        // standard BB
                        nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                    }
                }
                else {
                    // standard BB
                    nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j];
                }
                break;
            }
            case SLIP_DYN_WALL:
            {
                // velocity of the wall
                const tVect vel = nodes->u[ln_i];
                // variation in Bounce-Back due to moving object
                const double BBi = BBCoeff * nodes->n[an_i] * coeff[j] * vel.dot(v[j]);
                if (j > 6) {
                    const unsigned int nodeCheck1 = nodes->d[slip1Check[j] * nodes->count + an_i];
                    const unsigned int nodeCheck2 = nodes->d[slip2Check[j] * nodes->count + an_i];
                    // check for the environment
                    const bool active1 = nodeCheck1 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck1);
                    const bool active2 = nodeCheck2 != std::numeric_limits<unsigned int>::max() && nodes->isActive(nodeCheck2);
                    // given the environment, perform the right operation
                    if (active1 && !active2) {
                        // first
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck1 * lbmDirec + slip1[j]] + S2 * (nodes->fs[A_OFFSET + j] - BBi);
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += S2 * nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                    else if (!active1 && active2) {
                        // second
                        nodes->f[A_OFFSET + opp[j]] = S1 * nodes->fs[nodeCheck2 * lbmDirec + slip2[j]] + S2 * (nodes->fs[A_OFFSET + j] - BBi);
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += S2 * nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                    else {
                        // standard BB
                        nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                        // adding the extra mass to the surplus //@todo extraMass required for parity
                        // extraMass += nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                    }
                }
                else {
                    // standard BB
                    nodes->f[A_OFFSET + opp[j]] = nodes->fs[A_OFFSET + j] - BBi;
                    // adding the extra mass to the surplus //@todo extraMass required for parity
                    // extraMass += nodes->mass[an_i] * BBi;  // redistributeMass() currently not used, so this isn't implemented properly
                }
                break;
            }
            case UNUSED:
            case GAS:
            case PERIODIC:
            case CYL:
            default:
            {
                {
                    // @todo This may print out of order if multiple threads break in parallel
                    tVect pos = nodes->getPosition(an_i);
                    printf("%u(%f, %f, %f) %s TYPE ERROR:\n", an_i, pos.x, pos.y, pos.z, typeString(nodes->type[an_i]));
                    for (unsigned int k = 1; k < lbmDirec; ++k) {
                        printf("before error: j=%u link=%u\n", k, nodes->coord[nodes->d[k * nodes->count + an_i]]);
                    }
                    pos = nodes->getPosition(ln_i);
                    printf("(%f, %f, %f) %s TYPE ERROR\n", pos.x, pos.y, pos.z, typeString(nodes->type[ln_i]));
                    // @todo aborting from CUDA is harder, especially if the printf() is to be saved
#ifndef __CUDA_ARCH__
                    std::abort();
#endif
                    return;
                }
                break;

            }
            }
        }
    }
}
template<>
void LB2::streaming<CPU>() {
    // STREAMING STEP
    // Init forces to zero
    hd_walls.initForces<CPU>();
    hd_objects.initForces<CPU>();
    // Init streaming support vector
    hd_nodes.store<CPU>();

#pragma omp parallel for // @note extraMass reduction is not currently implemented
    for (unsigned int i = 0; i < d_nodes->activeCount; ++i) {
        common_streaming(i, d_nodes, d_walls);
    }

    // redistributing extra mass due to bounce back to interface cells
    // redistributeMass(extraMass);  // extraMass hasn't been implemented properly
}
#ifdef USE_CUDA
__global__ void d_streaming(Node2* d_nodes, Wall2* d_walls) {
    // Get unique CUDA thread index, which corresponds to active node 
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Kill excess threads early
    if (i >= d_nodes->activeCount) return;

    common_streaming(i, d_nodes, d_walls);
}
template<>
void LB2::streaming<CUDA>() {
    // STREAMING STEP
    // Init forces to zero
    hd_walls.initForces<CUDA>();
    hd_objects.initForces<CUDA>();
    // Init streaming support vector
    hd_nodes.store<CUDA>();

    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, h_nodes.activeCount);
    // Round up to accommodate required threads
    gridSize = (h_nodes.activeCount + blockSize - 1) / blockSize;
    d_streaming << <gridSize, blockSize >> > (d_nodes, d_walls);
    CUDA_CHECK();

#ifdef _DEBUG
    CUDA_CALL(hipMemcpy(h_nodes.f, hd_nodes.f, sizeof(double) * h_nodes.count * lbmDirec, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.activeI, hd_nodes.activeI, sizeof(unsigned int) * h_nodes.activeCount, hipMemcpyDeviceToHost));
    for (unsigned int in = 0; in < h_nodes.activeCount; ++in) {
        const unsigned int a_i = h_nodes.activeI[in];
        for (unsigned int j = 1; j < lbmDirec; ++j) {
            if (h_nodes.f[a_i * lbmDirec + j] == 0) {
                cout << "Error!" << endl;
            }
        }
    }
#endif

    // redistributing extra mass due to bounce back to interface cells
    // redistributeMass(extraMass);  // extraMass hasn't been implemented properly
}
#endif
template<>
void LB2::shiftToPhysical<CPU>() {
    for (unsigned int i = 0; i < d_elements->count; ++i) {
        d_elements->FHydro[i] *= PARAMS.unit.Force;
        d_elements->MHydro[i] *= PARAMS.unit.Torque;
        d_elements->fluidVolume[i] *= PARAMS.unit.Volume;
    }
    for (unsigned int i = 0; i < d_walls->count; ++i) {
        d_walls->FHydro[i] *= PARAMS.unit.Force;
    }
    for (unsigned int i = 0; i < d_objects->count; ++i) {
        d_objects->FHydro[i] *= PARAMS.unit.Force;
    }
}
#ifdef USE_CUDA
__global__ void d_shiftToPhysical(Element2* d_elements, Wall2* d_walls, Object2* d_objects) {
    // Get unique CUDA thread index
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_elements->count) {
        d_elements->FHydro[i] *= PARAMS.unit.Force;
        d_elements->MHydro[i] *= PARAMS.unit.Torque;
        d_elements->fluidVolume[i] *= PARAMS.unit.Volume;
    }
    if (i < d_walls->count) {
        d_walls->FHydro[i] *= PARAMS.unit.Force;
    }
    if (i < d_objects->count) {
        d_objects->FHydro[i] *= PARAMS.unit.Force;
    }
}
template<>
void LB2::shiftToPhysical<CUDA>() {
    // Launch enough threads to accomodate everything
    const unsigned int maxCount = std::max(std::max(h_elements.count, h_walls.count), h_objects.count);
    // Launch cuda kernel to update
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, d_initializeParticleBoundaries, 0, maxCount);
    // Round up to accommodate required threads
    gridSize = (maxCount + blockSize - 1) / blockSize;
    d_shiftToPhysical << <gridSize, blockSize >> > (d_elements, d_walls, d_objects);
    CUDA_CHECK();
}
#endif


void LB2::latticeBoltzmannCouplingStep(bool& newNeighbourList) {
    // identifies which nodes need to have an update due to particle movement
    // the complexity arises from trying to keep the scaling as close to linear as possible
    // maybe the best idea is to do this in two step:
    // 1) the first is to check for new active nodes and initialise them
    // 2) the second is checking for new solid nodes.
    // this automatically check also for the hideous case of particle to particle double transition

    /**
     * @todo The parallelisation of each of these methods should be reviewed
     *       Most are 2D loops, the range of each being unclear
     *       Likewise, can OpenMP parallel block be moved outside of each member?
     */

     // first we check if a new neighbour table has been defined. In that case, the indexing needs to be reinitialised
    if (newNeighbourList) {
        cout << endl << "New neighbour list" << endl;
        this->initializeParticleBoundaries<IMPL>();
        newNeighbourList = false;
    }
    else {
        // SOLID TO ACTIVE CHECK
        // @note Calling this directly after initializeParticleBoundaries() is redundant, hence else
        this->findNewActive<IMPL>();
    }

    // ACTIVE TO SOLID CHECK
    this->findNewSolid<IMPL>();

    if (PARAMS.freeSurface) {
        this->checkNewInterfaceParticles<IMPL>();
    }
}
void LB2::latticeBoltzmannStep() {
    // Reconstruct active list
    hd_nodes.cleanLists<IMPL>();

    // Initializing the elements forces (lattice units)
    hd_elements.initElements<IMPL>();

    // Initialise lattice boltzmann force vector
    if (!h_PARAMS.forceField) {
        h_PARAMS.lbF.reset();
        syncParams();
    }

    // reconstruct(), computeHydroForces(), collision()
    // Reconstruct macroscopic variables from microscopic distribution
    // Compute interaction forces with DEM elmts
    // Collision step
    this->reconstructHydroCollide<IMPL>();

    // Streaming operator
    this->streaming<IMPL>();

    // Shift element/wall/object forces and torques to physical units
    this->shiftToPhysical<IMPL>();
}
#endif

Node2& LB2::getNodes() {
#ifdef USE_CUDA
    // If using CUDA, data is on device by default, so sync back.
    if (hd_nodes.count > h_nodes.count) {
        // Resize main buffers
        if (h_nodes.f) free(h_nodes.f);
        h_nodes.f = static_cast<double*>(malloc(hd_nodes.count * lbmDirec * sizeof(double)));
        if (h_nodes.fs) free(h_nodes.fs);
        h_nodes.fs = static_cast<double*>(malloc(hd_nodes.count * lbmDirec * sizeof(double)));
        if (h_nodes.n) free(h_nodes.n);
        h_nodes.n = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.u) free(h_nodes.u);
        h_nodes.u = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.hydroForce) free(h_nodes.hydroForce);
        h_nodes.hydroForce = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.centrifugalForce) free(h_nodes.centrifugalForce);
        h_nodes.centrifugalForce = static_cast<tVect*>(malloc(hd_nodes.count * sizeof(tVect)));
        if (h_nodes.mass) free(h_nodes.mass);
        h_nodes.mass = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.visc) free(h_nodes.visc);
        h_nodes.visc = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.basal) free(h_nodes.basal);
        h_nodes.basal = static_cast<bool*>(malloc(hd_nodes.count * sizeof(bool)));
        if (h_nodes.friction) free(h_nodes.friction);
        h_nodes.friction = static_cast<double*>(malloc(hd_nodes.count * sizeof(double)));
        if (h_nodes.age) free(h_nodes.age);
        h_nodes.age = static_cast<float*>(malloc(hd_nodes.count * sizeof(float)));
        if (h_nodes.solidIndex) free(h_nodes.solidIndex);
        h_nodes.solidIndex = static_cast<unsigned int*>(malloc(hd_nodes.count * sizeof(unsigned int)));
        if (h_nodes.d) free(h_nodes.d);
        h_nodes.d = static_cast<unsigned int*>(malloc(hd_nodes.count * lbmDirec * sizeof(unsigned int)));
        if (h_nodes.curved) free(h_nodes.curved);
        h_nodes.curved = static_cast<unsigned int*>(malloc(hd_nodes.count * sizeof(unsigned int)));
        if (h_nodes.type) free(h_nodes.type);
        h_nodes.type = static_cast<types*>(malloc(hd_nodes.count * sizeof(types)));
        if (h_nodes.p) free(h_nodes.p);
        h_nodes.p = static_cast<bool*>(malloc(hd_nodes.count * sizeof(bool)));
    }
    h_nodes.count = hd_nodes.count;
    // Resize misc buffers
    if (hd_nodes.activeCount > h_nodes.activeAlloc) {
        if (h_nodes.activeI) free(h_nodes.activeI);
        h_nodes.activeI = static_cast<unsigned int*>(malloc(hd_nodes.activeCount * sizeof(unsigned int)));
    }
    h_nodes.activeCount = hd_nodes.activeCount;
    if (hd_nodes.interfaceCount > h_nodes.interfaceCount) {
        if (h_nodes.interfaceI) free(h_nodes.interfaceI);
        h_nodes.interfaceI = static_cast<unsigned int*>(malloc(hd_nodes.interfaceCount * sizeof(unsigned int)));
    }
    h_nodes.interfaceCount = hd_nodes.interfaceCount;
    if (hd_nodes.fluidCount > h_nodes.fluidCount) {
        if (h_nodes.fluidI) free(h_nodes.fluidI);
        h_nodes.fluidI = static_cast<unsigned int*>(malloc(hd_nodes.fluidCount * sizeof(unsigned int)));
    }
    h_nodes.fluidCount = hd_nodes.fluidCount;
    if (hd_nodes.wallCount > h_nodes.wallCount) {
        if (h_nodes.wallI) free(h_nodes.wallI);
        h_nodes.wallI = static_cast<unsigned int*>(malloc(hd_nodes.wallCount * sizeof(unsigned int)));
    }
    h_nodes.wallCount = hd_nodes.wallCount;
    if (hd_nodes.curveCount > h_nodes.curveCount) {
        if (h_nodes.curves) free(h_nodes.curves);
        h_nodes.curves = static_cast<curve*>(malloc(hd_nodes.curveCount * sizeof(curve)));
    }
    h_nodes.curveCount = hd_nodes.curveCount;
    // Copy main buffers back to host
    CUDA_CALL(hipMemcpy(h_nodes.coord, hd_nodes.coord, h_nodes.count * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.f, hd_nodes.f, h_nodes.count * lbmDirec * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.fs, hd_nodes.fs, h_nodes.count * lbmDirec * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.n, hd_nodes.n, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.u, hd_nodes.u, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.hydroForce, hd_nodes.hydroForce, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.centrifugalForce, hd_nodes.centrifugalForce, h_nodes.count * sizeof(tVect), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.mass, hd_nodes.mass, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.visc, hd_nodes.visc, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.basal, hd_nodes.basal, h_nodes.count * sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.friction, hd_nodes.friction, h_nodes.count * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.age, hd_nodes.age, h_nodes.count * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.solidIndex, hd_nodes.solidIndex, h_nodes.count * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.d, hd_nodes.d, h_nodes.count * lbmDirec * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.curved, hd_nodes.curved, h_nodes.count * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.type, hd_nodes.type, h_nodes.count * sizeof(types), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.p, hd_nodes.p, h_nodes.count * sizeof(bool), hipMemcpyDeviceToHost));
    // Copy misc buffers back to host
    CUDA_CALL(hipMemcpy(h_nodes.activeI, hd_nodes.activeI, h_nodes.activeCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.interfaceI, hd_nodes.interfaceI, h_nodes.interfaceCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.fluidI, hd_nodes.fluidI, h_nodes.fluidCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.wallI, hd_nodes.wallI, h_nodes.wallCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_nodes.curves, hd_nodes.curves, h_nodes.curveCount * sizeof(curve), hipMemcpyDeviceToHost));
#endif
    return h_nodes;
}
void LB2::initDeviceNodes() {
#ifdef USE_CUDA
    // Allocate the main storage
    if (d_nodes) {
        fprintf(stderr, "LB2::initDeviceNodes() should only be called once.");
        throw std::exception();
    }
    CUDA_CALL(hipMalloc(&d_nodes, sizeof(Node2)));
    // Build HD struct
    hd_nodes.activeCount = h_nodes.activeCount;
    hd_nodes.activeAlloc = h_nodes.activeCount;
    CUDA_CALL(hipMalloc(&hd_nodes.activeI, hd_nodes.activeCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.activeI, h_nodes.activeI, hd_nodes.activeCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.interfaceCount = h_nodes.interfaceCount;
    CUDA_CALL(hipMalloc(&hd_nodes.interfaceI, hd_nodes.interfaceCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.interfaceI, h_nodes.interfaceI, hd_nodes.interfaceCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.fluidCount = h_nodes.fluidCount;
    CUDA_CALL(hipMalloc(&hd_nodes.fluidI, hd_nodes.fluidCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.fluidI, h_nodes.fluidI, hd_nodes.fluidCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.wallCount = h_nodes.wallCount;
    CUDA_CALL(hipMalloc(&hd_nodes.wallI, hd_nodes.wallCount * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.wallI, h_nodes.wallI, hd_nodes.wallCount * sizeof(unsigned int), hipMemcpyHostToDevice));
    hd_nodes.curveCount = h_nodes.curveCount;
    CUDA_CALL(hipMalloc(&hd_nodes.curves, hd_nodes.curveCount * sizeof(curve)));
    CUDA_CALL(hipMemcpy(hd_nodes.curves, h_nodes.curves, hd_nodes.curveCount * sizeof(curve), hipMemcpyHostToDevice));
    hd_nodes.count = h_nodes.count;
    CUDA_CALL(hipMalloc(&hd_nodes.coord, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.coord, h_nodes.coord, hd_nodes.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.f, hd_nodes.count * lbmDirec * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.f, h_nodes.f, hd_nodes.count * lbmDirec * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.fs, hd_nodes.count * lbmDirec * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.fs, h_nodes.fs, hd_nodes.count * lbmDirec * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.n, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.n, h_nodes.n, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.u, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.u, h_nodes.u, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.hydroForce, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.hydroForce, h_nodes.hydroForce, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.centrifugalForce, hd_nodes.count * sizeof(tVect)));
    CUDA_CALL(hipMemcpy(hd_nodes.centrifugalForce, h_nodes.centrifugalForce, hd_nodes.count * sizeof(tVect), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.mass, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.mass, h_nodes.mass, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.visc, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.visc, h_nodes.visc, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.basal, hd_nodes.count * sizeof(bool)));
    CUDA_CALL(hipMemcpy(hd_nodes.basal, h_nodes.basal, hd_nodes.count * sizeof(bool), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.friction, hd_nodes.count * sizeof(double)));
    CUDA_CALL(hipMemcpy(hd_nodes.friction, h_nodes.friction, hd_nodes.count * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.age, hd_nodes.count * sizeof(float)));
    CUDA_CALL(hipMemcpy(hd_nodes.age, h_nodes.age, hd_nodes.count * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.solidIndex, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.solidIndex, h_nodes.solidIndex, hd_nodes.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.d, hd_nodes.count * lbmDirec * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.d, h_nodes.d, hd_nodes.count * lbmDirec * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.curved, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.curved, h_nodes.curved, hd_nodes.count * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.type, hd_nodes.count * sizeof(unsigned int)));
    CUDA_CALL(hipMemcpy(hd_nodes.type, h_nodes.type, hd_nodes.count * sizeof(types), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_nodes.p, hd_nodes.count * sizeof(bool)));
    CUDA_CALL(hipMemcpy(hd_nodes.p, h_nodes.p, hd_nodes.count * sizeof(bool), hipMemcpyHostToDevice));
    // Copy struct containing device pointers and counts
    CUDA_CALL(hipMemcpy(d_nodes, &hd_nodes, sizeof(Node2), hipMemcpyHostToDevice));
#endif
    
}

void LB2::init(cylinderList& cylinders, wallList& walls, particleList& particles, objectList& objects, bool externalSolveCoriolis, bool externalSolveCentrifugal) {
    // Convert from AoS format to SoA and copy to device
    syncCylinders<IMPL>(cylinders);
    syncWalls<IMPL>(walls);
    syncParticles<IMPL>(particles);
    syncObjects<IMPL>(objects);

    //  Lattice Boltzmann initialization steps

    // switchers for apparent accelerations
    h_PARAMS.solveCoriolis = externalSolveCoriolis;
    h_PARAMS.solveCentrifugal = externalSolveCentrifugal;

    // first comes the initialization of the data structures
    cout << "Initializing nodes containers and types" << endl;
    // total number of nodes
    h_PARAMS.totPossibleNodes = h_PARAMS.lbSize[0] * h_PARAMS.lbSize[1] * h_PARAMS.lbSize[2];

    // Count the number of nodes to be created, so memory can be pre-allocated
    std::map<unsigned int, NewNode> newNodes;
    // application of lattice boundaries
    countLatticeBoundaries(newNodes);
    // then the initial node type must be identified for every node (if not specified, it is already Fluid)
    countTypes(newNodes, walls, cylinders, objects);

    // Build a temporary buffer of curves
    std::vector<curve> curves;

    ifstream fluidFileID;
    if (h_PARAMS.lbRestart) {
        // open fluid restart file
        fluidFileID.open(init_params.lbRestartFile.c_str(), ios::in);
        ASSERT(fluidFileID.is_open());
        // check if the restart file size is ok
        unsigned int restartX, restartY, restartZ, restartNodes;
        fluidFileID >> restartX;
        fluidFileID >> restartY;
        fluidFileID >> restartZ;
        fluidFileID >> restartNodes;
        ASSERT(restartX == h_PARAMS.lbSize[0]);
        ASSERT(restartY == h_PARAMS.lbSize[1]);
        ASSERT(restartZ == h_PARAMS.lbSize[2]);
        // read active nodes from file and generate
        // @todo
        throw std::exception("lbRestart is not yet supported.");
        // restartInterface(fluidFileID, restartNodes);
    } else {
        // initialize interface
        countInterface(newNodes);
        // Create and initialize active nodes
        generateInitialNodes(newNodes, curves);
    }

    // initialize variables for wall nodes
    initializeWalls();

    // initializing curved properties
    initializeCurved(curves);

    // Allocate the curves storage
    h_nodes.curveCount = static_cast<unsigned int>(curves.size());
    h_nodes.curves = static_cast<curve*>(malloc(h_nodes.curveCount * sizeof(curve)));
    memcpy(h_nodes.curves, curves.data(), h_nodes.curveCount * sizeof(curve));

    // initialize h_nodes's fluid, interface and active lists
    initializeLists();

    // Setup hd_nodes, copy it to d_nodes
    initDeviceNodes();

    // application of particle initial position
    const double inside_mass = initializeParticleBoundaries<IMPL>();

    // in case mass needs to be kept constant, compute it here
    h_PARAMS.totalMass = 0.0;
    if (h_PARAMS.imposeFluidVolume) {
        // volume and mass is the same in lattice units
        h_PARAMS.totalMass = h_PARAMS.imposedFluidVolume / h_PARAMS.unit.Volume;
    } else {
        switch (problemName) {
        case DRUM:
        {
            h_PARAMS.totalMass = h_PARAMS.fluidMass / h_PARAMS.unit.Mass;
            break;
        }
        case STAVA:
        {
            h_PARAMS.totalMass = 200000.0 / h_PARAMS.unit.Volume;
            break;
        }
        default:
        {
            // @todo This needs to be calculated on device if using CUDA, h_nodes is out of date follow initializeParticleBoundaries
            for (int i = 0; i < h_nodes.activeCount; ++i) {
                const unsigned int a_i = h_nodes.activeI[i];
                if (!h_nodes.isInsideParticle(a_i)) {
                    h_PARAMS.totalMass += h_nodes.mass[a_i];
                }
            }
            break;
        }
        }
    }
    if (h_PARAMS.increaseVolume) {
        h_PARAMS.deltaVolume /= h_PARAMS.unit.Volume;
        h_PARAMS.deltaTime /= h_PARAMS.unit.Time;
    }

    syncParams();

    cout << "Done with initialization" << endl;
}

void LB2::countLatticeBoundaries(std::map<unsigned int, NewNode> &newNodes) {
    // Based on initialiseLatticeBoundaries()
    // XY
    for (unsigned int x = 0; x < h_PARAMS.lbSize[0]; ++x) {
        for (unsigned int y = 0; y < h_PARAMS.lbSize[1]; ++y) {
            // bottom
            newNodes.emplace(h_PARAMS.getIndex(x, y, 0), NewNode{ h_PARAMS.boundary[4] });
            // top
            newNodes.emplace(h_PARAMS.getIndex(x, y, h_PARAMS.lbSize[2] - 1), NewNode{ h_PARAMS.boundary[5] });
        }
    }

    // YZ
    for (unsigned int y = 0; y < h_PARAMS.lbSize[1]; ++y) {
        for (unsigned int z = 0; z < h_PARAMS.lbSize[2]; ++z) {
            // bottom
            newNodes.emplace(h_PARAMS.getIndex(0, y, z), NewNode{ h_PARAMS.boundary[0] });
            // top
            newNodes.emplace(h_PARAMS.getIndex(h_PARAMS.lbSize[0] - 1, y, z), NewNode{ h_PARAMS.boundary[1] });
        }
    }

    // ZX
    for (unsigned int z = 0; z < h_PARAMS.lbSize[2]; ++z) {
        for (unsigned int x = 0; x < h_PARAMS.lbSize[0]; ++x) {
            // bottom
            newNodes.emplace(h_PARAMS.getIndex(x, 0, z), NewNode{ h_PARAMS.boundary[2] });
            // top
            newNodes.emplace(h_PARAMS.getIndex(x, h_PARAMS.lbSize[1] - 1, z), NewNode{ h_PARAMS.boundary[3] });
        }
    }
}
void LB2::countTypes(std::map<unsigned int, NewNode> &newNodes, const wallList& walls, const cylinderList& cylinders, const objectList& objects) {
    countWallBoundaries(newNodes, walls);
    // application of solid cylinders
    countCylinderBoundaries(newNodes, cylinders);
    // application of objects
    countObjectBoundaries(newNodes, objects);
    // initializing topography if one is present
    countTopography(newNodes);
}
void LB2::countWallBoundaries(std::map<unsigned int, NewNode> &newNodes, const wallList& walls) {
    // Based on initializeWallBoundaries()
    // const double wallThickness = 2.0 * h_PARAMS.unit.Length;
    // SOLID WALLS ////////////////////////
    for (unsigned int iw = 0; iw < walls.size(); ++iw) {
        const tVect convertedWallp = walls[iw].p / h_PARAMS.unit.Length;
        const tVect normHere = walls[iw].n;
        const unsigned int indexHere = walls[iw].index;
        const bool slipHere = walls[iw].slip;
        const bool movingHere = walls[iw].moving;
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            // check if the node is solid
            // all walls have max thickness 2 nodes
            const tVect pos = h_PARAMS.getPosition(it);
            const double wallDistance = pos.distance2Plane(convertedWallp, normHere);
            if (wallDistance > -2.0 && wallDistance < 0.0) {
                //check for borders in limted walls
                if (walls[iw].limited) {
                    const double xHere = pos.x * h_PARAMS.unit.Length;
                    const double yHere = pos.y * h_PARAMS.unit.Length;
                    const double zHere = pos.z * h_PARAMS.unit.Length;
                    // check if beyond limits
                    if (xHere < walls[iw].xMin || xHere > walls[iw].xMax ||
                        yHere < walls[iw].yMin || yHere > walls[iw].yMax ||
                        zHere < walls[iw].zMin || zHere > walls[iw].zMax) {
                        continue;
                    }
                }
                // setting type: 5-6=slip, 7-8=no-slip
                if (slipHere) {
                    // setting type for slip: 5=static, 6=moving
                    if (movingHere) {
                        const auto nn = newNodes.emplace(it, NewNode{ SLIP_DYN_WALL, iw });
                        nn.first->second.solidIndex = iw; // Update solidIndex, even if node creation already requested
                    } else {
                        const auto nn = newNodes.emplace(it, NewNode{ SLIP_STAT_WALL, iw });
                        nn.first->second.solidIndex = iw; // Update solidIndex, even if node creation already requested
                    }
                } else {
                    // setting type for no-slip: 7=static, 8=moving
                    if (movingHere) {
                        const auto nn = newNodes.emplace(it, NewNode{ DYN_WALL, iw });
                        nn.first->second.solidIndex = iw; // Update solidIndex, even if node creation already requested
                    } else {
                        const auto nn = newNodes.emplace(it, NewNode{ STAT_WALL, iw });
                        nn.first->second.solidIndex = iw; // Update solidIndex, even if node creation already requested
                    }
                }
            }
        }
    }

}
void LB2::countObjectBoundaries(std::map<unsigned int, NewNode> &newNodes, const objectList& objects) {
    // Based on initializeObjectBoundaries()
    // SOLID WALLS ////////////////////////
    for (int io = 0; io < objects.size(); ++io) {
        const tVect convertedPosition = objects[io].x0 / h_PARAMS.unit.Length;
        const double convertedRadius = objects[io].r / h_PARAMS.unit.Length;
        const unsigned int indexHere = objects[io].index;
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            const tVect nodePosition = h_PARAMS.getPosition(it);
            if (nodePosition.insideSphere(convertedPosition, convertedRadius)) {
                newNodes.emplace(it, NewNode{ OBJ, indexHere });
            }
        }
    }
}
void LB2::countCylinderBoundaries(std::map<unsigned int, NewNode> &newNodes, const cylinderList& cylinders) {
    // Based on initializeCylinderBoundaries()
    // SOLID CYLINDERS ////////////////////////
    for (int ic = 0; ic < cylinders.size(); ++ic) {

        const tVect convertedCylinderp1 = cylinders[ic].p1 / h_PARAMS.unit.Length;
        const tVect naxesHere = cylinders[ic].naxes;
        const double convertedRadius = cylinders[ic].R / h_PARAMS.unit.Length;
        const unsigned int indexHere = cylinders[ic].index;
        const bool slipHere = cylinders[ic].slip;
        const bool movingHere = cylinders[ic].moving;
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            // creating solid cells
            const bool isOutside = h_PARAMS.getPosition(it).insideCylinder(convertedCylinderp1, naxesHere, convertedRadius, convertedRadius + 3.0);
            const bool isInside = h_PARAMS.getPosition(it).insideCylinder(convertedCylinderp1, naxesHere, max(convertedRadius - 3.0, 0.0), convertedRadius);
            if ((cylinders[ic].type == FULL && isInside) ||
                (cylinders[ic].type == EMPTY && isOutside)) {
                //check for borders in limted walls
                if (cylinders[ic].limited) {
                    const tVect here = h_PARAMS.getPosition(it) * h_PARAMS.unit.Length;
                    // check if beyond limits
                    if (here.x < cylinders[ic].xMin || here.x > cylinders[ic].xMax ||
                        here.y < cylinders[ic].yMin || here.y > cylinders[ic].yMax ||
                        here.z < cylinders[ic].zMin || here.z > cylinders[ic].zMax) {
                        continue;
                    }
                }
                // setting type: 5-6=slip, 7-8=no-slip
                if (slipHere) {
                    // setting type for slip: 5=static, 6=moving
                    if (movingHere) {
                        newNodes.emplace(it, NewNode{ SLIP_DYN_WALL, indexHere });
                    } else {
                        newNodes.emplace(it, NewNode{ SLIP_STAT_WALL, indexHere });
                    }
                } else {
                    // setting type for no-slip: 7=static, 8=moving
                    if (movingHere) {
                        newNodes.emplace(it, NewNode{ DYN_WALL, indexHere });
                    } else {
                        newNodes.emplace(it, NewNode{ STAT_WALL, indexHere });
                    }
                }
            }
        }
    }
}
void LB2::countTopography(std::map<unsigned int, NewNode> &newNodes) {
    // Based on initializeTopography()
    
    const double surfaceThickness = 1.75 * h_PARAMS.unit.Length;

    // TOPOGRAPHY ////////////////////////
    if (h_PARAMS.lbTopography) {
        lbTop.readFromFile(init_params.lbTopographyFile, h_PARAMS.translateTopographyX, h_PARAMS.translateTopographyY, h_PARAMS.translateTopographyZ);
        lbTop.show();
        // check if topography grid contains the fluid domain
        ASSERT(lbTop.coordX[0] < h_PARAMS.unit.Length);
        ASSERT(lbTop.coordY[0] < h_PARAMS.unit.Length);

        cout << "lbTop.coordX[lbTop.sizeX - 1]=" << lbTop.coordX[lbTop.sizeX - 1] << endl;
        cout << "lbSize[0]) * unit.Length=" << h_PARAMS.lbSize[0] * h_PARAMS.unit.Length << endl;
        ASSERT(lbTop.coordX[lbTop.sizeX - 1] > h_PARAMS.lbSize[0] * h_PARAMS.unit.Length);
        cout << "lbTop.coordY[lbTop.sizeY - 1]=" << lbTop.coordY[lbTop.sizeY - 1] << endl;
        cout << "lbSize[1]) * unit.Length=" << h_PARAMS.lbSize[1] * h_PARAMS.unit.Length << endl;
        ASSERT(lbTop.coordY[lbTop.sizeY - 1] > h_PARAMS.lbSize[1] * h_PARAMS.unit.Length);


        for (unsigned int ix = 1; ix < h_PARAMS.lbSize[0] - 1; ++ix) {
            for (unsigned int iy = 1; iy < h_PARAMS.lbSize[1] - 1; ++iy) {
                for (unsigned int iz = 1; iz < h_PARAMS.lbSize[2] - 1; ++iz) {
                    const tVect nodePosition = tVect(ix, iy, iz) * h_PARAMS.unit.Length;
                    const double distanceFromTopography = lbTop.distance(nodePosition);
                    
                    if (distanceFromTopography < 0.0 && distanceFromTopography>-1.0 * surfaceThickness) {
                        const unsigned int it = ix + iy * h_PARAMS.lbSize[0] + iz * h_PARAMS.lbSize[0] * h_PARAMS.lbSize[1];
                        newNodes.emplace(it, NewNode{ TOPO });
                    }
                }
            }
        }
    }
}
void LB2::countInterface(std::map<unsigned int, NewNode> &newNodes) {
    // Based on initializeInterface()
    // @Currently only default case is supported
    // creates an interface electing interface cells from active cells
    if (h_PARAMS.lbTopographySurface) {
        // Formerly setTopographySurface()
        for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
            if (newNodes.find(it) == newNodes.end()) {
                // control is done in real coordinates
                const tVect nodePosition = h_PARAMS.getPosition(it) * h_PARAMS.unit.Length;
                const double surfaceIsoparameterHere = lbTop.surfaceIsoparameter(nodePosition);
                if (surfaceIsoparameterHere > 0.0 && surfaceIsoparameterHere <= 1.0) {// setting solidIndex
                    newNodes.emplace(it, NewNode{ LIQUID });
                }
            }
        }
    } else {
        switch (problemName) {
        case NONE:
        case SHEARCELL:
        case AVALANCHE:
        case DRUM:
        case NET:
        case BARRIER:
        case ZHOU:
        case OPENBARRIER:
        case HONGKONG:
        case STVINCENT:
        case STAVA:
        case NIGRO:
        case CAROLINE:
        case DAMBREAK:
        case GRAY_DAMBREAK:
        case GRAY_DAMBREAK_2D:
        case INCLINEFLOW:
        case HOURGLASS:
        case IERVOLINO:
        case IERVOLINO_2D:
        case IERVOLINO_CYLINDERTEST:
        case HEAP:
        case TRIAXIAL:
        case JOP:
        case WILL:
        case WILL_SETTLING:
        case MANGENEY:
        case GRAY:
        case ESERCITAZIONE:
        case FILIPPO_SILOS:
        case HK_SMALL:
        case HK_LARGE:
        case KELVIN:
        case SHEARCELL2023:
        case INTRUDER:
        case OBJMOVING:
        default:
            {
                cout << "X=(" << double(h_PARAMS.freeSurfaceBorders[0]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[1]) * h_PARAMS.unit.Length << ")" << endl;
                cout << "Y=(" << double(h_PARAMS.freeSurfaceBorders[2]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[3]) * h_PARAMS.unit.Length << ")" << endl;
                cout << "Z=(" << double(h_PARAMS.freeSurfaceBorders[4]) * h_PARAMS.unit.Length << ", " << double(h_PARAMS.freeSurfaceBorders[5]) * h_PARAMS.unit.Length << ")" << endl;
                for (unsigned int it = 0; it < h_PARAMS.totPossibleNodes; ++it) {
                    if (newNodes.find(it) == newNodes.end()) {
                        // creating fluid cells
                        const tVect pos = h_PARAMS.getPosition(it);
                        if ((pos.x > h_PARAMS.freeSurfaceBorders[0]) &&
                            (pos.x < h_PARAMS.freeSurfaceBorders[1]) &&
                            (pos.y > h_PARAMS.freeSurfaceBorders[2]) &&
                            (pos.y < h_PARAMS.freeSurfaceBorders[3]) &&
                            (pos.z > h_PARAMS.freeSurfaceBorders[4]) &&
                            (pos.z < h_PARAMS.freeSurfaceBorders[5])) {
                            newNodes.emplace(it, NewNode{ LIQUID });
                        }
                    }
                }
                break;
            }
        }
    }
}
void LB2::generateInitialNodes(const std::map<unsigned int, NewNode> &newNodes, std::vector<curve> &curves) {
    // Allocate enough memory for these nodes
    assert(h_nodes.count == 0);  // No nodes should exist at the time this is called
    h_nodes.count = newNodes.size();
    // Allocate host buffers
    h_nodes.coord = static_cast<unsigned int*>(malloc(h_nodes.count * sizeof(unsigned int)));
    h_nodes.f = static_cast<double*>(malloc(h_nodes.count * lbmDirec * sizeof(double)));
    h_nodes.fs = static_cast<double*>(malloc(h_nodes.count * lbmDirec * sizeof(double)));
    h_nodes.n = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.u = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.hydroForce = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.centrifugalForce = static_cast<tVect*>(malloc(h_nodes.count * sizeof(tVect)));
    h_nodes.mass = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.visc = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.basal = static_cast<bool*>(malloc(h_nodes.count * sizeof(bool)));
    h_nodes.friction = static_cast<double*>(malloc(h_nodes.count * sizeof(double)));
    h_nodes.age = static_cast<float*>(malloc(h_nodes.count * sizeof(float)));
    h_nodes.solidIndex = static_cast<unsigned int*>(malloc(h_nodes.count * sizeof(unsigned int)));
    h_nodes.d = static_cast<unsigned int*>(malloc(h_nodes.count * lbmDirec * sizeof(unsigned int)));
    h_nodes.curved = static_cast<unsigned int*>(malloc(h_nodes.count * sizeof(unsigned int)));
    h_nodes.type = static_cast<types*>(malloc(h_nodes.count * sizeof(types)));
    h_nodes.p = static_cast<bool*>(malloc(h_nodes.count * sizeof(bool)));
    // Zero initialisation
    memset(h_nodes.f, 0, h_nodes.count * lbmDirec * sizeof(double));
    memset(h_nodes.fs, 0, h_nodes.count * lbmDirec * sizeof(double));
    memset(h_nodes.n, 0, h_nodes.count * sizeof(double));
    memset(h_nodes.u, 0, h_nodes.count * sizeof(tVect));
    memset(h_nodes.hydroForce, 0, h_nodes.count * sizeof(tVect));
    memset(h_nodes.mass, 0, h_nodes.count * sizeof(double));
    // h_nodes.visc is instead init to 1 below
    memset(h_nodes.basal, 0, h_nodes.count * sizeof(bool));
    memset(h_nodes.friction, 0, h_nodes.count * sizeof(double));
    memset(h_nodes.age, 0, h_nodes.count * sizeof(float));
    memset(h_nodes.curved, std::numeric_limits<unsigned int>::max(), h_nodes.count * sizeof(unsigned int));
    memset(h_nodes.p, 0, h_nodes.count * sizeof(bool));
    // Perform the generateNode() loop for each item in newNodes
    std::map<unsigned int, unsigned int> idIndexMap;
    {
        unsigned int i = 0;
        for (const auto& [id, nn] : newNodes) {
            idIndexMap.emplace(id, i);
            h_nodes.visc[i] = 1;
            h_nodes.coord[i] = id;
            h_nodes.type[i] = nn.type;
            h_nodes.solidIndex[i] = nn.solidIndex;
            // set centrifugal acceleration
            h_nodes.centrifugalForce[i] = computeCentrifugal(h_nodes.getPosition(i), PARAMS.rotationCenter, PARAMS.rotationSpeed);
            ++i;
        }
    }
    // Perform a second pass for handling neighbours
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        // findNeighbors()
        std::array<unsigned int, lbmDirec> neighbourCoord;
        for (int j = 1; j < lbmDirec; ++j) {
            neighbourCoord[j] = h_nodes.coord[i] + PARAMS.ne[j];
        }
        // Boundary conditions
        if (h_nodes.isWall(i)) {
            const std::array<int, 3> pos = h_nodes.getGridPosition(i);// getPosition() adds + 0.5x
            if (pos[0] == 0) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Xp) < 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            } else if (pos[0] == static_cast<int>(PARAMS.lbSize[0] - 1)) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Xp) > 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            }
            if (pos[1] == 0) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Yp) < 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            } else if (pos[1] == static_cast<int>(PARAMS.lbSize[1] - 1)) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Yp) > 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            }
            if (pos[2] == 0) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Zp) < 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            } else if (pos[2] == static_cast<int>(PARAMS.lbSize[2] - 1)) {
                for (unsigned int j = 1; j < lbmDirec; ++j) {
                    if (v[j].dot(Zp) > 0.0) {
                        neighbourCoord[j] = h_nodes.coord[i];
                    }
                }
            }
        } else if (h_nodes.isActive(i)) {
            /*
            // PERIODICITY ////////////////////////////////////////////////
            // assigning periodicity conditions (this needs to be done after applying boundary conditions)
            // runs through free cells and identifies neighboring cells. If neighbor cell is
            // a special cell (periodic) then the proper neighboring condition is applied
            // calculates the effect of periodicity
            */
            // 
            // neighboring and periodicity vector for boundary update
            std::array<unsigned int, lbmDirec> pbc = {};

            auto f = idIndexMap.find(neighbourCoord[1]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Xp) > 0.0) {
                            pbc[j] -= PARAMS.domain[0];
                        }
                    }
                }
            }
            f = idIndexMap.find(neighbourCoord[2]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Xp) < 0.0) {
                            pbc[j] += PARAMS.domain[0];
                        }
                    }
                }
            }
            f = idIndexMap.find(neighbourCoord[3]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Yp) > 0.0) {
                            pbc[j] -= PARAMS.domain[1];
                        }
                    }
                }
            }
            f = idIndexMap.find(neighbourCoord[4]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Yp) < 0.0) {
                            pbc[j] += PARAMS.domain[1];
                        }
                    }
                }
            }
            f = idIndexMap.find(neighbourCoord[5]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Zp) > 0.0) {
                            pbc[j] -= PARAMS.domain[2];
                        }
                    }
                }
            }
            f = idIndexMap.find(neighbourCoord[6]);
            if (f != idIndexMap.end()) {
                if (h_nodes.type[f->second] == PERIODIC) {
                    for (int j = 1; j < lbmDirec; ++j) {
                        if (v[j].dot(Zp) < 0.0) {
                            pbc[j] += PARAMS.domain[2];
                        }
                    }
                }
            }

            // apply periodicity
            for (int j = 1; j < lbmDirec; ++j) {
                neighbourCoord[j] += pbc[j];
            }
        }
        // assign neighbour nodes
        for (int j = 1; j < lbmDirec; ++j) {
            auto f = idIndexMap.find(neighbourCoord[j]);
            // check if node at that location exists
            if (f != idIndexMap.end()) {
                const unsigned int l_i = f->second;
                // assign neighbor for local node
                h_nodes.d[j * h_nodes.count + i] = l_i;
                // if neighbor node is also active, link it to local node
                if (h_nodes.isActive(i)) {
                    h_nodes.d[opp[j] * h_nodes.count + l_i] = i;
                    // if the neighbor is a curved wall, set parameters accordingly
                    if (h_nodes.type[l_i] == TOPO) {
                        if (h_nodes.curved[i] == std::numeric_limits<unsigned int>::max()) {
                            h_nodes.curved[i] = static_cast<unsigned int>(curves.size());
                            curves.emplace_back();
                        }
                        // set curved
                        const tVect nodePosHere = PARAMS.unit.Length * h_nodes.getPosition(i);
                        // xf - xw
                        const double topographyDistance = 1.0 * lbTop.directionalDistance(nodePosHere, vDirec[j]) / PARAMS.unit.Length;
                        // wall normal
                        curves.back().wallNormal = lbTop.surfaceNormal(nodePosHere);
                        //cout << topographyDistance << endl;
                        const double deltaHere = topographyDistance / vNorm[j];
                        curves.back().delta[j] = std::min(0.99, std::max(0.01, deltaHere));
                        curves.back().computeCoefficients();
                    }
                    if (h_nodes.isWall(l_i)) {
                        h_nodes.basal[i] = true;
                    }
                }
            } else {
                // Neighbour is gas
                h_nodes.d[j * h_nodes.count + i] = std::numeric_limits<unsigned int>::max();
            }
        }
    }

    // Formerly initializeVariables()    
    cout << "Initializing variables" << endl;
    // note that interface is not defined here. All fluid, interface and gas cells are uninitialized at the moment
    // calculate maximum height of the fluid

    // find "taller" and "deepest" points
    double minProjection = std::numeric_limits<double>::max();
    double maxProjection = -std::numeric_limits<double>::max();
        
    if (!PARAMS.solveCentrifugal) {
        for (unsigned int i = 0; i < h_nodes.count; ++i) {
            if (h_nodes.isActive(i)) {
                const tVect position = h_nodes.getPosition(i);
                const double projection = position.dot(PARAMS.lbF);
                minProjection = std::min(minProjection, projection);
                maxProjection = std::max(maxProjection, projection);
            }
        }
        cout << "minProjection = " << minProjection << endl;
    } else {
        for (unsigned int i = 0; i < h_nodes.count; ++i) {
            if (h_nodes.isActive(i)) {
                const tVect position = h_nodes.getPosition(i);
                const double projection = position.dot(h_nodes.centrifugalForce[i]);
                minProjection = std::min(minProjection, projection);
                maxProjection = std::max(maxProjection, projection);
            }
        }
        cout << "minProjection = " << minProjection << endl;
    }

    // checking for boundary between gas and fluid and assigning interface properties
    // at this point fluid cells contain actual fluid cells and potential interface cells, so we create the node anyway
    double massFluid = 0.0;
    double massInterface = 0.0;
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.type[i] == LIQUID) {
            // check if it is interface
            for (int j = 1; j < lbmDirec; ++j) {
                unsigned int linkNode = h_nodes.d[j * h_nodes.count + i];
                if (linkNode == std::numeric_limits<unsigned int>::max()) {
                    h_nodes.type[i] = INTERFACE;
                    break;
                }
            }
        }
        // now assign macroscopic quantities accordingly
        // FLUID NODES ////
        if (h_nodes.type[i] == LIQUID) {
            massFluid += 1.0;
            // setting macroscopic variables
            // density is calculated using hydrostatic profile
            const tVect position = h_nodes.getPosition(i);
            if (!PARAMS.solveCentrifugal) {
                const double projection = position.dot(PARAMS.lbF);
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity + 3.0 * PARAMS.fluidMaterial.initDensity * (projection-minProjection), PARAMS.initVelocity, PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, Zero);
            } else {
                const double projection = position.dot(h_nodes.centrifugalForce[i]);
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity + 3.0 * PARAMS.fluidMaterial.initDensity * (projection-minProjection), PARAMS.initVelocity, PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, PARAMS.rotationSpeed);
            }
        }// INTERFACE NODES ////
        else if (h_nodes.type[i] == INTERFACE) {
            massInterface += 0.5;
            // setting macroscopic variables
            h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, PARAMS.initVelocity, 0.5 * PARAMS.fluidMaterial.initDensity, PARAMS.fluidMaterial.initDynVisc, PARAMS.lbF, 1.0, PARAMS.rotationSpeed);
        }

    }
    cout << "Approximate volume = " << massFluid * PARAMS.unit.Volume << " (fluid body), " << massInterface * PARAMS.unit.Volume << " (interface), " << (massFluid + massInterface) * PARAMS.unit.Volume << " (tot), " << endl;
}
void LB2::initializeWalls() {
    cout << "Initializing wall nodes" << endl;
    const double zero = 0.0;

    std::vector<unsigned int> wallNodes;

    // initializing wall nodes
    // note that, in the hypothesis that these walls are not evolving, only nodes at the interface need creation
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.isWall(i)) {
            // initialize node
            // STATIC WALL NODES ////
            if (h_nodes.type[i] == STAT_WALL ||
                h_nodes.type[i] == SLIP_STAT_WALL ||
                h_nodes.type[i] ==  OBJ ||
                h_nodes.type[i] == TOPO) {
                // reset velocity and mass (useful for plotting)
                // density=0.0; velocity=(0.0,0.0,0.0), mass=0.0; viscosity=0.0; force=(0.0,0.0,0.0)
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, Zero, zero, zero, Zero, 1.0, Zero);
            }// DYNAMIC WALL NODES ////
            else if (h_nodes.type[i] == DYN_WALL || 
                     h_nodes.type[i] == SLIP_DYN_WALL || 
                     h_nodes.type[i] == CYL) {
                // need to define velocity. It could be part of a cylinder or wall, we check both
                tVect solidVelocity;
                const tVect nodePosition = h_nodes.getPosition(i);
                unsigned int solidIndex = h_nodes.solidIndex[i];
                // wall
                if (solidIndex < h_walls.count && nodePosition.insidePlane(h_walls.p[solidIndex] / PARAMS.unit.Length, h_walls.n[solidIndex])) {
                    solidVelocity = h_walls.getSpeed(solidIndex, nodePosition * PARAMS.unit.Length) / PARAMS.unit.Speed;
                }// cylinder
                else if (solidIndex < h_cylinders.count && !nodePosition.insideCylinder(h_cylinders.p1[solidIndex] / PARAMS.unit.Length, h_cylinders.naxes[solidIndex], 0.0, h_cylinders.R[solidIndex] / PARAMS.unit.Length)) {
                    solidVelocity = h_cylinders.getSpeed(solidIndex, nodePosition * PARAMS.unit.Length) / PARAMS.unit.Speed;
                }// objects
                else if (solidIndex < h_objects.count && nodePosition.insideSphere(h_objects.x0[solidIndex] / PARAMS.unit.Length, h_objects.r[solidIndex] / PARAMS.unit.Length)) {
                    solidVelocity = h_objects.x1[solidIndex] / PARAMS.unit.Speed;
                }
                // reset velocity and mass (useful for plotting)
                // density=0.0; velocity=solidVelocity, mass=0.0; viscosity=0.0; force=(0.0,0.0,0.0)
                h_nodes.initialize(i, PARAMS.fluidMaterial.initDensity, solidVelocity, zero, zero, Zero, 1.0, PARAMS.rotationSpeed);
            }
            // add node to list
            wallNodes.push_back(i);
        }
    }
    // Allocate the wall nodes storage
    h_nodes.wallCount = static_cast<unsigned int>(wallNodes.size());
    h_nodes.wallI = static_cast<unsigned int*>(malloc(h_nodes.wallCount * sizeof(unsigned int)));
    memcpy(h_nodes.wallI, wallNodes.data(), h_nodes.wallCount * sizeof(unsigned int));    
}
void LB2::initializeCurved(std::vector<curve> &curves) {
    cout << "Initializing curved boundaries" << endl;
    for (unsigned int i = 0; i < h_nodes.wallCount; ++i) {
        const unsigned int w_i = h_nodes.wallI[i];
        if (h_nodes.type[w_i] == CYL) {
            assert(h_nodes.curved[w_i] == std::numeric_limits<unsigned int>::max());
            h_nodes.curved[w_i] = static_cast<unsigned int>(curves.size());
            curves.emplace_back();
            const tVect nodePos = PARAMS.unit.Length * h_nodes.getPosition(w_i);
            for (int j = 1; j < lbmDirec; ++j) {
                curves.back().delta[j] = 1.0 - h_cylinders.segmentIntercept(0, nodePos, PARAMS.unit.Length * v[j]);
                curves.back().computeCoefficients();
            }
        }
    }
}
void LB2::initializeLists() {
    cout << "Resetting lists ...";

    // note that interface is not defined here. All fluid, interface and gas cells are 0 at the moment
    std::vector<unsigned int> fluidNodes;
    std::vector<unsigned int> interfaceNodes;

    // creating list and initialize macroscopic variables for all nodes except walls
    for (unsigned int i = 0; i < h_nodes.count; ++i) {
        if (h_nodes.type[i] == LIQUID) {
            fluidNodes.push_back(i);
        } else if (h_nodes.type[i] == INTERFACE) {
            interfaceNodes.push_back(i);
        }

    }

    // Array to Buffer
    assert(!h_nodes.fluidI);
    h_nodes.fluidCount = static_cast<unsigned int>(fluidNodes.size());
    h_nodes.fluidI = static_cast<unsigned int*>(malloc(h_nodes.fluidCount * sizeof(unsigned int)));
    memcpy(h_nodes.fluidI, fluidNodes.data(), h_nodes.fluidCount * sizeof(unsigned int));

    assert(!h_nodes.interfaceI);
    h_nodes.interfaceCount = static_cast<unsigned int>(interfaceNodes.size());
    h_nodes.interfaceI = static_cast<unsigned int*>(malloc(h_nodes.interfaceCount * sizeof(unsigned int)));
    memcpy(h_nodes.interfaceI, interfaceNodes.data(), h_nodes.interfaceCount * sizeof(unsigned int));

    // Build a sorted active nodes list
    fluidNodes.insert(fluidNodes.end(), interfaceNodes.begin(), interfaceNodes.end());
    std::sort(fluidNodes.begin(), fluidNodes.end());

    // Array to buffer
    assert(!h_nodes.activeI);
    h_nodes.activeCount = static_cast<unsigned int>(fluidNodes.size());
    h_nodes.activeI = static_cast<unsigned int*>(malloc(h_nodes.activeCount * sizeof(unsigned int)));
    memcpy(h_nodes.activeI, fluidNodes.data(), h_nodes.activeCount * sizeof(unsigned int));
    
    cout << " done" << endl;
}
void LB2::step(const DEM &dem, bool io_demSolver) {
    this->syncDEM(dem.elmts, dem.particles, dem.walls, dem.objects);

    if (io_demSolver) {
        this->latticeBoltzmannCouplingStep(dem.newNeighborList);
    }

    if (dem.demTime >= dem.demInitialRepeat) {
        this->latticeBoltzmannStep();

        // Lattice Boltzmann core steps @todo after latticeBoltzmannStep() has been tested
        // if (this->freeSurface) {
        //     this->latticeBoltzmannFreeSurfaceStep();
        // }
    }
}

void LB2::syncDEM(const elmtList &elmts, const particleList &particles, const wallList &walls, const objectList &objects) {
    // Sync DEM data to structure of arrays format (and device memory)
    syncElements<IMPL>(elmts);
    syncParticles<IMPL>(particles);
    syncWalls<IMPL>(walls);
    syncObjects<IMPL>(objects);
}
void LB2::setParams(const LBParams& params, const LBInitParams& initParams, bool skip_sync) {
    // CPU
    h_PARAMS = params;
    init_params = initParams;
    // CUDA
    if (!skip_sync)
        syncParams();
}
void LB2::syncParams() {
#ifdef USE_CUDA
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PARAMS), &h_PARAMS, sizeof(LBParams)));
#endif
}
