#include "hip/hip_runtime.h"
#include "DEMParams.h"

#include <string>

#include "getpot.h"
#include "macros.h"
#include "Object2.h"
#include "Element2.h"

#ifdef USE_CUDA
__constant__ DEMParams d_DEM_P;
#endif
DEMParams h_DEM_P;

void DEMParams::discreteElementGet(GetPot& configFile, GetPot& commandLine, Element2& elmts, Object2& objects) {
    // getting material properties
    PARSE_CLASS_MEMBER(configFile, sphereMat.density, "particleDensity", 0.0);
    ASSERT(sphereMat.density > 0.0);

    std::string contactModelString;
    PARSE_CLASS_MEMBER(configFile, contactModelString, "contactModel", "none");
    if (contactModelString == "HERTZIAN") sphereMat.contactModel = HERTZIAN;
    else if (contactModelString == "LINEAR") sphereMat.contactModel = LINEAR;


    // Hertzian contact model /////////////
    PARSE_CLASS_MEMBER(configFile, sphereMat.youngMod, "youngMod", 1.0);
    ASSERT(sphereMat.youngMod > 0.0);
    PARSE_CLASS_MEMBER(configFile, sphereMat.poisson, "poisson", 0.3);
    ASSERT(sphereMat.poisson >= 0.0);
    // normal stiffness (constant part, calculated here to avoid repetition)
    sphereMat.knConst = 2.0 / 3.0 * sphereMat.youngMod / (1.0 - sphereMat.poisson * sphereMat.poisson);
    // there was a bracket mistake here
    sphereMat.ksConst = 2.0 * sphereMat.youngMod / (2.0 - sphereMat.poisson) / (1.0 + sphereMat.poisson);

    // linear contact model /////////////
    PARSE_CLASS_MEMBER(configFile, sphereMat.linearStiff, "linearStiff", 1.0);
    ASSERT(sphereMat.linearStiff >= 0.0);

    // normal damping ///////////////////////
    PARSE_CLASS_MEMBER(configFile, sphereMat.restitution, "restitution", 1.0);
    ASSERT(sphereMat.restitution > 0.0);
    ASSERT(sphereMat.restitution <= 1.0);
    // calculating coefficient for normal damping
    switch (sphereMat.contactModel) {
        case HERTZIAN:
        {
            // see "On the Determination of the Damping Coefficient of Non-linear Spring-dashpot System to Model Hertz Contact for Simulation by Discrete Element Method"
            // Hu, Hu, Jian, Liu, Wan, Journal of Computers, 6 (2011) OR BETTER Antypov & Elliott
            sphereMat.dampCoeff = -1.0 * sqrt(5) * log(sphereMat.restitution) / sqrt((log(sphereMat.restitution) * log(sphereMat.restitution) + M_PI * M_PI));
            break;
        }
        case LINEAR:
        {
            //sphereMat.dampCoeff=-1.0*sqrt(2.0)*log(sphereMat.restitution)/sqrt((log(sphereMat.restitution)*log(sphereMat.restitution)+M_PI));
            sphereMat.dampCoeff = -1.0 * log(sphereMat.restitution) / sqrt((log(sphereMat.restitution) * log(sphereMat.restitution) + M_PI * M_PI));
            ASSERT(sphereMat.dampCoeff < 1.0);
            break;
        }
    }

    // tangential model //////////////////////
    PARSE_CLASS_MEMBER(configFile, sphereMat.viscTang, "viscTang", 0.0);
    ASSERT(sphereMat.viscTang >= 0.0);
    PARSE_CLASS_MEMBER(configFile, sphereMat.frictionCoefPart, "frictionCoefPart", 0.0);
    ASSERT(sphereMat.frictionCoefPart >= 0.0);
    PARSE_CLASS_MEMBER(configFile, sphereMat.frictionCoefWall, "frictionCoefWall", 0.0);
    ASSERT(sphereMat.frictionCoefWall >= 0.0);
    PARSE_CLASS_MEMBER(configFile, sphereMat.frictionCoefObj, "frictionCoefObj", 0.0);
    if (sphereMat.frictionCoefObj == 0.0) {
        sphereMat.frictionCoefObj = sphereMat.frictionCoefWall;
    }
    // rolling model //////////////////////
    PARSE_CLASS_MEMBER(configFile, sphereMat.rollingCoefPart, "rollingCoefPart", 0.0);
    ASSERT(sphereMat.rollingCoefPart >= 0.0);

    // particle initial state //////////////////////
    std::string particleFile;
    PARSE_CLASS_MEMBER(configFile, particleFile, "particleFile", "particles.dat");
    double translateX(0.0), translateY(0.0), translateZ(0.0);
    PARSE_CLASS_MEMBER(configFile, translateX, "particleTranslateX", 0.0);
    PARSE_CLASS_MEMBER(configFile, translateY, "particleTranslateY", 0.0);
    PARSE_CLASS_MEMBER(configFile, translateZ, "particleTranslateZ", 0.0);
    tVect translate(translateX, translateY, translateZ);
    double scale = 1.0;
    PARSE_CLASS_MEMBER(configFile, scale, "particleScale", 1.0);

    ifstream particleFileID;
    particleFileID.open(particleFile.c_str(), ios::in);
    cout << "Reading " << particleFile.c_str() << "...";
    ASSERT(particleFileID.is_open());
    particleFileID>>elmts.count;
    // Allocate memory for elements
    elmts.memoryAlloc<CPU>(elmts.count);
    elmts.componentsIndex[0] = 0;
    for (int n = 0; n < elmts.count; ++n) {
        elmt dummyElmt;

        // import variables
        particleFileID >> elmts.index[n];
        particleFileID >> elmts.size[n];
        // Calculate components index
        elmts.componentsIndex[n + 1] = elmts.componentsIndex[n] + elmts.size[n];
        particleFileID >> elmts.radius[n];
        elmts.radius[n] = elmts.radius[n] * scale;
        // position
        particleFileID>>elmts.x0[n].x;
        particleFileID>>elmts.x0[n].y;
        particleFileID>>elmts.x0[n].z;
        elmts.x0[n] *= scale;
        elmts.x0[n] += translate;
        // translational velocity
        particleFileID>>elmts.x1[n].x;
        particleFileID>>elmts.x1[n].y;
        particleFileID>>elmts.x1[n].z;
        // rotational velocity
        particleFileID>>elmts.w0[n].x;
        particleFileID>>elmts.w0[n].y;
        particleFileID>>elmts.w0[n].z;
        // orientation
        particleFileID>>elmts.q0[n].q0;
        particleFileID>>elmts.q0[n].q1;
        particleFileID>>elmts.q0[n].q2;
        particleFileID>>elmts.q0[n].q3;
        // translational velocity (in quaternion rates))
        particleFileID>>elmts.q1[n].q0;
        particleFileID>>elmts.q1[n].q1;
        particleFileID>>elmts.q1[n].q2;
        particleFileID>>elmts.q1[n].q3;
        elmts.active[n] = true;
    }
    // Allocate memory for componentsData
    elmts.allocComponentsData();
    cout << " done" << endl;

    // objects initial state //////////////////////
    string objectFile;
    PARSE_CLASS_MEMBER(configFile, objectFile, "objectFile", "objects.dat");
    ifstream objectFileID;
    objectFileID.open(objectFile.c_str(), ios::in);
    ASSERT(objectFileID.is_open());
    cout << "Reading " << objectFile.c_str() << "...";
    objectFileID>>objects.count;
    // Allocate memory for elements
    objects.allocObjects<CPU>(objects.count);
    for (int n = 0; n < objects.count; ++n) {
        // import variables
        //objectFileID >> objects.index[n];
        double trash;
        objectFileID >> trash; // @note index is no longer stored
        // this is used to identify objects belonging to different groups
        objectFileID >> objects.ElID[n]; // must be one
        objectFileID >> objects.r[n];
        objectFileID >> objects.x0[n].x;
        objectFileID >> objects.x0[n].y;
        objectFileID >> objects.x0[n].z;
        objectFileID >> objects.x1[n].x;
        objectFileID >> objects.x1[n].y;
        objectFileID >> objects.x1[n].z;
        // the next eight values are for rotation, and are not used
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        objectFileID>>trash;
        //objects.originalIndex[n] = objects.index[n]; // @note originalIndex/index are no longer stored
    }
    cout << " done" << endl;

    // numerical viscosity for stability
    PARSE_CLASS_MEMBER(configFile, numVisc, "numVisc", 0.0);
    // set multiplication number (sets the number of DEM steps between two fluid steps)
    PARSE_CLASS_MEMBER(configFile, multiStep, "multiStep", 1);
    // set ratio between time step and estimated duration of contacts (only if multiStep=0)
    PARSE_CLASS_MEMBER(configFile, criticalRatio, "criticalRatio", 0.1);
}


void DEMParams::init_prototypeC1C2() {
    c1 = { deltat, deltat * deltat / 2.0, deltat * deltat * deltat / 6.0, deltat * deltat * deltat * deltat / 24.0, deltat * deltat * deltat * deltat * deltat / 120.0 };
    c2 = { deltat, deltat * deltat / 2.0, deltat * deltat * deltat / 6.0, deltat * deltat * deltat * deltat / 24.0, deltat * deltat * deltat * deltat * deltat / 120.0 };
    //    c[0] = deltat;
    //    c[1] = c[0] * deltat / 2.0;
    //    c[2] = c[1] * deltat / 3.0;
    //    c[3] = c[2] * deltat / 4.0;
    //    c[4] = c[3] * deltat / 5.0;
}